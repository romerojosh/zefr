#include "hip/hip_runtime.h"
/* Copyright (C) 2016 Aerospace Computing Laboratory (ACL).
 * See AUTHORS for contributors to this source code.
 *
 * This file is part of ZEFR.
 *
 * ZEFR is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ZEFR is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with ZEFR.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "elements_kernels.h"
#include "flux.hpp"
#include "input.hpp"
#include "mdvector_gpu.h"

#define HOLE 0
#define FRINGE -1
#define NORMAL 1

static const unsigned int MAX_GRID_DIM = 65535;

__device__
double determinant(double* mat, unsigned int M)
{
  double Det = 0;

  switch(M)
  {
    case 0:
      break;

    case 1:
      Det = mat[0];
      break;

    case 2:
      Det = mat[0]*mat[M+1] - mat[1]*mat[M*1];
      break;

    default:
    {
      unsigned int N = M;
      // Use minor-matrix recursion

      int sign = -1;
      double *Minor = new double[(M-1)*(M-1)];

      for (int row = 0; row < M; row++)
      {
        sign *= -1;
        // Setup the minor matrix (expanding along first column)
        int i0 = 0;
        for (int i = 0; i < M; i++)
        {
          if (i == row) continue;
          for (int j = 1; j < N; j++)
          {
            Minor[(M-1)*i0+j-1] = mat[M*i+j];
          }
          i0++;
        }
        // Add in the minor's determinant
        Det += sign*determinant(Minor,M-1)*mat[M*row+0];
      }

      delete[] Minor;

      break;
    }
  }
  return Det;
}

__device__
void device_adjoint(double *mat, double *adj, int M)
{
  unsigned int N = M;

  int signRow = -1;
  double *Minor = new double[(M-1)*(M-1)];

  for (int row = 0; row < M; row++)
  {
    signRow *= -1;
    int sign = -1*signRow;
    for (int col = 0; col < N; col++)
    {
      sign *= -1;
      // Setup the minor matrix (expanding along row, col)
      int i0 = 0;
      for (int i = 0; i < M; i++)
      {
        if (i == row) continue;
        int j0 = 0;
        for (int j = 0; j < N; j++)
        {
          if (j == col) continue;
          Minor[(M-1)*i0+j0] = mat[M*i+j];
          j0++;
        }
        i0++;
      }
      // Recall: adjoint is TRANSPOSE of cofactor matrix
      adj[M*col+row] = sign*determinant(Minor,M-1);
    }
  }

  delete[] Minor;
}

__device__
void device_adjoint_4x4(double *mat, double *adj)
{
  double a11 = mat[0],  a12 = mat[1],  a13 = mat[2],  a14 = mat[3];
  double a21 = mat[4],  a22 = mat[5],  a23 = mat[6],  a24 = mat[7];
  double a31 = mat[8],  a32 = mat[9],  a33 = mat[10], a34 = mat[11];
  double a41 = mat[12], a42 = mat[13], a43 = mat[14], a44 = mat[15];

  adj[0] = -a24*a33*a42 + a23*a34*a42 + a24*a32*a43 - a22*a34*a43 - a23*a32*a44 + a22*a33*a44;
  adj[1] =  a14*a33*a42 - a13*a34*a42 - a14*a32*a43 + a12*a34*a43 + a13*a32*a44 - a12*a33*a44;
  adj[2] = -a14*a23*a42 + a13*a24*a42 + a14*a22*a43 - a12*a24*a43 - a13*a22*a44 + a12*a23*a44;
  adj[3] =  a14*a23*a32 - a13*a24*a32 - a14*a22*a33 + a12*a24*a33 + a13*a22*a34 - a12*a23*a34;

  adj[4] =  a24*a33*a41 - a23*a34*a41 - a24*a31*a43 + a21*a34*a43 + a23*a31*a44 - a21*a33*a44;
  adj[5] = -a14*a33*a41 + a13*a34*a41 + a14*a31*a43 - a11*a34*a43 - a13*a31*a44 + a11*a33*a44;
  adj[6] =  a14*a23*a41 - a13*a24*a41 - a14*a21*a43 + a11*a24*a43 + a13*a21*a44 - a11*a23*a44;
  adj[7] = -a14*a23*a31 + a13*a24*a31 + a14*a21*a33 - a11*a24*a33 - a13*a21*a34 + a11*a23*a34;

  adj[8] = -a24*a32*a41 + a22*a34*a41 + a24*a31*a42 - a21*a34*a42 - a22*a31*a44 + a21*a32*a44;
  adj[9] =  a14*a32*a41 - a12*a34*a41 - a14*a31*a42 + a11*a34*a42 + a12*a31*a44 - a11*a32*a44;
  adj[10]= -a14*a22*a41 + a12*a24*a41 + a14*a21*a42 - a11*a24*a42 - a12*a21*a44 + a11*a22*a44;
  adj[11]=  a14*a22*a31 - a12*a24*a31 - a14*a21*a32 + a11*a24*a32 + a12*a21*a34 - a11*a22*a34;

  adj[12]=  a23*a32*a41 - a22*a33*a41 - a23*a31*a42 + a21*a33*a42 + a22*a31*a43 - a21*a32*a43;
  adj[13]= -a13*a32*a41 + a12*a33*a41 + a13*a31*a42 - a11*a33*a42 - a12*a31*a43 + a11*a32*a43;
  adj[14]=  a13*a22*a41 - a12*a23*a41 - a13*a21*a42 + a11*a23*a42 + a12*a21*a43 - a11*a22*a43;
  adj[15]= -a13*a22*a31 + a12*a23*a31 + a13*a21*a32 - a11*a23*a32 - a12*a21*a33 + a11*a22*a33;
}

template<unsigned int nVars, unsigned int nDims, unsigned int equation>
__global__
void compute_F(mdvector_gpu<double> F_spts, 
    const mdvector_gpu<double> U_spts, mdvector_gpu<double> dU_spts, const mdvector_gpu<double> Vg_spts,
    const mdvector_gpu<double> inv_jaco_spts, const mdvector_gpu<double> jaco_det_spts, unsigned int nSpts, unsigned int nEles,
    const mdvector_gpu<double> AdvDiff_A, double AdvDiff_D, double gamma,
    double prandtl, double mu_in, double c_sth, double rt, bool fix_vis, bool viscous, bool grad_via_div,
    bool overset = false, const int* iblank = NULL, bool motion = false)
{

  const unsigned int ele = (blockDim.x * blockIdx.x + threadIdx.x);
  const unsigned int spt = (blockDim.y * blockIdx.y + threadIdx.y);

  if (ele >= nEles || spt >= nSpts) 
    return;

  if (overset && iblank[ele] != 1)
      return;

  double U[nVars];
  double tdU[nVars][nDims];
  double dU[nVars][nDims];
  double F[nVars][nDims];
  double inv_jaco[nDims][nDims];
  double tF[nVars][nDims];
  double Vg[nDims] = {0.0};

  //for (unsigned int spt = 0; spt < nSpts; spt++)
  {

    /* Get state variables and reference space gradients */
    for (unsigned int var = 0; var < nVars; var++)
      U[var] = U_spts(spt, var, ele);

    if (viscous) 
    {
      for(unsigned int dim = 0; dim < nDims; dim++)
        for (unsigned int var = 0; var < nVars; var++)
          tdU[var][dim] = dU_spts(dim, spt, var, ele);
    }

    if (motion)
    {
      for (unsigned int dim = 0; dim < nDims; dim++)
        Vg[dim] = Vg_spts(spt, dim, ele);
    }

    /* Get metric terms */
    for (int dim1 = 0; dim1 < nDims; dim1++)
      for (int dim2 = 0; dim2 < nDims; dim2++)
        inv_jaco[dim1][dim2] = inv_jaco_spts(dim1, spt, dim2, ele);

    if (viscous)
    {
      /* Transform gradient to physical space */
      double inv_jaco_det = 1.0 / jaco_det_spts(spt,ele);

      for (unsigned int var = 0; var < nVars; var++)
      {
        for (int dim1 = 0; dim1 < nDims; dim1++)
        {
          if (!grad_via_div)
          {
            dU[var][dim1] = (tdU[var][0] * inv_jaco[0][dim1]);
            for (int dim2 = 1; dim2 < nDims; dim2++)
              dU[var][dim1] += (tdU[var][dim2] * inv_jaco[dim2][dim1]);

            dU[var][dim1] *= inv_jaco_det;

          }
          else
          {
            dU[var][dim1] = tdU[var][dim1] * inv_jaco_det;
          }

          /* Write physical gradient to global memory */
          dU_spts(dim1, spt, var, ele) = dU[var][dim1];

        }
      }
    }

    /* Compute fluxes */
    if (equation == AdvDiff)
    {
      double A[nDims];
      for(unsigned int dim = 0; dim < nDims; dim++)
        A[dim] = AdvDiff_A(dim);

      compute_Fconv_AdvDiff<nVars, nDims>(U, F, A, Vg);
      if(viscous) 
        compute_Fvisc_AdvDiff_add<nVars, nDims>(dU, F, AdvDiff_D);

    }
    else if (equation == EulerNS)
    {
      double P;
      compute_Fconv_EulerNS<nVars, nDims>(U, F, Vg, P, gamma);
      if(viscous) 
        compute_Fvisc_EulerNS_add<nVars, nDims>(U, dU, F, gamma, prandtl, mu_in,
          rt, c_sth, fix_vis);
    }

    /* Transform flux to reference space */
    for (unsigned int var = 0; var < nVars; var++)
    {
      for (unsigned int dim1 = 0; dim1 < nDims; dim1++)
      {
        tF[var][dim1] = F[var][0] * inv_jaco[dim1][0];

        for (unsigned int dim2 = 1; dim2 < nDims; dim2++)
        {
          tF[var][dim1] += F[var][dim2] * inv_jaco[dim1][dim2];
        }
      }
    }

    /* Write out transformed fluxes */
    for(unsigned int dim = 0; dim < nDims; dim++)
    {
      for (unsigned int var = 0; var < nVars; var++)
      {
        F_spts(dim, spt, var, ele) = tF[var][dim];
      }
    }
  }
}

void compute_F_wrapper(mdvector_gpu<double> &F_spts, 
    mdvector_gpu<double> &U_spts, mdvector_gpu<double> &dU_spts, mdvector_gpu<double> &grid_vel_spts,
    mdvector_gpu<double> &inv_jaco_spts, mdvector_gpu<double> &jaco_det_spts,
    unsigned int nSpts, unsigned int nEles, unsigned int nDims,
    unsigned int equation, mdvector_gpu<double> &AdvDiff_A, double AdvDiff_D, double gamma,
    double prandtl, double mu_in, double c_sth, double rt, bool fix_vis, bool viscous,
    bool grad_via_div, bool overset, int* iblank, bool motion)
{
  //unsigned int threads = 128;
  //unsigned int blocks = (nEles + threads - 1)/threads;
  dim3 threads(32, 4);
  dim3 blocks((nEles + threads.x - 1)/threads.x, (nSpts + threads.y -1)/threads.y);

  if (equation == AdvDiff)
  {
    if (nDims == 2)
      compute_F<1, 2, AdvDiff><<<blocks, threads>>>(F_spts, U_spts, dU_spts, grid_vel_spts, inv_jaco_spts, jaco_det_spts, nSpts, nEles, AdvDiff_A,
          AdvDiff_D, gamma, prandtl, mu_in, c_sth, rt, fix_vis, viscous, grad_via_div, overset, iblank, motion);
    else if (nDims == 3)
      compute_F<1, 3, AdvDiff><<<blocks, threads>>>(F_spts, U_spts, dU_spts, grid_vel_spts, inv_jaco_spts, jaco_det_spts, nSpts, nEles, AdvDiff_A,
          AdvDiff_D, gamma, prandtl, mu_in, c_sth, rt, fix_vis, viscous, grad_via_div, overset, iblank, motion);
  }
  else if (equation == EulerNS)
  {
    if (nDims == 2)
      compute_F<4, 2, EulerNS><<<blocks, threads>>>(F_spts, U_spts, dU_spts, grid_vel_spts, inv_jaco_spts, jaco_det_spts, nSpts, nEles, AdvDiff_A,
          AdvDiff_D, gamma, prandtl, mu_in, c_sth, rt, fix_vis, viscous, grad_via_div, overset, iblank, motion);
    else if (nDims == 3)
      compute_F<5, 3, EulerNS><<<blocks, threads>>>(F_spts, U_spts, dU_spts, grid_vel_spts, inv_jaco_spts, jaco_det_spts, nSpts, nEles, AdvDiff_A,
          AdvDiff_D, gamma, prandtl, mu_in, c_sth, rt, fix_vis, viscous, grad_via_div, overset, iblank, motion);
  }

}

template <unsigned int nDims, unsigned int nVars>
__global__
void common_U_to_F(mdvector_gpu<double> Fcomm, mdvector_gpu<double> Ucomm, mdvector_gpu<double> norm_fpts, 
    mdvector_gpu<double> dA_fpts, unsigned int nEles, unsigned int nFpts, unsigned int dim)
{
  const unsigned int ele = (blockDim.x * blockIdx.x + threadIdx.x);
  const unsigned int fpt = (blockDim.y * blockIdx.y + threadIdx.y);

  if (ele >= nEles || fpt >= nFpts) 
    return;

  double n = norm_fpts(dim, fpt, ele);
  double dA = dA_fpts(fpt, ele); 

  for (unsigned int var = 0; var < nVars; var++)
  {
    Fcomm(fpt, var, ele) = Ucomm(fpt, var, ele) * n * dA;
  }
}

void common_U_to_F_wrapper(mdvector_gpu<double> &Fcomm, mdvector_gpu<double> &Ucomm, mdvector_gpu<double> &norm_fpts, 
    mdvector_gpu<double> &dA_fpts, unsigned int nEles, unsigned int nFpts, unsigned int nVars, unsigned int nDims, unsigned int equation,
    unsigned int dim)
{
  dim3 threads(32, 4);
  dim3 blocks((nEles + threads.x - 1)/threads.x, (nFpts + threads.y -1)/threads.y);

  if (equation == AdvDiff)
  {
    if (nDims == 2)
      common_U_to_F<2, 1><<<blocks, threads>>>(Fcomm, Ucomm, norm_fpts, dA_fpts, nEles, nFpts, dim);
    else
      common_U_to_F<3, 1><<<blocks, threads>>>(Fcomm, Ucomm, norm_fpts, dA_fpts, nEles, nFpts, dim);
  }
  else if (equation == EulerNS)
  {
    if (nDims == 2)
      common_U_to_F<2, 4><<<blocks, threads>>>(Fcomm, Ucomm, norm_fpts, dA_fpts, nEles, nFpts, dim);
    else
      common_U_to_F<3, 5><<<blocks, threads>>>(Fcomm, Ucomm, norm_fpts, dA_fpts, nEles, nFpts, dim);
  }

}

template<unsigned int nVars, unsigned int nDims>
__global__
void compute_unit_advF(mdvector_gpu<double> F_spts, mdvector_gpu<double> U_spts, mdvector_gpu<double> inv_jaco_spts,
    unsigned int nSpts, unsigned int nEles, unsigned int dim)
{

  const unsigned int ele = (blockDim.x * blockIdx.x + threadIdx.x);
  const unsigned int spt = (blockDim.y * blockIdx.y + threadIdx.y);

  if (ele >= nEles || spt >= nSpts) 
    return;

  double U[nVars];
  double inv_jaco[nDims];

  /* Get state variables */
  for (unsigned int var = 0; var < nVars; var++)
  {
    U[var] = U_spts(spt, var, ele);
  }

  /* Get required metric terms */
  for (unsigned int dim1 = 0; dim1 < nDims; dim1++)
  {
      inv_jaco[dim1] = inv_jaco_spts(dim1, spt, dim, ele);
  }

  /* Compute transformed unit advection flux along provided dimension */
  for (unsigned int var = 0; var < nVars; var++)
  {
    for (unsigned int dim1 = 0; dim1 < nDims; dim1++)
    {
        F_spts(dim1, spt, var, ele) = U[var] * inv_jaco[dim1];
    }
  }
}

void compute_unit_advF_wrapper(mdvector_gpu<double>& F_spts, mdvector_gpu<double>& U_spts, mdvector_gpu<double>& inv_jaco_spts, 
    unsigned int nSpts, unsigned int nEles, unsigned int nDims, unsigned int equation, unsigned int dim)
{
  dim3 threads(32, 4);
  dim3 blocks((nEles + threads.x - 1)/threads.x, (nSpts + threads.y -1)/threads.y);

  if (equation == AdvDiff)
  {
    if (nDims == 2)
      compute_unit_advF<1, 2><<<blocks, threads>>>(F_spts, U_spts, inv_jaco_spts, nSpts, nEles, dim);
    else if (nDims == 3)
      compute_unit_advF<1, 3><<<blocks, threads>>>(F_spts, U_spts, inv_jaco_spts, nSpts, nEles, dim);
  }
  else if (equation == EulerNS)
  {
    if (nDims == 2)
      compute_unit_advF<4, 2><<<blocks, threads>>>(F_spts, U_spts, inv_jaco_spts, nSpts, nEles, dim);
    else if (nDims == 3)
      compute_unit_advF<5, 3><<<blocks, threads>>>(F_spts, U_spts, inv_jaco_spts, nSpts, nEles, dim);
  }
}


__global__
void compute_inv_Jac_fpts(mdvector_gpu<double> LHS, mdvector_gpu<double> oppDiv_fpts, 
    mdvector_gpu<double> oppE, mdvector_gpu<double> dFcdU, unsigned int nSpts, unsigned int nFpts, 
    unsigned int nVars, unsigned int nEles)
{
  const unsigned int tidx = blockIdx.x * blockDim.x  + threadIdx.x;
  const unsigned int tidy = blockIdx.y * blockDim.y  + threadIdx.y;

  for (unsigned int elevarj = tidy; elevarj < nEles * nVars; elevarj += gridDim.y * blockDim.y)
  {
    const unsigned int ele = elevarj / nVars;
    const unsigned int varj = elevarj % nVars;

    for (unsigned int varispti = tidx; varispti < nSpts * nVars; varispti += blockDim.x)
    {
      const unsigned int vari = varispti / nSpts;
      const unsigned int spti = varispti % nSpts;

      for (unsigned int sptj = 0; sptj < nSpts; sptj++)
      {
        double sum = 0.0;
        for (unsigned int fptk = 0; fptk < nFpts; fptk++)
          sum += oppDiv_fpts(spti, fptk) * dFcdU(ele, vari, varj, fptk) * oppE(fptk, sptj);
        LHS(ele, varj, sptj, vari, spti) = sum;
      }
    }

    __syncthreads(); /* To avoid divergence */
  }
}

void compute_inv_Jac_fpts_wrapper(mdvector_gpu<double> LHS, mdvector_gpu<double> oppDiv_fpts, 
    mdvector_gpu<double> oppE, mdvector_gpu<double> dFcdU, unsigned int nSpts, unsigned int nFpts, 
    unsigned int nVars, unsigned int nEles)
{
  dim3 threads(32, 6);
  dim3 blocks(1, std::min((nVars * nEles + threads.y - 1) / threads.y, MAX_GRID_DIM));

  compute_inv_Jac_fpts<<<blocks, threads>>>(LHS, oppDiv_fpts, oppE, dFcdU, nSpts, nFpts, nVars, nEles);
}

__global__
void compute_inv_Jac_spts(mdvector_gpu<double> LHS, mdvector_gpu<double> oppD, 
    mdvector_gpu<double> dFdU_spts, unsigned int nSpts, unsigned int nVars, unsigned int nEles,
    unsigned int nDims)
{
  const unsigned int tidx = blockIdx.x * blockDim.x  + threadIdx.x;
  const unsigned int tidy = blockIdx.y * blockDim.y  + threadIdx.y;

  for (unsigned int elevarj = tidy; elevarj < nEles * nVars; elevarj += gridDim.y * blockDim.y)
  {
    const unsigned int ele = elevarj / nVars;
    const unsigned int varj = elevarj % nVars;

    for (unsigned int varispti = tidx; varispti < nSpts * nVars; varispti += blockDim.x)
    {
      const unsigned int vari = varispti / nSpts;
      const unsigned int spti = varispti % nSpts;

      for (unsigned int sptj = 0; sptj < nSpts; sptj++)
        for (unsigned int dim = 0; dim < nDims; dim++)
          LHS(ele, varj, sptj, vari, spti) += oppD(dim, spti, sptj) * dFdU_spts(ele, vari, varj, dim, sptj);
    }

    __syncthreads(); /* To avoid divergence */
  }
}

void compute_inv_Jac_spts_wrapper(mdvector_gpu<double> &LHS, mdvector_gpu<double> &oppD, 
    mdvector_gpu<double> &dFdU_spts, unsigned int nSpts, unsigned int nVars, unsigned int nEles,
    unsigned int nDims)
{
  dim3 threads(32, 6);
  dim3 blocks(1, std::min((nVars * nEles + threads.y - 1) / threads.y, MAX_GRID_DIM));

  compute_inv_Jac_spts<<<blocks, threads>>>(LHS, oppD, dFdU_spts, nSpts, nVars, nEles, nDims);
}

__global__
void scale_Jac(mdvector_gpu<double> LHS, mdvector_gpu<double> jaco_det_spts, 
    unsigned int nSpts, unsigned int nVars, unsigned int nEles)
{
  const unsigned int tidx = blockIdx.x * blockDim.x  + threadIdx.x;
  const unsigned int tidy = blockIdx.y * blockDim.y  + threadIdx.y;

  for (unsigned int elevarj = tidy; elevarj < nEles * nVars; elevarj += gridDim.y * blockDim.y)
  {
    const unsigned int ele = elevarj / nVars;
    const unsigned int varj = elevarj % nVars;

    for (unsigned int varispti = tidx; varispti < nSpts * nVars; varispti += blockDim.x)
    {
      const unsigned int vari = varispti / nSpts;
      const unsigned int spti = varispti % nSpts;
      const double jaco_det = jaco_det_spts(spti, ele);

      for (unsigned int sptj = 0; sptj < nSpts; sptj++)
        LHS(ele, varj, sptj, vari, spti) /= jaco_det;
    }

    __syncthreads(); /* To avoid divergence */
  }
}

void scale_Jac_wrapper(mdvector_gpu<double> &LHS, mdvector_gpu<double> &jaco_det_spts, 
    unsigned int nSpts, unsigned int nVars, unsigned int nEles)
{
  dim3 threads(32, 6);
  dim3 blocks(1, std::min((nVars * nEles + threads.y - 1) / threads.y, MAX_GRID_DIM));

  scale_Jac<<<blocks, threads>>>(LHS, jaco_det_spts, nSpts, nVars, nEles);
}

template<unsigned int nVars, unsigned int nDims, unsigned int equation>
__global__
void compute_dFdU(mdvector_gpu<double> dFdU_spts, mdvector_gpu<double> dFddU_spts,
    const mdvector_gpu<double> U_spts, const mdvector_gpu<double> dU_spts,
    const mdvector_gpu<double> inv_jaco_spts, unsigned int nSpts, unsigned int nEles,
    const mdvector_gpu<double> AdvDiff_A, double AdvDiff_D, double gamma,
    double prandtl, double mu, bool viscous)
{
  const unsigned int ele = (blockDim.x * blockIdx.x + threadIdx.x);
  const unsigned int spt = (blockDim.y * blockIdx.y + threadIdx.y);

  if (ele >= nEles || spt >= nSpts) 
    return;

  double U[nVars];
  double dU[nVars][nDims];
  double dFdU[nVars][nVars][nDims] = {0};
  double dFddU[nVars][nVars][nDims][nDims] = {0};
  double inv_jaco[nDims][nDims];
  double tdFdU[nVars][nVars][nDims] = {{0.0}};

  /* Get state variables and physical space gradients */
  for (unsigned int var = 0; var < nVars; var++)
  {
    U[var] = U_spts(spt, var, ele);

    if(viscous) 
      for(unsigned int dim = 0; dim < nDims; dim++)
        dU[var][dim] = dU_spts(dim, spt, var, ele);
  }

  /* Compute flux derivatives */
  if (equation == AdvDiff)
  {
    double A[nDims];
    for(unsigned int dim = 0; dim < nDims; dim++)
      A[dim] = AdvDiff_A(dim);

    compute_dFdUconv_AdvDiff<nVars, nDims>(dFdU, A);
    if(viscous) compute_dFddUvisc_AdvDiff<nVars, nDims>(dFddU, AdvDiff_D);
  }
  else if (equation == EulerNS)
  {
    compute_dFdUconv_EulerNS<nVars, nDims>(U, dFdU, gamma);
    if(viscous)
    {
      compute_dFdUvisc_EulerNS_add<nVars, nDims>(U, dU, dFdU, gamma, prandtl, mu);
      compute_dFddUvisc_EulerNS<nVars, nDims>(U, dFddU, gamma, prandtl, mu);
    }
  }

  /* Get metric terms */
  for (int dim1 = 0; dim1 < nDims; dim1++)
    for (int dim2 = 0; dim2 < nDims; dim2++)
      inv_jaco[dim1][dim2] = inv_jaco_spts(dim1, spt, dim2, ele);

  /* Transform flux derivative to reference space */
  for (unsigned int vari = 0; vari < nVars; vari++)
    for (unsigned int varj = 0; varj < nVars; varj++)
      for (unsigned int dim1 = 0; dim1 < nDims; dim1++)
        for (unsigned int dim2 = 0; dim2 < nDims; dim2++)
          tdFdU[vari][varj][dim1] += dFdU[vari][varj][dim2] * inv_jaco[dim1][dim2];

  /* Write out transformed flux derivatives */
  for (unsigned int vari = 0; vari < nVars; vari++)
    for (unsigned int varj = 0; varj < nVars; varj++)
      for (unsigned int dim = 0; dim < nDims; dim++)
        dFdU_spts(ele, vari, varj, dim, spt) = tdFdU[vari][varj][dim];

  if(viscous)
    for (unsigned int vari = 0; vari < nVars; vari++)
      for (unsigned int varj = 0; varj < nVars; varj++)
        for (unsigned int dimi = 0; dimi < nDims; dimi++)
          for (unsigned int dimj = 0; dimj < nDims; dimj++)
            dFddU_spts(ele, dimi, dimj, vari, varj, spt) = dFddU[vari][varj][dimi][dimj];
}

void compute_dFdU_wrapper(mdvector_gpu<double> &dFdU_spts, mdvector_gpu<double> &dFddU_spts,
    mdvector_gpu<double> &U_spts, mdvector_gpu<double> &dU_spts,
    mdvector_gpu<double> &inv_jaco_spts, unsigned int nSpts, unsigned int nEles, unsigned int nDims,
    unsigned int equation, mdvector_gpu<double> &AdvDiff_A, double AdvDiff_D, double gamma,
    double prandtl, double mu, bool viscous)
{
  //unsigned int threads = 128;
  //unsigned int blocks = (nEles + threads - 1)/threads;
  dim3 threads(32, 4);
  dim3 blocks((nEles + threads.x - 1)/threads.x, (nSpts + threads.y -1)/threads.y);

  if (equation == AdvDiff)
  {
    if (nDims == 2)
      compute_dFdU<1, 2, AdvDiff><<<blocks, threads>>>(dFdU_spts, dFddU_spts, U_spts, dU_spts, inv_jaco_spts, nSpts, nEles, AdvDiff_A,
          AdvDiff_D, gamma, prandtl, mu, viscous);
    else if (nDims == 3)
      compute_dFdU<1, 3, AdvDiff><<<blocks, threads>>>(dFdU_spts, dFddU_spts, U_spts, dU_spts, inv_jaco_spts, nSpts, nEles, AdvDiff_A,
          AdvDiff_D, gamma, prandtl, mu, viscous);
  }
  else if (equation == EulerNS)
  {
    if (nDims == 2)
      compute_dFdU<4, 2, EulerNS><<<blocks, threads>>>(dFdU_spts, dFddU_spts, U_spts, dU_spts, inv_jaco_spts, nSpts, nEles, AdvDiff_A,
          AdvDiff_D, gamma, prandtl, mu, viscous);
    else if (nDims == 3)
      compute_dFdU<5, 3, EulerNS><<<blocks, threads>>>(dFdU_spts, dFddU_spts, U_spts, dU_spts, inv_jaco_spts, nSpts, nEles, AdvDiff_A,
          AdvDiff_D, gamma, prandtl, mu, viscous);
  }
}


__global__
void normal_flux(mdvector_gpu<double> tempF, mdvector_gpu<double> dFn,
    mdvector_gpu<double> norm, mdvector_gpu<double> dA,
    mdvector_gpu<int> fpt2gfpt, mdvector_gpu<char> fpt2slot, unsigned int nFpts,
    unsigned int nEles, unsigned int dim, unsigned int var)
{
  const unsigned int fpt = (blockDim.x * blockIdx.x + threadIdx.x) % nFpts;
  const unsigned int ele = (blockDim.x * blockIdx.x + threadIdx.x) / nFpts;

  if (ele >= nEles)
    return;

  int gfpt = fpt2gfpt(fpt,ele);
  int slot = fpt2slot(fpt,ele);

  if (gfpt < 0)
    return;

  double fac = (slot == 1) ? -1 : 1; // factor to negate normal if "right" element (slot = 1)
  dFn(fpt,ele,var) -= tempF(fpt,ele) * fac * norm(gfpt,dim) * dA(gfpt);
}

__global__
void compute_Uavg(mdvector_gpu<double> U_spts, 
    mdvector_gpu<double> Uavg, mdvector_gpu<double> jaco_det_spts, 
    mdvector_gpu<double> weights_spts, mdvector_gpu<double> vol, unsigned int nSpts, 
    unsigned int nEles, unsigned int nVars, unsigned int nDims, int order)
{
  const unsigned int ele = (blockDim.x * blockIdx.x + threadIdx.x);

  if (ele >= nEles)
    return;

  /* Compute average solution using quadrature */
  for (unsigned int n = 0; n < nVars; n++)
  {
    double sum = 0.0;

    for (unsigned int spt = 0; spt < nSpts; spt++)
    {
      sum += weights_spts(spt) * jaco_det_spts(spt, ele) * U_spts(spt, n, ele);
    }

    Uavg(n, ele) = sum / vol(ele); 

  }

}

void compute_Uavg_wrapper(mdvector_gpu<double> &U_spts, 
    mdvector_gpu<double> &Uavg, mdvector_gpu<double> &jaco_det_spts, 
    mdvector_gpu<double> &weights_spts, mdvector_gpu<double> &vol, unsigned int nSpts, 
    unsigned int nEles, unsigned int nVars, unsigned int nDims, int order)
{
  unsigned int threads = 128;
  unsigned int blocks = (nEles + threads - 1)/ threads;

  compute_Uavg<<<blocks, threads>>>(U_spts, Uavg, jaco_det_spts, weights_spts, vol, nSpts, nEles, nVars, nDims, order);

  check_error();
}

__global__
void poly_squeeze(mdvector_gpu<double> U_spts, 
    mdvector_gpu<double> U_fpts, mdvector_gpu<double> Uavg, 
    double gamma, double exps0, unsigned int nSpts, 
    unsigned int nFpts, unsigned int nEles, unsigned int nVars,
    unsigned int nDims)
{
  const unsigned int ele = (blockDim.x * blockIdx.x + threadIdx.x);

  if (ele >= nEles)
    return;

  double V[3]; 

  /* For each element, check for negative density at solution and flux points */
  double tol = 1e-10;

  bool negRho = false;
  double minRho = U_spts(0, 0, ele);

  for (unsigned int spt = 0; spt < nSpts; spt++)
  {
    if (U_spts(spt, 0, ele) < 0)
    {
      negRho = true;
      minRho = min(minRho, U_spts(spt, 0, ele));
    }
  }
  
  for (unsigned int fpt = 0; fpt < nFpts; fpt++)
  {
    if (U_fpts(fpt, 0, ele) < 0)
    {
      negRho = true;
      minRho = min(minRho, U_fpts(fpt, 0, ele));
    }
  }

  /* If negative density found, squeeze density */
  if (negRho)
  {
    double theta = (Uavg(0, ele) - tol) / (Uavg(0, ele) - minRho); 

    for (unsigned int spt = 0; spt < nSpts; spt++)
      U_spts(spt, 0, ele) = theta * U_spts(spt, 0, ele) + (1.0 - theta) * Uavg(0, ele);

    for (unsigned int fpt = 0; fpt < nFpts; fpt++)
      U_fpts(fpt, 0, ele) = theta * U_fpts(fpt, 0, ele) + (1.0 - theta) * Uavg(0, ele);
    
  }

  /* For each element, check for entropy loss */
  double minTau = 1.0;

  /* Get minimum tau value */
  for (unsigned int spt = 0; spt < nSpts; spt++)
  {
    double rho = U_spts(spt, 0, ele);
    double momF = 0.0;
    for (unsigned int dim = 0; dim < nDims; dim++)
      momF += U_spts(spt, dim + 1, ele) * U_spts(spt, dim + 1, ele);

    momF /= rho;
    double P = (gamma - 1.0) * (U_spts(spt, nDims + 1, ele) - 0.5 * momF);

    double tau = P - exps0 * pow(rho, gamma);
    minTau = min(minTau, tau);

  }
  
  for (unsigned int fpt = 0; fpt < nFpts; fpt++)
  {
    double rho = U_fpts(fpt, 0, ele);
    double momF = 0.0;
    for (unsigned int dim = 0; dim < nDims; dim++)
      momF += U_fpts(fpt, dim + 1, ele) * U_fpts(fpt, dim + 1, ele);

    momF /= U_fpts(fpt, 0, ele);
    double P = (gamma - 1.0) * (U_fpts(fpt, nDims + 1, ele) - 0.5 * momF);

    double tau = P - exps0 * pow(rho, gamma);
    minTau = min(minTau, tau);

  }

  /* If minTau is negative, squeeze solution */
  if (minTau < 0)
  {
    double rho = Uavg(0, ele);
    double Vsq = 0.0;
    for (unsigned int dim = 0; dim < nDims; dim++)
    {
      V[dim] = Uavg(dim + 1, ele) / rho;
      Vsq += V[dim] * V[dim];
    }

    double e = Uavg(nDims + 1, ele);
    double P = (gamma - 1.0) * (e - 0.5 * rho * Vsq);

    double eps = minTau / (minTau - P + exps0 * pow(rho, gamma));

//      if (P < input->exps0 * std::pow(rho, input->gamma))
//        std::cout << "Constraint violated. Lower CFL?" << std::endl;

    for (unsigned int spt = 0; spt < nSpts; spt++)
    {
      for (unsigned int n = 0; n < nVars; n++)
      {
        U_spts(spt, n, ele) = eps * Uavg(n, ele) + (1.0 - eps) * U_spts(spt, n, ele);
      }
    }

    for (unsigned int fpt = 0; fpt < nFpts; fpt++)
    {
      for (unsigned int n = 0; n < nVars; n++)
      {
        U_fpts(fpt, n, ele) = eps * Uavg(n, ele) + (1.0 - eps) * U_fpts(fpt, n, ele);
      }
    }
  }

}

void poly_squeeze_wrapper(mdvector_gpu<double> &U_spts, 
    mdvector_gpu<double> &U_fpts, mdvector_gpu<double> &Uavg, 
    double gamma, double exps0, unsigned int nSpts, 
    unsigned int nFpts, unsigned int nEles, unsigned int nVars,
    unsigned int nDims)
{
  unsigned int threads = 128;
  unsigned int blocks = (nEles + threads - 1)/ threads;

  poly_squeeze<<<blocks, threads>>>(U_spts, U_fpts, Uavg, gamma, exps0, nSpts, nFpts,
      nEles, nVars, nDims);

  check_error();
}

//! Copy node positions from GeoStruct's array to ele's array
__global__
void copy_coords_ele(mdvector_gpu<double> nodes, mdvector_gpu<double> g_nodes,
    mdvector_gpu<int> ele2node, unsigned int nEles, unsigned int nNodes)
{
  int node = (blockDim.x * blockIdx.x + threadIdx.x) % nNodes;
  int ele =  (blockDim.x * blockIdx.x + threadIdx.x) / nNodes;
  int dim = blockIdx.y;

  if (ele >= nEles)
    return;

  nodes(node, dim, ele) = g_nodes(ele2node(ele, node), dim);
}

//! Copy fpt positions from ele's array to face's array
template<int nDims>
__global__
void copy_coords_face(mdvector_gpu<double> coord, mdvector_gpu<double> e_coord,
    mdvector_gpu<int> fpt2gfpt, unsigned int nEles, unsigned int nFpts)
{
  int fpt = (blockDim.y * blockIdx.y + threadIdx.y);
  int ele =  (blockDim.x * blockIdx.x + threadIdx.x);

  if (ele >= nEles || fpt >= nFpts)
    return;

  int gfpt = fpt2gfpt(fpt,ele);

  if (gfpt < 0) return;

  for (int dim = 0; dim < nDims; dim++)
    coord(dim, gfpt) = e_coord(fpt, dim, ele);
}

void copy_coords_ele_wrapper(mdvector_gpu<double> &nodes,
    mdvector_gpu<double> &g_nodes, mdvector_gpu<int> &ele2node,
    unsigned int nNodes, unsigned int nEles, unsigned int nDims)
{
  int threads = 128;
  dim3 blocksE((nEles * nNodes + threads - 1) / threads, nDims);

  copy_coords_ele<<<blocksE,threads>>>(nodes, g_nodes, ele2node, nEles, nNodes);
}

void update_coords_wrapper(mdvector_gpu<double> &nodes,
    mdvector_gpu<double> &g_nodes,  mdvector_gpu<double> &shape_spts,
    mdvector_gpu<double> &shape_fpts, mdvector_gpu<double> &coord_spts,
    mdvector_gpu<double> &coord_fpts, mdvector_gpu<double> &coord_faces,
    mdvector_gpu<int> &ele2node, mdvector_gpu<int> &fpt2gfpt, unsigned int nSpts,
    unsigned int nFpts, unsigned int nNodes, unsigned int nEles,
    unsigned int nDims)
{
  int threads = 128;
  dim3 blocksE((nEles * nNodes + threads - 1) / threads, nDims);

  copy_coords_ele<<<blocksE,threads>>>(nodes, g_nodes, ele2node, nEles, nNodes);

  int m = nEles * nDims;
  int k = nNodes;
  int ns = nSpts;
  int nf = nFpts;

  double *A = nodes.data();

  double *Bs = shape_spts.data();
  double *Cs = coord_spts.data();

  cublasDGEMM_wrapper(m, ns, k, 1.0, A, nodes.ldim(), Bs, shape_spts.ldim(),
                      0.0, Cs, coord_spts.ldim());

  double *Bf = shape_fpts.data();
  double *Cf = coord_fpts.data();

  cublasDGEMM_wrapper(m, nf, k, 1.0, A, nodes.ldim(), Bf, shape_fpts.ldim(),
                      0.0, Cf, coord_fpts.ldim());

  dim3 threadsF(32, 4);
  dim3 blocksF((nEles + threadsF.x - 1) / threadsF.x, (nFpts + threadsF.y - 1) / threadsF.y);

  if (nDims == 3)
    copy_coords_face<3><<<blocksF,threadsF>>>(coord_faces, coord_fpts, fpt2gfpt, nEles, nFpts);
  else
    copy_coords_face<2><<<blocksF,threadsF>>>(coord_faces, coord_fpts, fpt2gfpt, nEles, nFpts);

  check_error();
}

template<unsigned int nDims>
__global__
void add_cg_offset(mdvector_gpu<double> nodes, mdvector_gpu<double> x_cg, unsigned int nNodes)
{
  int node = blockDim.x * blockIdx.x + threadIdx.x;

  if (node >= nNodes)
    return;

  for (unsigned int i = node; i < nNodes; i += gridDim.x * blockDim.x)
    for (unsigned int d = 0; d < nDims; d++)
      nodes(i,d) += x_cg(d);
}

template<unsigned int nDims>
__global__
void update_h_ref(mdvector_gpu<double> h_ref, mdvector_gpu<double> coord_fpts,
    unsigned int nEles, unsigned int nFpts, unsigned int nPts1D)
{
  int fpt = (blockDim.x * blockIdx.x + threadIdx.x) % nFpts;
  int ele = (blockDim.x * blockIdx.x + threadIdx.x) / nFpts;

  if (ele >= nEles)
    return;

  if (nDims == 2)
  {
    /* Some indexing to pair up opposing flux points in 2D (on Quad) */
    unsigned int idx = fpt % nPts1D;
    unsigned int fpt1 = fpt;
    unsigned int fpt2 = (fpt / nPts1D + 3) * nPts1D - idx - 1;

    double dx = coord_fpts(fpt1,0,ele) - coord_fpts(fpt2,0,ele);
    double dy = coord_fpts(fpt1,1,ele) - coord_fpts(fpt2,1,ele);
    double dist = std::sqrt(dx*dx + dy*dy);

    h_ref(fpt1, ele) = dist;
    h_ref(fpt2, ele) = dist;
  }
  else
  {
    /// TODO
  }
}

void update_h_ref_wrapper(mdvector_gpu<double> &h_ref,
    mdvector_gpu<double> &coord_fpts, unsigned int nEles, unsigned int nFpts,
    unsigned int nPts1D, unsigned int nDims)
{
  int threads = 128;
  int blocks = (nEles * nFpts + threads - 1) / threads;

  if (nDims == 2)
  {
    update_h_ref<2><<<blocks,threads>>>(h_ref, coord_fpts, nEles, nFpts, nPts1D);
  }
  else
  {
    update_h_ref<3><<<blocks,threads>>>(h_ref, coord_fpts, nEles, nFpts, nPts1D);
  }
}

__global__
void inverse_transform_quad(mdvector_gpu<double> jaco,
    mdvector_gpu<double> inv_jaco, double *jaco_det, int nEles, int nPts)
{
  int pt = (blockDim.x * blockIdx.x + threadIdx.x) % nPts;
  int ele = (blockDim.x * blockIdx.x + threadIdx.x) / nPts;

  if (ele >= nEles)
    return;

  // Determinant of transformation matrix
  if (jaco_det != NULL)
    jaco_det[pt+nPts*ele] = jaco(0,pt,0,ele)*jaco(1,pt,1,ele)-jaco(0,pt,1,ele)*jaco(1,pt,0,ele);

  // Inverse of transformation matrix (times its determinant)
  inv_jaco(0,pt,0,ele) = jaco(1,pt,1,ele);  inv_jaco(0,pt,1,ele) =-jaco(1,pt,0,ele);
  inv_jaco(1,pt,0,ele) =-jaco(0,pt,1,ele);  inv_jaco(1,pt,1,ele) = jaco(0,pt,0,ele);
}

__global__
void inverse_transform_hexa(mdvector_gpu<double> jaco,
    mdvector_gpu<double> inv_jaco, double* jaco_det, int nEles, int nPts)
{
  int pt = (blockDim.x * blockIdx.x + threadIdx.x) % nPts;
  int ele = (blockDim.x * blockIdx.x + threadIdx.x) / nPts;

  if (ele >= nEles)
    return;

  double xr = jaco(0,pt,0,ele);  double xs = jaco(1,pt,0,ele);  double xt = jaco(2,pt,0,ele);
  double yr = jaco(0,pt,1,ele);  double ys = jaco(1,pt,1,ele);  double yt = jaco(2,pt,1,ele);
  double zr = jaco(0,pt,2,ele);  double zs = jaco(1,pt,2,ele);  double zt = jaco(2,pt,2,ele);

  // Determinant of transformation matrix (not always needed)
  if (jaco_det != NULL)
    jaco_det[pt+nPts*ele] = xr*(ys*zt - yt*zs) - xs*(yr*zt - yt*zr) + xt*(yr*zs - ys*zr);

  // Inverse of transformation matrix (times its determinant)
  inv_jaco(0,pt,0,ele) = ys*zt - yt*zs;  inv_jaco(0,pt,1,ele) = xt*zs - xs*zt;  inv_jaco(0,pt,2,ele) = xs*yt - xt*ys;
  inv_jaco(1,pt,0,ele) = yt*zr - yr*zt;  inv_jaco(1,pt,1,ele) = xr*zt - xt*zr;  inv_jaco(1,pt,2,ele) = xt*yr - xr*yt;
  inv_jaco(2,pt,0,ele) = yr*zs - ys*zr;  inv_jaco(2,pt,1,ele) = xs*zr - xr*zs;  inv_jaco(2,pt,2,ele) = xr*ys - xs*yr;
}

void calc_transforms_wrapper(mdvector_gpu<double> &nodes, mdvector_gpu<double> &jaco_spts,
    mdvector_gpu<double> &jaco_fpts, mdvector_gpu<double> &inv_jaco_spts,
    mdvector_gpu<double> &inv_jaco_fpts, mdvector_gpu<double> &jaco_det_spts,
    mdvector_gpu<double> &dshape_spts, mdvector_gpu<double> &dshape_fpts,
    int nSpts, int nFpts, int nNodes, int nEles, int nDims)
{
  // Calculate forward transform (reference -> physical)
  int ms = nSpts * nDims;
  int mf = nFpts * nDims;
  int k = nNodes;
  int n = nEles * nDims;

  double* B = nodes.data();

  double *As = dshape_spts.data();
  double *Af = dshape_fpts.data();
  double *Cs = jaco_spts.data();
  double *Cf = jaco_fpts.data();

  cublasDGEMM_wrapper(n, ms, k, 1.0, B, n, As, k, 0.0, Cs, n);
  cublasDGEMM_wrapper(n, mf, k, 1.0, B, n, Af, k, 0.0, Cf, n);

  // Calculate inverse transform (physical -> reference) at spts, fpts
  int threads = 128;

  int blocksS = (nSpts * nEles + threads - 1) / threads;
  int blocksF = (nFpts * nEles + threads - 1) / threads;

  if (nDims == 2)
  {
    inverse_transform_quad<<<blocksS,threads>>>(jaco_spts,inv_jaco_spts,
        jaco_det_spts.data(),nEles,nSpts);

    inverse_transform_quad<<<blocksF,threads>>>(jaco_fpts,inv_jaco_fpts,
        NULL,nEles,nFpts);
  }
  else
  {
    inverse_transform_hexa<<<blocksS,threads>>>(jaco_spts,inv_jaco_spts,
        jaco_det_spts.data(),nEles,nSpts);

    inverse_transform_hexa<<<blocksF,threads>>>(jaco_fpts,inv_jaco_fpts,
        NULL,nEles,nFpts);
  }

  check_error();
}

template<unsigned int nDims>
__global__
void update_transform_rmat(mdvector_gpu<double> jaco_init, mdvector_gpu<double> jaco,
    mdvector_gpu<double> Rmat, unsigned int nEles, unsigned int nSpts)
{
  const int ele = (blockDim.x * blockIdx.x + threadIdx.x) % nEles;
  const int spt = (blockDim.x * blockIdx.x + threadIdx.x) / nEles;

  if (spt >= nSpts) return;

  double J[nDims][nDims] = {{0.0}};
  double R[nDims][nDims];

  for (int i = 0; i < nDims; i++)
    for (int j = 0; j < nDims; j++)
      R[i][j] = Rmat(i,j);

  for (int i = 0; i < nDims; i++)
    for (int j = 0; j < nDims; j++)
      for (int k = 0; k < nDims; k++)
        J[i][j] += R[j][k] * jaco_init(i,spt,k,ele);

  for (int i = 0; i < nDims; i++)
    for (int j = 0; j < nDims; j++)
      jaco(i,spt,j,ele) = J[i][j];
}

template<unsigned int nDims>
__global__
void update_inv_transform_rmat(mdvector_gpu<double> jaco_init, mdvector_gpu<double> jaco,
    mdvector_gpu<double> inv_jaco, mdvector_gpu<double> Rmat, unsigned int nEles,
    unsigned int nSpts)
{
  const int ele = (blockDim.x * blockIdx.x + threadIdx.x) % nEles;
  const int spt = (blockDim.x * blockIdx.x + threadIdx.x) / nEles;

  if (spt >= nSpts) return;

  double J[nDims][nDims] = {{0.0}};
  double R[nDims][nDims];

  for (int i = 0; i < nDims; i++)
    for (int j = 0; j < nDims; j++)
      R[i][j] = Rmat(i,j);

  for (int i = 0; i < nDims; i++)
    for (int j = 0; j < nDims; j++)
      for (int k = 0; k < nDims; k++)
        J[i][j] += R[j][k] * jaco_init(i,spt,k,ele);

  for (int i = 0; i < nDims; i++)
    for (int j = 0; j < nDims; j++)
      jaco(i,spt,j,ele) = J[i][j];

  double xr = J[0][0];  double xs = J[1][0];  double xt = J[2][0];
  double yr = J[0][1];  double ys = J[1][1];  double yt = J[2][1];
  double zr = J[0][2];  double zs = J[1][2];  double zt = J[2][2];

  // Inverse of transformation matrix (times its determinant)
  inv_jaco(0,spt,0,ele) = ys*zt - yt*zs;  inv_jaco(0,spt,1,ele) = xt*zs - xs*zt;  inv_jaco(0,spt,2,ele) = xs*yt - xt*ys;
  inv_jaco(1,spt,0,ele) = yt*zr - yr*zt;  inv_jaco(1,spt,1,ele) = xr*zt - xt*zr;  inv_jaco(1,spt,2,ele) = xt*yr - xr*yt;
  inv_jaco(2,spt,0,ele) = yr*zs - ys*zr;  inv_jaco(2,spt,1,ele) = xs*zr - xr*zs;  inv_jaco(2,spt,2,ele) = xr*ys - xs*yr;
}

void update_transforms_rigid_wrapper(mdvector_gpu<double> &jaco_spts_init,
    mdvector_gpu<double> &jaco_spts, mdvector_gpu<double> &inv_jaco_spts, mdvector_gpu<double> &norm_init,
    mdvector_gpu<double> &norm, mdvector_gpu<double> &Rmat, unsigned int nSpts,
    unsigned int nFpts, unsigned int nEles, unsigned int nDims, bool need_inv)
{
  /* WARNING: Hex elements only right now! */

  // Apply rotation matrix to body-frame jacobian
  int threads = 128;
  int blocks = (nSpts*nEles + threads - 1) / threads;
  if (need_inv)
  {
    update_inv_transform_rmat<3><<<blocks,threads>>>(jaco_spts_init,jaco_spts,inv_jaco_spts,Rmat,nEles,nSpts);
  }
  else
  {
    update_transform_rmat<3><<<blocks,threads>>>(jaco_spts_init,jaco_spts,Rmat,nEles,nSpts);
  }

  // Apply rotation matrix to body-frame normals
  double* A = norm_init.data();
  double* B = Rmat.data();
  double* C = norm.data();

  cublasDGEMM_wrapper(nFpts, nDims, nDims, 1.0, A, norm_init.ldim(),
      B, Rmat.ldim(), 0.0, C, norm.ldim());

  check_error();
}

void update_nodes_rigid_wrapper(mdvector_gpu<double> &nodes_init, mdvector_gpu<double> &nodes,
    mdvector_gpu<double> &Rmat, mdvector_gpu<double> &x_cg, unsigned int nNodes, unsigned int nDims)
{
  // Apply rotation matrix to body-frame nodes
  double *A = Rmat.data();
  double *B = nodes_init.data();
  double *C = nodes.data();

  cublasDGEMM_transA_wrapper(nDims, nNodes, nDims, 1.0, A, nDims, B, nDims, 0.0, C, nDims);

  // Add in translation of body's CG
  int threads = 192;
  int blocks = min((nNodes + threads - 1) / threads, MAX_GRID_DIM);

  if (nDims == 3)
    add_cg_offset<3><<<blocks,threads>>>(nodes, x_cg, nNodes);
  else
    add_cg_offset<2><<<blocks,threads>>>(nodes, x_cg, nNodes);

  check_error();
}

__global__
void calc_normals(mdvector_gpu<double> norm, mdvector_gpu<double> dA,
    mdvector_gpu<double> inv_jaco, mdvector_gpu<double> tnorm,
    mdvector_gpu<int> fpt2gfpt, mdvector_gpu<char> fpt2slot, int nFpts, int nEles, int nDims)
{
  int fpt = (blockDim.x * blockIdx.x + threadIdx.x) % nFpts;
  int ele = (blockDim.x * blockIdx.x + threadIdx.x) / nFpts;

  if (ele >= nEles)
    return;

  int gfpt = fpt2gfpt(fpt,ele);

  if (gfpt < 0)
    return;

  int slot = fpt2slot(fpt,ele);

  double DA = 0.0;
  if (slot == 0)
  {
    for (int dim1 = 0; dim1 < nDims; dim1++)
    {
      norm(dim1,gfpt) = 0.0;
      for (int dim2 = 0; dim2 < nDims; dim2++)
      {
        norm(dim1, gfpt) += inv_jaco(dim2, fpt, dim1, ele) * tnorm(fpt,dim2);
      }

      DA += norm(dim1, gfpt) * norm(dim1, gfpt);
    }

    DA = sqrt(DA);

    for (int dim = 0; dim < nDims; dim++)
    {
      norm(dim, gfpt) /= DA;
    }

    dA(gfpt) = DA;
  }
}

void calc_normals_wrapper(mdvector_gpu<double> &norm, mdvector_gpu<double> &dA,
    mdvector_gpu<double> &inv_jaco, mdvector_gpu<double> &tnorm,
    mdvector_gpu<int> &fpt2gfpt, mdvector_gpu<char> &fpt2slot, int nFpts,
    int nEles, int nDims)
{
  int threads = 128;
  int blocks = (nFpts * nEles + threads - 1) / threads;

  calc_normals<<<blocks,threads>>>(norm,dA,inv_jaco,tnorm,fpt2gfpt,fpt2slot,
      nFpts,nEles,nDims);

  check_error();
}

template <unsigned int nVars>
__global__
void pack_donor_u(mdvector_gpu<double> U_spts, mdvector_gpu<double> U_donors,
    int* donorIDs, int nDonors, unsigned int nSpts)
{
  const unsigned int spt   = (blockDim.x * blockIdx.x + threadIdx.x) % nSpts;
  const unsigned int donor = (blockDim.x * blockIdx.x + threadIdx.x) / nSpts;
//  const unsigned int spt  = threadIdx.x;
//  const unsigned int donor= blockIdx.x;

  if (spt >= nSpts || donor >= nDonors)
    return;

  const unsigned int ele = donorIDs[donor];
  for (unsigned int var = 0; var < nVars; var++)
  {
    U_donors(spt, donor, var) = U_spts(spt, var, ele);
  }
}

void pack_donor_u_wrapper(mdvector_gpu<double> &U_spts,
    mdvector_gpu<double> &U_donors, int* donorIDs, int nDonors,
    unsigned int nSpts, unsigned int nVars)
{
  int threads = 192;
  int blocks = (nSpts * nDonors + threads - 1) / threads;

  switch (nVars)
  {
    case 1:
      pack_donor_u<1><<<blocks, threads>>>(U_spts, U_donors, donorIDs, nDonors, nSpts);
      break;

    case 4:
      pack_donor_u<4><<<blocks, threads>>>(U_spts, U_donors, donorIDs, nDonors, nSpts);
      break;

    case 5:
      pack_donor_u<5><<<blocks, threads>>>(U_spts, U_donors, donorIDs, nDonors, nSpts);
      break;
  }
}

template <unsigned int nVars>
__global__
void pack_donor_grad(mdvector_gpu<double> dU_spts,
    mdvector_gpu<double> dU_donors, int* donorIDs, int nDonors,
    unsigned int nSpts)
{
  const unsigned int dim = blockIdx.y;
  const unsigned int spt   = (blockDim.x * blockIdx.x + threadIdx.x) % nSpts;
  const unsigned int donor = (blockDim.x * blockIdx.x + threadIdx.x) / nSpts;

  if (spt >= nSpts || donor >= nDonors || dim >= 3)
    return;

  const unsigned int ele = donorIDs[donor];

  for (unsigned int var = 0; var < nVars; var++)
  {
    dU_donors(spt, donor, var, dim) = dU_spts(dim, spt, var, ele);
  }
}

void pack_donor_grad_wrapper(mdvector_gpu<double> &dU_spts,
    mdvector_gpu<double> &dU_donors, int* donorIDs, int nDonors,
    unsigned int nSpts, unsigned int nVars, unsigned int nDims)
{
  int threads = 128;
  int nblock_x = (nDonors * nSpts + threads - 1) / threads;
  dim3 blocks(nblock_x, nDims);

  switch (nVars)
  {
    case 1:
      pack_donor_grad<1><<<blocks, threads>>>(dU_spts, dU_donors, donorIDs,
                                              nDonors, nSpts);
      break;

    case 4:
      pack_donor_grad<4><<<blocks, threads>>>(dU_spts, dU_donors, donorIDs,
                                              nDonors, nSpts);
      break;

    case 5:
      pack_donor_grad<5><<<blocks, threads>>>(dU_spts, dU_donors, donorIDs,
                                              nDonors, nSpts);
      break;
  }
}
