#include "hip/hip_runtime.h"
#include "elements_kernels.h"
#include "mdvector_gpu.h"

__global__
void compute_Fconv_spts_2D_EulerNS(mdvector_gpu<double> F, mdvector_gpu<double> U, 
    unsigned int nSpts, unsigned int nEles, double gamma)
{
  const unsigned int spt = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int ele = blockDim.y * blockIdx.y + threadIdx.y;

  if (spt >= nSpts || ele >= nEles)
    return;

  /* Compute some primitive variables */
  double momF = (U(spt, ele, 1) * U(spt,ele,1) + U(spt, ele, 2) * 
      U(spt, ele,2)) / U(spt, ele, 0);
  double P = (gamma - 1.0) * (U(spt, ele, 3) - 0.5 * momF);
  double H = (U(spt, ele, 3) + P) / U(spt, ele, 0);


  F(spt, ele, 0, 0) = U(spt, ele, 1);
  F(spt, ele, 1, 0) = U(spt, ele, 1) * U(spt, ele, 1) / U(spt, ele, 0) + P;
  F(spt, ele, 2, 0) = U(spt, ele, 1) * U(spt, ele, 2) / U(spt, ele, 0);
  F(spt, ele, 3, 0) = U(spt, ele, 1) * H;

  F(spt, ele, 0, 1) = U(spt, ele, 2);
  F(spt, ele, 1, 1) = U(spt, ele, 1) * U(spt, ele, 2) / U(spt, ele, 0);
  F(spt, ele, 2, 1) = U(spt, ele, 2) * U(spt, ele, 2) / U(spt, ele, 0) + P;
  F(spt, ele, 3, 1) = U(spt, ele, 2) * H;
 
}

void compute_Fconv_spts_2D_EulerNS_wrapper(mdvector_gpu<double> F_spts, 
    mdvector_gpu<double> U_spts, unsigned int nSpts, unsigned int nEles,
    double gamma)
{
  dim3 threads(32,32);
  dim3 blocks((nSpts + threads.x - 1)/threads.x, (nEles + threads.y - 1) / 
      threads.y);

  compute_Fconv_spts_2D_EulerNS<<<blocks, threads>>>(F_spts, U_spts, nSpts, 
      nEles, gamma);
}

__global__
void transform_flux_quad(mdvector_gpu<double> F_spts, 
    mdvector_gpu<double> jaco_spts, unsigned int nSpts, 
    unsigned int nEles, unsigned int nVars)
{
  const unsigned int spt = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int ele = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned int var = blockDim.z * blockIdx.z + threadIdx.z;

  if (spt >= nSpts || ele >= nEles || var >= nVars)
    return;

  double Ftemp = F_spts(spt, ele, var, 0);

  F_spts(spt, ele, var, 0) = F_spts(spt, ele, var, 0) * jaco_spts(1, 1, spt, ele) -
                           F_spts(spt, ele, var, 1) * jaco_spts(0, 1, spt, ele);
  F_spts(spt, ele, var, 1) = F_spts(spt, ele, var, 1) * jaco_spts(0, 0, spt, ele) -
                           Ftemp * jaco_spts(1, 0, spt, ele);

}

void transform_flux_quad_wrapper(mdvector_gpu<double> F_spts, 
    mdvector_gpu<double> jaco_spts, unsigned int nSpts, 
    unsigned int nEles, unsigned int nVars)
{
  dim3 threads(16, 16, 4);
  dim3 blocks((nSpts + threads.x - 1) / threads.x, (nEles + threads.y - 1) / 
      threads.y, (nVars + threads.z - 1) / threads.z);

  transform_flux_quad<<<threads, blocks>>>(F_spts, jaco_spts, nSpts, nEles, nVars);
}
