#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include "hipblas.h"

#include <cusp/krylov/gmres.h>
#include <cusp/krylov/cg.h>
#include <cusp/krylov/bicgstab.h>
#include <cusp/csr_matrix.h>
#include <cusp/monitor.h>
#include <cusp/print.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>

#ifdef _MPI
#include "mpi.h"
#endif

#include "input.hpp"
#include "macros.hpp"
#include "mdvector_gpu.h"
#include "spmatrix_gpu.h"
#include "solver_kernels.h"
#include "funcs_kernels.cu"

void check_error()
{
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    ThrowException(hipGetErrorString(err));
  }
}

static hipblasHandle_t cublas_handle;
void start_cublas()
{
  hipblasCreate(&cublas_handle);
}

template <typename T>
void allocate_device_data(T* &device_data, unsigned int size)
{
  hipMalloc((void**)&device_data, size*sizeof(T));
  check_error();
}

template void allocate_device_data<double>(double* &device_data, unsigned int size);
template void allocate_device_data<unsigned int>(unsigned int* &device_data, unsigned int size);
template void allocate_device_data<int>(int* &device_data, unsigned int size);


template <typename T>
void free_device_data(T* &device_data)
{
  hipFree(device_data);
  check_error();
}

template void free_device_data<double>(double* &device_data);
template void free_device_data<unsigned int>(unsigned int* &device_data);
template void free_device_data<int>(int* &device_data);

template <typename T>
void copy_to_device(T* device_data, const T* host_data, unsigned int size)
{
  hipMemcpy(device_data, host_data, size * sizeof(T), hipMemcpyHostToDevice);
  check_error();
}

template void copy_to_device<double>(double* device_data, const double* host_data, unsigned int size);
template void copy_to_device<unsigned int>(unsigned int* device_data, const unsigned int* host_data, unsigned int size);
template void copy_to_device<int>(int* device_data, const int* host_data, unsigned int size);

template <typename T>
void copy_from_device(T* host_data, const T* device_data, unsigned int size)
{
  hipMemcpy(host_data, device_data, size * sizeof(T), hipMemcpyDeviceToHost);
  check_error();
}

template void copy_from_device<double>(double* host_data, const double* device_data, unsigned int size);
template void copy_from_device<unsigned int>(unsigned int* host_data, const unsigned int* device_data, unsigned int size);
template void copy_from_device<int>(int* host_data, const int* device_data, unsigned int size);

__global__
void copy_kernel(mdvector_gpu<double> vec1, mdvector_gpu<double> vec2, unsigned int size)
{

  const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if (idx >= size)
    return;

  vec1(idx) = vec2(idx);

}

void device_copy(mdvector_gpu<double> &vec1, mdvector_gpu<double> &vec2, unsigned int size)
{
  unsigned int threads = 192;
  unsigned int blocks = (size + threads - 1) /threads;
  copy_kernel<<<blocks, threads>>>(vec1, vec2, size);
}

__global__
void add_kernel(mdvector_gpu<double> vec1, mdvector_gpu<double> vec2, unsigned int size)
{
  const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if (idx >= size)
    return;

  vec1(idx) += vec2(idx);
}

void device_add(mdvector_gpu<double> &vec1, mdvector_gpu<double> &vec2, unsigned int size)
{
  unsigned int threads = 192;
  unsigned int blocks = (size + threads - 1) /threads;
  add_kernel<<<blocks, threads>>>(vec1, vec2, size);
}

__global__
void subtract_kernel(mdvector_gpu<double> vec1, mdvector_gpu<double> vec2, unsigned int size)
{
  const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if (idx >= size)
    return;

  vec1(idx) -= vec2(idx);
}
void device_subtract(mdvector_gpu<double> &vec1, mdvector_gpu<double> &vec2, unsigned int size)
{
  unsigned int threads = 192;
  unsigned int blocks = (size + threads - 1) /threads;
  subtract_kernel<<<blocks, threads>>>(vec1, vec2, size);
}

void cublasDGEMM_wrapper(int M, int N, int K, const double alpha, const double* A, 
    int lda, const double* B, int ldb, const double beta, double *C, int ldc)
{
    hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, A, lda, B, ldb, &beta, C, ldc);
}

template <unsigned int nVars>
__global__
void U_to_faces(mdvector_gpu<double> U_fpts, mdvector_gpu<double> U_gfpts, mdvector_gpu<double> Ucomm, mdvector_gpu<int> fpt2gfpt, 
    mdvector_gpu<int> fpt2gfpt_slot, unsigned int nEles, unsigned int nFpts, bool viscous)
{  
  const unsigned int fpt = (blockDim.x * blockIdx.x + threadIdx.x) % nFpts;
  const unsigned int ele = (blockDim.x * blockIdx.x + threadIdx.x) / nFpts;

  if (fpt >= nFpts || ele >= nEles)
    return;

  int gfpt = fpt2gfpt(fpt,ele);
  /* Check if flux point is on ghost edge */
  if (gfpt == -1)
  {
    if (viscous) // if viscous, put extrapolated solution into Ucomm
    {
      for (unsigned int var = 0; var < nVars; var++)
        Ucomm(fpt, ele, var) = U_fpts(fpt, ele, var);
    }
    return;
  }

  int slot = fpt2gfpt_slot(fpt,ele);

  for (unsigned int var = 0; var < nVars; var++)
    U_gfpts(gfpt, var, slot) = U_fpts(fpt, ele, var);

}

void U_to_faces_wrapper(mdvector_gpu<double> &U_fpts, mdvector_gpu<double> &U_gfpts, 
    mdvector_gpu<double> &Ucomm, mdvector_gpu<int> &fpt2gfpt, mdvector_gpu<int> &fpt2gfpt_slot, 
    unsigned int nVars, unsigned int nEles, unsigned int nFpts, unsigned int nDims, unsigned int equation, 
    bool viscous)
{
  unsigned int threads= 192;
  unsigned int blocks = ((nFpts * nEles) + threads - 1)/ threads;

  if (equation == AdvDiff)
  {
    U_to_faces<1><<<blocks, threads>>>(U_fpts, U_gfpts, Ucomm, fpt2gfpt, fpt2gfpt_slot, nEles, nFpts, viscous);
  }
  else if (equation == EulerNS)
  {
    if (nDims == 2)
      U_to_faces<4><<<blocks, threads>>>(U_fpts, U_gfpts, Ucomm, fpt2gfpt, fpt2gfpt_slot, nEles, nFpts, viscous);
    else if (nDims == 3)
      U_to_faces<5><<<blocks, threads>>>(U_fpts, U_gfpts, Ucomm, fpt2gfpt, fpt2gfpt_slot, nEles, nFpts, viscous);
  }
}

template <unsigned int nVars>
__global__
void U_from_faces(mdvector_gpu<double> Ucomm_gfpts, mdvector_gpu<double> Ucomm_fpts, mdvector_gpu<int> fpt2gfpt, 
    mdvector_gpu<int> fpt2gfpt_slot, unsigned int nEles, unsigned int nFpts)
{
  const unsigned int fpt = (blockDim.x * blockIdx.x + threadIdx.x) % nFpts;
  const unsigned int ele = (blockDim.x * blockIdx.x + threadIdx.x) / nFpts;

  if (fpt >= nFpts || ele >= nEles)
    return;

  int gfpt = fpt2gfpt(fpt,ele);

  /* Check if flux point is on ghost edge */
  if (gfpt == -1)
    return;

  int slot = fpt2gfpt_slot(fpt,ele);

  for (unsigned int var = 0; var < nVars; var++)
    Ucomm_fpts(fpt, ele, var) = Ucomm_gfpts(gfpt, var, slot);

}

void U_from_faces_wrapper(mdvector_gpu<double> &Ucomm_gfpts, mdvector_gpu<double> &Ucomm_fpts, 
    mdvector_gpu<int> &fpt2gfpt, mdvector_gpu<int> &fpt2gfpt_slot, unsigned int nVars, 
    unsigned int nEles, unsigned int nFpts, unsigned int nDims, unsigned int equation)
{
  unsigned int threads= 192;
  unsigned int blocks = ((nFpts * nEles) + threads - 1)/ threads;

  if (equation == AdvDiff)
  {
    U_from_faces<1><<<blocks, threads>>>(Ucomm_gfpts, Ucomm_fpts, fpt2gfpt, fpt2gfpt_slot, nEles, nFpts);
  }
  else if (equation == EulerNS)
  {
    if (nDims == 2)
      U_from_faces<4><<<blocks, threads>>>(Ucomm_gfpts, Ucomm_fpts, fpt2gfpt, fpt2gfpt_slot, nEles, nFpts);
    else
      U_from_faces<5><<<blocks, threads>>>(Ucomm_gfpts, Ucomm_fpts, fpt2gfpt, fpt2gfpt_slot, nEles, nFpts);
  }

}

template <unsigned int nVars, unsigned int nDims>
__global__
void dU_to_faces(mdvector_gpu<double> dU_fpts, mdvector_gpu<double> dU_gfpts, mdvector_gpu<int> fpt2gfpt, 
    mdvector_gpu<int> fpt2gfpt_slot, unsigned int nEles, unsigned int nFpts)
{
  const unsigned int fpt = (blockDim.x * blockIdx.x + threadIdx.x) % nFpts;
  const unsigned int ele = (blockDim.x * blockIdx.x + threadIdx.x) / nFpts;

  if (fpt >= nFpts || ele >= nEles)
    return;

  int gfpt = fpt2gfpt(fpt,ele);

  /* Check if flux point is on ghost edge */
  if (gfpt == -1)
    return;

  int slot = fpt2gfpt_slot(fpt,ele);

  for (unsigned int dim = 0; dim < nDims; dim++)
  {
    for (unsigned int var = 0; var < nVars; var++)
    {
      dU_gfpts(gfpt, var, dim, slot) = dU_fpts(fpt, ele, var, dim);
    }
  }

}

void dU_to_faces_wrapper(mdvector_gpu<double> &dU_fpts, mdvector_gpu<double> &dU_gfpts, 
    mdvector_gpu<int> &fpt2gfpt, mdvector_gpu<int> &fpt2gfpt_slot, unsigned int nVars, 
    unsigned int nEles, unsigned int nFpts, unsigned int nDims, unsigned int equation)
{
  unsigned int threads= 192;
  unsigned int blocks = ((nFpts * nEles) + threads - 1)/ threads;
  

  if (equation == AdvDiff)
  {
    if (nDims == 2)
      dU_to_faces<1, 2><<<blocks, threads>>>(dU_fpts, dU_gfpts, fpt2gfpt, fpt2gfpt_slot, nEles, nFpts);
    else
      dU_to_faces<1, 3><<<blocks, threads>>>(dU_fpts, dU_gfpts, fpt2gfpt, fpt2gfpt_slot, nEles, nFpts);
  }
  else if (equation == EulerNS)
  {
    if (nDims == 2)
      dU_to_faces<4, 2><<<blocks, threads>>>(dU_fpts, dU_gfpts, fpt2gfpt, fpt2gfpt_slot, nEles, nFpts);
    else
      dU_to_faces<5, 3><<<blocks, threads>>>(dU_fpts, dU_gfpts, fpt2gfpt, fpt2gfpt_slot, nEles, nFpts);
  }
}

template <unsigned int nVars, unsigned int nDims>
__global__
void compute_divF(mdvector_gpu<double> divF, mdvector_gpu<double> dF_spts, 
    unsigned int nSpts, unsigned int nEles, unsigned int stage)
{
  const unsigned int spt = (blockDim.x * blockIdx.x + threadIdx.x) % nSpts;
  const unsigned int ele = (blockDim.x * blockIdx.x + threadIdx.x) / nSpts;

  if (spt >= nSpts || ele >= nEles)
    return;

  double sum[nVars];

  for (unsigned int var = 0; var < nVars; var++)
    sum[var] = 0.0;

  for (unsigned int dim = 0; dim < nDims; dim++)
    for (unsigned int var = 0; var < nVars; var++)
      sum[var] += dF_spts(spt, ele, var, dim);

  for (unsigned int var = 0; var < nVars; var++)
    divF(spt, ele, var, stage) = sum[var];


}

void compute_divF_wrapper(mdvector_gpu<double> &divF, mdvector_gpu<double> &dF_spts, 
    unsigned int nSpts, unsigned int nVars, unsigned int nEles, unsigned int nDims,
    unsigned int equation, unsigned int stage)
{
  unsigned int threads= 192;
  unsigned int blocks = ((nSpts * nEles) + threads - 1)/ threads;

  if (equation == AdvDiff)
  {
    if (nDims == 2)
      compute_divF<1,2><<<blocks, threads>>>(divF, dF_spts, nSpts, nEles, stage);
    else
      compute_divF<1,3><<<blocks, threads>>>(divF, dF_spts, nSpts, nEles, stage);
  }
  else if (equation == EulerNS)
  {
    if (nDims == 2)
      compute_divF<4,2><<<blocks, threads>>>(divF, dF_spts, nSpts, nEles, stage);
    else
      compute_divF<5,3><<<blocks, threads>>>(divF, dF_spts, nSpts, nEles, stage);
  }
}

template <unsigned int nVars>
__global__
void RK_update(mdvector_gpu<double> U_spts, mdvector_gpu<double> U_ini, 
    mdvector_gpu<double> divF, mdvector_gpu<double> jaco_det_spts, mdvector_gpu<double> dt_in, 
    mdvector_gpu<double> rk_coeff, unsigned int dt_type, unsigned int nSpts, unsigned int nEles, 
    unsigned int stage, unsigned int nStages, bool last_stage)
{
  const unsigned int spt = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int ele = blockDim.y * blockIdx.y + threadIdx.y;

  if (spt >= nSpts || ele >= nEles)
    return;

  double dt;
  if (dt_type != 2)
    dt = dt_in(0);
  else
    dt = dt_in(ele);

  double jaco_det = jaco_det_spts(spt,ele);

  if (!last_stage)
  {
    double coeff = rk_coeff(stage);
    for (unsigned int var = 0; var < nVars; var ++)
      U_spts(spt, ele, var) = U_ini(spt, ele, var) - coeff * dt / 
          jaco_det * divF(spt, ele, var, stage);
  }
  else
  {
    double sum[nVars];
    for (unsigned int var = 0; var < nVars; var++)
      sum[var] = 0.;

    for (unsigned int n = 0; n < nStages; n++)
    {
      double coeff = rk_coeff(n);
      for (unsigned int var = 0; var < nVars; var++)
      {
        sum[var] -= coeff * dt / jaco_det * divF(spt, ele, var, n);
      }
    }

    for (unsigned int var = 0; var < nVars; var++)
      U_spts(spt,ele,var) += sum[var];

  }
}

void RK_update_wrapper(mdvector_gpu<double> &U_spts, mdvector_gpu<double> &U_ini, 
    mdvector_gpu<double> &divF, mdvector_gpu<double> &jaco_det_spts, mdvector_gpu<double> &dt, 
    mdvector_gpu<double> &rk_coeff, unsigned int dt_type, unsigned int nSpts, unsigned int nEles, 
    unsigned int nVars, unsigned int nDims, unsigned int equation, unsigned int stage, 
    unsigned int nStages, bool last_stage)
{
  dim3 threads(16,12);
  dim3 blocks((nSpts + threads.x - 1)/threads.x, (nEles + threads.y - 1)/
      threads.y);

  if (equation == AdvDiff)
  {
      RK_update<1><<<blocks, threads>>>(U_spts, U_ini, divF, jaco_det_spts, dt, 
          rk_coeff, dt_type, nSpts, nEles, stage, nStages, last_stage);
  }
  else if (equation == EulerNS)
  {
    if (nDims == 2)
      RK_update<4><<<blocks, threads>>>(U_spts, U_ini, divF, jaco_det_spts, dt, 
          rk_coeff, dt_type, nSpts, nEles, stage, nStages, last_stage);
    else
      RK_update<5><<<blocks, threads>>>(U_spts, U_ini, divF, jaco_det_spts, dt, 
          rk_coeff, dt_type, nSpts, nEles, stage, nStages, last_stage);
  }
}

template <unsigned int nVars>
__global__
void RK_update_source(mdvector_gpu<double> U_spts, mdvector_gpu<double> U_ini, 
    mdvector_gpu<double> divF, mdvector_gpu<double> source, mdvector_gpu<double> jaco_det_spts, 
    mdvector_gpu<double> dt_in, mdvector_gpu<double> rk_coeff, unsigned int dt_type, 
    unsigned int nSpts, unsigned int nEles, unsigned int stage, unsigned int nStages, 
    bool last_stage)
{
  const unsigned int spt = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int ele = blockDim.y * blockIdx.y + threadIdx.y;

  if (spt >= nSpts || ele >= nEles)
    return;

  double dt;
  if (dt_type != 2)
    dt = dt_in(0);
  else
    dt = dt_in(ele);

  double jaco_det = jaco_det_spts(spt,ele);

  if (!last_stage)
  {
    double coeff = rk_coeff(stage);
    for (unsigned int var = 0; var < nVars; var ++)
      U_spts(spt, ele, var) = U_ini(spt, ele, var) - coeff * dt / 
          jaco_det * (divF(spt, ele, var, stage) + source(spt, ele, var));
  }
  else
  {
    double sum[nVars];
    for (unsigned int var = 0; var < nVars; var++)
      sum[var] = 0.;

    for (unsigned int n = 0; n < nStages; n++)
    {
      double coeff = rk_coeff(n);
      for (unsigned int var = 0; var < nVars; var++)
      {
        sum[var] -= coeff * dt / jaco_det * (divF(spt, ele, var, n) + source(spt, ele, var));
      }
    }

    for (unsigned int var = 0; var < nVars; var++)
      U_spts(spt,ele,var) += sum[var];

  }
}

void RK_update_source_wrapper(mdvector_gpu<double> &U_spts, mdvector_gpu<double> &U_ini, 
    mdvector_gpu<double> &divF, mdvector_gpu<double> &source, mdvector_gpu<double> &jaco_det_spts, 
    mdvector_gpu<double> &dt, mdvector_gpu<double> &rk_coeff, unsigned int dt_type, 
    unsigned int nSpts, unsigned int nEles, unsigned int nVars, unsigned int nDims, 
    unsigned int equation, unsigned int stage, unsigned int nStages, bool last_stage)
{
  dim3 threads(16,12);
  dim3 blocks((nSpts + threads.x - 1)/threads.x, (nEles + threads.y - 1)/
      threads.y);

  if (equation == AdvDiff)
  {
      RK_update_source<1><<<blocks, threads>>>(U_spts, U_ini, divF, source, jaco_det_spts, dt, 
          rk_coeff, dt_type, nSpts, nEles, stage, nStages, last_stage);
  }
  else if (equation == EulerNS)
  {
    if (nDims == 2)
      RK_update_source<4><<<blocks, threads>>>(U_spts, U_ini, divF, source, jaco_det_spts, dt, 
          rk_coeff, dt_type, nSpts, nEles, stage, nStages, last_stage);
    else
      RK_update_source<5><<<blocks, threads>>>(U_spts, U_ini, divF, source, jaco_det_spts, dt, 
          rk_coeff, dt_type, nSpts, nEles, stage, nStages, last_stage);
  }
}

__global__
void compute_element_dt(mdvector_gpu<double> dt, mdvector_gpu<double> waveSp_gfpts, 
    mdvector_gpu<double> dA, mdvector_gpu<int> fpt2gfpt, double CFL, int order, 
    unsigned int nFpts, unsigned int nEles)
{
  const unsigned int ele = blockDim.x * blockIdx.x + threadIdx.x;

  if (ele >= nEles)
    return;

  double waveSp_max = 0.0;

  /* Compute maximum wavespeed */
  for (unsigned int fpt = 0; fpt <nFpts; fpt++)
  {
    /* Skip if on ghost edge. */
    int gfpt = fpt2gfpt(fpt,ele);
    if (gfpt == -1)
      continue;

    double waveSp = waveSp_gfpts(gfpt) / dA(gfpt);

    waveSp_max = max(waveSp, waveSp_max);
  }

  /* Note: CFL is applied to parent space element with width 2 */
  dt(ele) = (CFL) * get_cfl_limit_dev(order) * (2.0 / (waveSp_max+1.e-10));
}

void compute_element_dt_wrapper(mdvector_gpu<double> &dt, mdvector_gpu<double> &waveSp, 
    mdvector_gpu<double> &dA, mdvector_gpu<int> &fpt2gfpt, double CFL, int order, 
    unsigned int dt_type, unsigned int nFpts, unsigned int nEles)
{
  unsigned int threads = 192;
  unsigned int blocks = (nEles + threads - 1) / threads;

  compute_element_dt<<<blocks, threads>>>(dt, waveSp, dA, fpt2gfpt, CFL, order, 
      nFpts, nEles);

  if (dt_type == 1)
  {
    /* Get min dt using thrust (pretty slow) */
    thrust::device_ptr<double> dt_ptr = thrust::device_pointer_cast(dt.data());
    thrust::device_ptr<double> min_ptr = thrust::min_element(dt_ptr, dt_ptr + nEles);

#ifdef _MPI
    double min_dt = min_ptr[0];
    MPI_Allreduce(MPI_IN_PLACE, &min_dt, 1, MPI_DOUBLE, MPI_MIN, MPI_COMM_WORLD);
    dt_ptr[0] = min_dt;
#else
    dt_ptr[0] = min_ptr[0];
    //thrust::copy(min_ptr, min_ptr+1, dt_ptr);
#endif

  }

}

template<unsigned int nVars, unsigned int nDims>
__global__
void add_source(mdvector_gpu<double> divF_spts, mdvector_gpu<double> jaco_det_spts, mdvector_gpu<double> coord_spts, 
    unsigned int nSpts, unsigned int nEles, unsigned int equation, 
    double flow_time, unsigned int stage)
{
  const unsigned int spt = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int ele = blockDim.y * blockIdx.y + threadIdx.y;

  if (spt >= nSpts || ele >= nEles)
    return;

  double x = coord_spts(spt, ele, 0);
  double y = coord_spts(spt, ele, 1);
  double z = 0;
  if (nDims == 3)
    z = coord_spts(spt, ele, 2);

  double jaco_det = jaco_det_spts(spt, ele);

  for (unsigned int n = 0; n < nVars; n++)
  {
    divF_spts(spt, ele, n, stage) += compute_source_term_dev(x, y, z, flow_time, n, nDims, equation) * jaco_det;
  }
}

void add_source_wrapper(mdvector_gpu<double> &divF_spts, mdvector_gpu<double> &jaco_det_spts, mdvector_gpu<double> &coord_spts, 
    unsigned int nSpts, unsigned int nEles, unsigned int nVars, unsigned int nDims, unsigned int equation, 
    double flow_time, unsigned int stage)
{
  dim3 threads(16,12);
  dim3 blocks((nSpts + threads.x - 1)/threads.x, (nEles + threads.y - 1)/
      threads.y);

  if (nDims == 2)
  {
    if (equation == AdvDiff)
      add_source<1, 2><<<blocks, threads>>>(divF_spts, jaco_det_spts, coord_spts, nSpts, nEles, equation,
          flow_time, stage);
    else
      add_source<4, 2><<<blocks, threads>>>(divF_spts, jaco_det_spts, coord_spts, nSpts, nEles, equation,
          flow_time, stage);
  }
  else
  {
    if (equation == AdvDiff)
      add_source<1, 3><<<blocks, threads>>>(divF_spts, jaco_det_spts, coord_spts, nSpts, nEles, equation,
          flow_time, stage);
    else
      add_source<5, 3><<<blocks, threads>>>(divF_spts, jaco_det_spts, coord_spts, nSpts, nEles, equation,
          flow_time, stage);
  }
}

__global__
void compute_RHS(mdvector_gpu<double> divF_spts, mdvector_gpu<double> jaco_det_spts, mdvector_gpu<double> dt, 
    mdvector_gpu<double> b, unsigned int nSpts, unsigned int nEles, unsigned int nVars)
{
  const unsigned int spt = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int ele = blockDim.y * blockIdx.y + threadIdx.y;

  if (spt >= nSpts || ele >= nEles)
    return;

  for (unsigned int n = 0; n < nVars; n++)
  {
    b(spt, ele, n) = -(dt(0) * divF_spts(spt, ele, n, 0))/jaco_det_spts(spt, ele);
    //b(spt, ele, n) = 0;
  }
}

void compute_RHS_wrapper(mdvector_gpu<double> &divF_spts, mdvector_gpu<double> &jaco_det_spts, mdvector_gpu<double> &dt, 
    mdvector_gpu<double> &b, unsigned int nSpts, unsigned int nEles, unsigned int nVars)
{
  dim3 threads(16,12);
  dim3 blocks((nSpts + threads.x - 1)/threads.x, (nEles + threads.y - 1)/threads.y);

  compute_RHS<<<blocks, threads>>>(divF_spts, jaco_det_spts, dt, b, nSpts, nEles, nVars);
}

void compute_deltaU_wrapper(spmatrix_gpu<double> &A, mdvector_gpu<double> &deltaU, mdvector_gpu<double> &b) 
{
  /* Experiment: Use CUSP library to solve system */
  /* First, wrap device arrays using Thrust */
  thrust::device_ptr<double> vals_w = thrust::device_pointer_cast(A.getVals());
  thrust::device_ptr<int> row_ptr_w = thrust::device_pointer_cast(A.getRowPtr());
  thrust::device_ptr<int> col_idx_w = thrust::device_pointer_cast(A.getColIdx());
  thrust::device_ptr<double> deltaU_w = thrust::device_pointer_cast(deltaU.data());
  thrust::device_ptr<double> b_w = thrust::device_pointer_cast(b.data());

  /* Next, create CSR array view using CUSP */
  typedef typename cusp::array1d_view<thrust::device_ptr<int>> IntArray;
  typedef typename cusp::array1d_view<thrust::device_ptr<double>> DoubleArray;
  IntArray row_ptr(row_ptr_w, row_ptr_w + A.getNrows() + 1);
  IntArray col_idx(col_idx_w, col_idx_w + A.getNnonzeros());
  DoubleArray vals(vals_w, vals_w + A.getNnonzeros());
  DoubleArray delU(deltaU_w, deltaU_w + A.getNrows());
  DoubleArray RHS(b_w, b_w + A.getNrows());

  /* Create cusp CSR view */
  typedef cusp::csr_matrix_view<IntArray, IntArray, DoubleArray> CSRMat;
  CSRMat Acsr(A.getNrows(), A.getNrows(), A.getNnonzeros(), row_ptr, col_idx, vals);

  //std::cout << "\nA" << std::endl;
  //cusp::print(Acsr);
  //std::cout << "\nU"<< std::endl;
  //cusp::print(U);
  //std::cout << "\nRHS" << std::endl;
  //cusp::print(RHS);


  /* Setup linear solver */
  // set stopping criteria:
  //  iteration_limit    = 100
  //  relative_tolerance = 1e-6
  //  absolute_tolerance = 0
  //  print = false
  cusp::monitor<double> monitor(RHS, 1000, 1e-6, 0, false);
  int restart = 50;
  // set preconditioner (identity)
  cusp::identity_operator<double, cusp::device_memory> M(Acsr.num_rows, Acsr.num_rows);

  // solve the linear system A U = RHS
  cusp::krylov::gmres(Acsr, delU, RHS, restart, monitor, M);
  //cusp::krylov::bicgstab(Acsr, delU, RHS, monitor, M);

  /*
  if (monitor.converged())
  {
    std::cout << "Solver converged to " << monitor.relative_tolerance() << " relative tolerance";
    std::cout << " after " << monitor.iteration_count() << " iterations" << std::endl;
  }
  else
  {
    std::cout << "Solver reached iteration limit " << monitor.iteration_limit() << " before converging";
    std::cout << " to " << monitor.relative_tolerance() << " relative tolerance " << std::endl;
  }
  */
}
