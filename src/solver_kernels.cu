#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include "hipblas.h"

#include <thrust/device_vector.h>
#include <thrust/extrema.h>

#include "input.hpp"
#include "macros.hpp"
#include "mdvector_gpu.h"
#include "solver_kernels.h"

void check_error()
{
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    ThrowException(hipGetErrorString(err));
  }
}

static hipblasHandle_t cublas_handle;
void start_cublas()
{
  hipblasCreate(&cublas_handle);
}

template <typename T>
void allocate_device_data(T* &device_data, unsigned int size)
{
  hipMalloc((void**)&device_data, size*sizeof(T));
  check_error();
}

template void allocate_device_data<double>(double* &device_data, unsigned int size);
template void allocate_device_data<unsigned int>(unsigned int* &device_data, unsigned int size);
template void allocate_device_data<int>(int* &device_data, unsigned int size);


template <typename T>
void free_device_data(T* &device_data)
{
  hipFree(device_data);
  check_error();
}

template void free_device_data<double>(double* &device_data);
template void free_device_data<unsigned int>(unsigned int* &device_data);
template void free_device_data<int>(int* &device_data);

template <typename T>
void copy_to_device(T* device_data, const T* host_data, unsigned int size)
{
  hipMemcpy(device_data, host_data, size * sizeof(T), hipMemcpyHostToDevice);
  check_error();
}

template void copy_to_device<double>(double* device_data, const double* host_data, unsigned int size);
template void copy_to_device<unsigned int>(unsigned int* device_data, const unsigned int* host_data, unsigned int size);
template void copy_to_device<int>(int* device_data, const int* host_data, unsigned int size);

template <typename T>
void copy_from_device(T* host_data, const T* device_data, unsigned int size)
{
  hipMemcpy(host_data, device_data, size * sizeof(T), hipMemcpyDeviceToHost);
  check_error();
}

template void copy_from_device<double>(double* host_data, const double* device_data, unsigned int size);
template void copy_from_device<unsigned int>(unsigned int* host_data, const unsigned int* device_data, unsigned int size);
template void copy_from_device<int>(int* host_data, const int* device_data, unsigned int size);

__global__
void copy_kernel(mdvector_gpu<double> vec1, mdvector_gpu<double> vec2, unsigned int size)
{

  const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if (idx >= size)
    return;

  vec1(idx) = vec2(idx);

}

void device_copy(mdvector_gpu<double> vec1, mdvector_gpu<double> vec2, unsigned int size)
{
  unsigned int threads = 192;
  unsigned int blocks = (size + threads - 1) /threads;
  copy_kernel<<<blocks, threads>>>(vec1, vec2, size);
}

__global__
void add_kernel(mdvector_gpu<double> vec1, mdvector_gpu<double> vec2, unsigned int size)
{
  const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if (idx >= size)
    return;

  vec1(idx) += vec2(idx);
}

void device_add(mdvector_gpu<double> vec1, mdvector_gpu<double> vec2, unsigned int size)
{
  unsigned int threads = 192;
  unsigned int blocks = (size + threads - 1) /threads;
  add_kernel<<<blocks, threads>>>(vec1, vec2, size);
}

__global__
void subtract_kernel(mdvector_gpu<double> vec1, mdvector_gpu<double> vec2, unsigned int size)
{
  const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if (idx >= size)
    return;

  vec1(idx) -= vec2(idx);
}
void device_subtract(mdvector_gpu<double> vec1, mdvector_gpu<double> vec2, unsigned int size)
{
  unsigned int threads = 192;
  unsigned int blocks = (size + threads - 1) /threads;
  subtract_kernel<<<blocks, threads>>>(vec1, vec2, size);
}

void cublasDGEMM_wrapper(int M, int N, int K, const double alpha, const double* A, 
    int lda, const double* B, int ldb, const double beta, double *C, int ldc)
{
    hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, A, lda, B, ldb, &beta, C, ldc);
}

template <unsigned int nVars>
__global__
void U_to_faces(mdvector_gpu<double> U_fpts, mdvector_gpu<double> U_gfpts, mdvector_gpu<double> Ucomm, mdvector_gpu<int> fpt2gfpt, 
    mdvector_gpu<int> fpt2gfpt_slot, unsigned int nEles, unsigned int nFpts, bool viscous)
{  
  const unsigned int fpt = (blockDim.x * blockIdx.x + threadIdx.x) % nFpts;
  const unsigned int ele = (blockDim.x * blockIdx.x + threadIdx.x) / nFpts;

  if (fpt >= nFpts || ele >= nEles)
    return;

  int gfpt = fpt2gfpt(fpt,ele);
  /* Check if flux point is on ghost edge */
  if (gfpt == -1)
  {
    if (viscous) // if viscous, put extrapolated solution into Ucomm
    {
      for (unsigned int var = 0; var < nVars; var++)
        Ucomm(fpt, ele, var) = U_fpts(fpt, ele, var);
    }
    return;
  }

  int slot = fpt2gfpt_slot(fpt,ele);

  for (unsigned int var = 0; var < nVars; var++)
    U_gfpts(gfpt, var, slot) = U_fpts(fpt, ele, var);

}

void U_to_faces_wrapper(mdvector_gpu<double> &U_fpts, mdvector_gpu<double> &U_gfpts, 
    mdvector_gpu<double> &Ucomm, mdvector_gpu<int> &fpt2gfpt, mdvector_gpu<int> &fpt2gfpt_slot, 
    unsigned int nVars, unsigned int nEles, unsigned int nFpts, unsigned int nDims, unsigned int equation, 
    bool viscous)
{
  unsigned int threads= 192;
  unsigned int blocks = ((nFpts * nEles) + threads - 1)/ threads;

  if (equation == AdvDiff)
  {
    U_to_faces<1><<<blocks, threads>>>(U_fpts, U_gfpts, Ucomm, fpt2gfpt, fpt2gfpt_slot, nEles, nFpts, viscous);
  }
  else if (equation == EulerNS)
  {
    if (nDims == 2)
      U_to_faces<4><<<blocks, threads>>>(U_fpts, U_gfpts, Ucomm, fpt2gfpt, fpt2gfpt_slot, nEles, nFpts, viscous);
    else if (nDims == 3)
      ThrowException("Under Construction!");
  }
}

template <unsigned int nVars>
__global__
void U_from_faces(mdvector_gpu<double> Ucomm_gfpts, mdvector_gpu<double> Ucomm_fpts, mdvector_gpu<int> fpt2gfpt, 
    mdvector_gpu<int> fpt2gfpt_slot, unsigned int nEles, unsigned int nFpts)
{
  const unsigned int fpt = (blockDim.x * blockIdx.x + threadIdx.x) % nFpts;
  const unsigned int ele = (blockDim.x * blockIdx.x + threadIdx.x) / nFpts;

  if (fpt >= nFpts || ele >= nEles)
    return;

  int gfpt = fpt2gfpt(fpt,ele);

  /* Check if flux point is on ghost edge */
  if (gfpt == -1)
    return;

  int slot = fpt2gfpt_slot(fpt,ele);

  for (unsigned int var = 0; var < nVars; var++)
    Ucomm_fpts(fpt, ele, var) = Ucomm_gfpts(gfpt, var, slot);

}

void U_from_faces_wrapper(mdvector_gpu<double> &Ucomm_gfpts, mdvector_gpu<double> &Ucomm_fpts, 
    mdvector_gpu<int> &fpt2gfpt, mdvector_gpu<int> &fpt2gfpt_slot, unsigned int nVars, 
    unsigned int nEles, unsigned int nFpts, unsigned int nDims, unsigned int equation)
{
  unsigned int threads= 192;
  unsigned int blocks = ((nFpts * nEles) + threads - 1)/ threads;

  if (equation == AdvDiff)
  {
    U_from_faces<1><<<blocks, threads>>>(Ucomm_gfpts, Ucomm_fpts, fpt2gfpt, fpt2gfpt_slot, nEles, nFpts);
  }
  else if (equation == EulerNS)
  {
    if (nDims == 2)
      U_from_faces<4><<<blocks, threads>>>(Ucomm_gfpts, Ucomm_fpts, fpt2gfpt, fpt2gfpt_slot, nEles, nFpts);
    else
      ThrowException("Under Construction");
  }

}

template <unsigned int nVars, unsigned int nDims>
__global__
void dU_to_faces(mdvector_gpu<double> dU_fpts, mdvector_gpu<double> dU_gfpts, mdvector_gpu<int> fpt2gfpt, 
    mdvector_gpu<int> fpt2gfpt_slot, unsigned int nEles, unsigned int nFpts)
{
  const unsigned int fpt = (blockDim.x * blockIdx.x + threadIdx.x) % nFpts;
  const unsigned int ele = (blockDim.x * blockIdx.x + threadIdx.x) / nFpts;

  if (fpt >= nFpts || ele >= nEles)
    return;

  int gfpt = fpt2gfpt(fpt,ele);

  /* Check if flux point is on ghost edge */
  if (gfpt == -1)
    return;

  int slot = fpt2gfpt_slot(fpt,ele);

  for (unsigned int dim = 0; dim < nDims; dim++)
  {
    for (unsigned int var = 0; var < nVars; var++)
    {
      dU_gfpts(gfpt, var, dim, slot) = dU_fpts(fpt, ele, var, dim);
    }
  }

}

void dU_to_faces_wrapper(mdvector_gpu<double> &dU_fpts, mdvector_gpu<double> &dU_gfpts, 
    mdvector_gpu<int> &fpt2gfpt, mdvector_gpu<int> &fpt2gfpt_slot, unsigned int nVars, 
    unsigned int nEles, unsigned int nFpts, unsigned int nDims, unsigned int equation)
{
  unsigned int threads= 192;
  unsigned int blocks = ((nFpts * nEles) + threads - 1)/ threads;
  

  if (equation == AdvDiff)
  {
    if (nDims == 2)
      dU_to_faces<1, 2><<<blocks, threads>>>(dU_fpts, dU_gfpts, fpt2gfpt, fpt2gfpt_slot, nEles, nFpts);
    else
      ThrowException("Under Construction");
  }
  else if (equation == EulerNS)
  {
    if (nDims == 2)
      dU_to_faces<4, 2><<<blocks, threads>>>(dU_fpts, dU_gfpts, fpt2gfpt, fpt2gfpt_slot, nEles, nFpts);
    else
      ThrowException("Under Construction");
  }
}

template <unsigned int nVars, unsigned int nDims>
__global__
void compute_divF(mdvector_gpu<double> divF, mdvector_gpu<double> dF_spts, 
    unsigned int nSpts, unsigned int nEles, unsigned int stage)
{
  const unsigned int spt = (blockDim.x * blockIdx.x + threadIdx.x) % nSpts;
  const unsigned int ele = (blockDim.x * blockIdx.x + threadIdx.x) / nSpts;

  if (spt >= nSpts || ele >= nEles)
    return;

  double sum[nVars];

  for (unsigned int var = 0; var < nVars; var++)
    sum[var] = 0.0;

  for (unsigned int dim = 0; dim < nDims; dim++)
    for (unsigned int var = 0; var < nVars; var++)
      sum[var] += dF_spts(spt, ele, var, dim);

  for (unsigned int var = 0; var < nVars; var++)
    divF(spt, ele, var, stage) = sum[var];


}

void compute_divF_wrapper(mdvector_gpu<double> &divF, mdvector_gpu<double> &dF_spts, 
    unsigned int nSpts, unsigned int nVars, unsigned int nEles, unsigned int nDims,
    unsigned int equation, unsigned int stage)
{
  unsigned int threads= 192;
  unsigned int blocks = ((nSpts * nEles) + threads - 1)/ threads;

  if (equation == AdvDiff)
  {
    if (nDims == 2)
      compute_divF<1,2><<<blocks, threads>>>(divF, dF_spts, nSpts, nEles, stage);
    else
      ThrowException("Under construction!");
  }
  else if (equation == EulerNS)
  {
    if (nDims == 2)
      compute_divF<4,2><<<blocks, threads>>>(divF, dF_spts, nSpts, nEles, stage);
    else
      ThrowException("Under construction!");
  }
}

template <unsigned int nVars>
__global__
void RK_update(mdvector_gpu<double> U_spts, mdvector_gpu<double> U_ini, 
    mdvector_gpu<double> divF, mdvector_gpu<double> jaco_det_spts, mdvector_gpu<double> dt_in, 
    mdvector_gpu<double> rk_coeff, unsigned int dt_type, unsigned int nSpts, unsigned int nEles, 
    unsigned int stage, unsigned int nStages, bool last_stage)
{
  const unsigned int spt = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int ele = blockDim.y * blockIdx.y + threadIdx.y;

  if (spt >= nSpts || ele >= nEles)
    return;

  double dt;
  if (dt_type != 2)
    dt = dt_in(0);
  else
    dt = dt_in(ele);

  double jaco_det = jaco_det_spts(spt,ele);

  if (!last_stage)
  {
    double coeff = rk_coeff(stage);
    for (unsigned int var = 0; var < nVars; var ++)
      U_spts(spt, ele, var) = U_ini(spt, ele, var) - coeff * dt / 
          jaco_det * divF(spt, ele, var, stage);
  }
  else
  {
    double sum[nVars];
    for (unsigned int var = 0; var < nVars; var++)
      sum[var] = 0.;

    for (unsigned int n = 0; n < nStages; n++)
    {
      double coeff = rk_coeff(n);
      for (unsigned int var = 0; var < nVars; var++)
      {
        sum[var] -= coeff * dt / jaco_det * divF(spt, ele, var, n);
      }
    }

    for (unsigned int var = 0; var < nVars; var++)
      U_spts(spt,ele,var) += sum[var];

  }
}

void RK_update_wrapper(mdvector_gpu<double> &U_spts, mdvector_gpu<double> &U_ini, 
    mdvector_gpu<double> &divF, mdvector_gpu<double> &jaco_det_spts, mdvector_gpu<double> &dt, 
    mdvector_gpu<double> &rk_coeff, unsigned int dt_type, unsigned int nSpts, unsigned int nEles, 
    unsigned int nVars, unsigned int nDims, unsigned int equation, unsigned int stage, 
    unsigned int nStages, bool last_stage)
{
  dim3 threads(16,12);
  dim3 blocks((nSpts + threads.x - 1)/threads.x, (nEles + threads.y - 1)/
      threads.y);

  if (equation == AdvDiff)
  {
      RK_update<1><<<blocks, threads>>>(U_spts, U_ini, divF, jaco_det_spts, dt, 
          rk_coeff, dt_type, nSpts, nEles, stage, nStages, last_stage);
  }
  else if (equation == EulerNS)
  {
    if (nDims == 2)
      RK_update<4><<<blocks, threads>>>(U_spts, U_ini, divF, jaco_det_spts, dt, 
          rk_coeff, dt_type, nSpts, nEles, stage, nStages, last_stage);
    else
      ThrowException("Under Construction");
  }
}

template <unsigned int nVars>
__global__
void RK_update_source(mdvector_gpu<double> U_spts, mdvector_gpu<double> U_ini, 
    mdvector_gpu<double> divF, mdvector_gpu<double> source, mdvector_gpu<double> jaco_det_spts, 
    mdvector_gpu<double> dt_in, mdvector_gpu<double> rk_coeff, unsigned int dt_type, 
    unsigned int nSpts, unsigned int nEles, unsigned int stage, unsigned int nStages, 
    bool last_stage)
{
  const unsigned int spt = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int ele = blockDim.y * blockIdx.y + threadIdx.y;

  if (spt >= nSpts || ele >= nEles)
    return;

  double dt;
  if (dt_type != 2)
    dt = dt_in(0);
  else
    dt = dt_in(ele);

  double jaco_det = jaco_det_spts(spt,ele);

  if (!last_stage)
  {
    double coeff = rk_coeff(stage);
    for (unsigned int var = 0; var < nVars; var ++)
      U_spts(spt, ele, var) = U_ini(spt, ele, var) - coeff * dt / 
          jaco_det * (divF(spt, ele, var, stage) + source(spt, ele, var));
  }
  else
  {
    double sum[nVars];
    for (unsigned int var = 0; var < nVars; var++)
      sum[var] = 0.;

    for (unsigned int n = 0; n < nStages; n++)
    {
      double coeff = rk_coeff(n);
      for (unsigned int var = 0; var < nVars; var++)
      {
        sum[var] -= coeff * dt / jaco_det * (divF(spt, ele, var, n) + source(spt, ele, var));
      }
    }

    for (unsigned int var = 0; var < nVars; var++)
      U_spts(spt,ele,var) += sum[var];

  }
}

void RK_update_source_wrapper(mdvector_gpu<double> &U_spts, mdvector_gpu<double> &U_ini, 
    mdvector_gpu<double> &divF, mdvector_gpu<double> &source, mdvector_gpu<double> &jaco_det_spts, 
    mdvector_gpu<double> &dt, mdvector_gpu<double> &rk_coeff, unsigned int dt_type, 
    unsigned int nSpts, unsigned int nEles, unsigned int nVars, unsigned int nDims, 
    unsigned int equation, unsigned int stage, unsigned int nStages, bool last_stage)
{
  dim3 threads(16,12);
  dim3 blocks((nSpts + threads.x - 1)/threads.x, (nEles + threads.y - 1)/
      threads.y);

  if (equation == AdvDiff)
  {
      RK_update_source<1><<<blocks, threads>>>(U_spts, U_ini, divF, source, jaco_det_spts, dt, 
          rk_coeff, dt_type, nSpts, nEles, stage, nStages, last_stage);
  }
  else if (equation == EulerNS)
  {
    if (nDims == 2)
      RK_update_source<4><<<blocks, threads>>>(U_spts, U_ini, divF, source, jaco_det_spts, dt, 
          rk_coeff, dt_type, nSpts, nEles, stage, nStages, last_stage);
    else
      ThrowException("Under Construction");
  }
}

__device__
double get_cfl_limit_dev(int order)
{
  switch(order)
  {
    case 0:
      return 1.393;

    case 1:
      return 0.464; 

    case 2:
      return 0.235;

    case 3:
      return 0.139;

    case 4:
      return 0.100;

    case 5:
      return 0.068;
  }
}


__global__
void compute_element_dt(mdvector_gpu<double> dt, mdvector_gpu<double> waveSp_gfpts, 
    mdvector_gpu<double> dA, mdvector_gpu<int> fpt2gfpt, double CFL, int order, 
    unsigned int nFpts, unsigned int nEles)
{
  const unsigned int ele = blockDim.x * blockIdx.x + threadIdx.x;

  if (ele >= nEles)
    return;

  double waveSp_max = 0.0;

  /* Compute maximum wavespeed */
  for (unsigned int fpt = 0; fpt <nFpts; fpt++)
  {
    /* Skip if on ghost edge. */
    int gfpt = fpt2gfpt(fpt,ele);
    if (gfpt == -1)
      continue;

    double waveSp = waveSp_gfpts(gfpt) / dA(gfpt);

    waveSp_max = max(waveSp, waveSp_max);
  }

  /* Note: CFL is applied to parent space element with width 2 */
  dt(ele) = (CFL) * get_cfl_limit_dev(order) * (2.0 / (waveSp_max+1.e-10));
}

void compute_element_dt_wrapper(mdvector_gpu<double> &dt, mdvector_gpu<double> &waveSp, 
    mdvector_gpu<double> &dA, mdvector_gpu<int> &fpt2gfpt, double CFL, int order, 
    unsigned int dt_type, unsigned int nFpts, unsigned int nEles)
{
  unsigned int threads = 192;
  unsigned int blocks = (nEles + threads - 1) / threads;

  compute_element_dt<<<blocks, threads>>>(dt, waveSp, dA, fpt2gfpt, CFL, order, 
      nFpts, nEles);

  if (dt_type == 1)
  {
    /* Get min dt using thrust (pretty slow) */
    thrust::device_ptr<double> dt_ptr = thrust::device_pointer_cast(dt.data());
    thrust::device_ptr<double> min_ptr = thrust::min_element(dt_ptr, dt_ptr + nEles);
    //dt_ptr[0] = min_ptr[0];
    thrust::copy(min_ptr, min_ptr+1, dt_ptr);
  }

}
