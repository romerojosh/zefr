#include "hip/hip_runtime.h"
#include "input.hpp"
#include "faces_kernels.h"
#include "mdvector_gpu.h"

template <unsigned int nDims>
__global__
void compute_Fconv_fpts_AdvDiff(mdvector_gpu<double> F, mdvector_gpu<double> U, 
    unsigned int nFpts, mdvector_gpu<double> AdvDiff_A, 
    unsigned int startFpt, unsigned int endFpt)
{
  const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x + startFpt;

  if (fpt >= endFpt)
    return;

  for (unsigned int dim = 0; dim < nDims; dim++)
  {
      F(fpt, 0, dim, 0) = AdvDiff_A(dim) * U(fpt, 0, 0);

      F(fpt, 0, dim, 1) = AdvDiff_A(dim) * U(fpt, 0, 1);
  }

}

void compute_Fconv_fpts_AdvDiff_wrapper(mdvector_gpu<double> &F, 
    mdvector_gpu<double> &U, unsigned int nFpts, unsigned int nDims, 
    mdvector_gpu<double> &AdvDiff_A, unsigned int startFpt,
    unsigned int endFpt)
{
  unsigned int threads = 192;
  unsigned int blocks = ((endFpt - startFpt + 1) + threads - 1)/threads;

  if (nDims == 2)
    compute_Fconv_fpts_AdvDiff<2><<<blocks, threads>>>(F, U, nFpts, AdvDiff_A, 
        startFpt, endFpt);
  else 
    compute_Fconv_fpts_AdvDiff<3><<<blocks, threads>>>(F, U, nFpts, AdvDiff_A,
        startFpt, endFpt);

}

template <unsigned int nDims>
__global__
void compute_Fconv_fpts_Burgers(mdvector_gpu<double> F, mdvector_gpu<double> U, 
    unsigned int nFpts, unsigned int startFpt, unsigned int endFpt)
{
  const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x + startFpt;

  if (fpt >= endFpt)
    return;

  for (unsigned int dim = 0; dim < nDims; dim++)
  {
      F(fpt, 0, dim, 0) = 0.5 * U(fpt, 0, 0) * U(fpt, 0, 0);

      F(fpt, 0, dim, 1) = 0.5 * U(fpt, 0, 1) * U(fpt, 0, 1);
  }

}

void compute_Fconv_fpts_Burgers_wrapper(mdvector_gpu<double> &F, 
    mdvector_gpu<double> &U, unsigned int nFpts, unsigned int nDims, 
    unsigned int startFpt, unsigned int endFpt)
{
  unsigned int threads = 192;
  unsigned int blocks = ((endFpt - startFpt + 1) + threads - 1)/threads;

  if (nDims == 2)
    compute_Fconv_fpts_Burgers<2><<<blocks, threads>>>(F, U, nFpts, startFpt, endFpt);
  else 
    compute_Fconv_fpts_Burgers<3><<<blocks, threads>>>(F, U, nFpts, startFpt, endFpt);

}

__global__
void compute_Fconv_fpts_2D_EulerNS(mdvector_gpu<double> F, mdvector_gpu<double> U, mdvector_gpu<double> P, 
    unsigned int nFpts, double gamma, unsigned int startFpt, unsigned int endFpt)
{
  const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x + startFpt;

  if (fpt >= endFpt)
    return;

   for (unsigned int slot = 0; slot < 2; slot ++)
   {
     /* Get states */
     double rho = U(fpt, 0, slot);
     double momx = U(fpt, 1, slot);
     double momy = U(fpt, 2, slot);
     double ene = U(fpt, 3, slot);

     /* Compute some primitive variables (keep pressure)*/
     double momF = (momx * momx + momy * momy) / rho;

     double P_d = (gamma - 1.0) * (ene - 0.5 * momF);
     P(fpt, slot) = P_d;

     double H = (ene + P_d) / rho;

     F(fpt, 0, 0, slot) = momx;
     F(fpt, 1, 0, slot) = momx * momx / rho + P_d;
     F(fpt, 2, 0, slot) = momx * momy / rho;
     F(fpt, 3, 0, slot) = momx * H;

     F(fpt, 0, 1, slot) = momy;
     F(fpt, 1, 1, slot) = momy * momx / rho;
     F(fpt, 2, 1, slot) = momy * momy / rho + P_d;
     F(fpt, 3, 1, slot) = momy * H;
   }
}

__global__
void compute_Fconv_fpts_3D_EulerNS(mdvector_gpu<double> F, mdvector_gpu<double> U, mdvector_gpu<double> P, 
    unsigned int nFpts, double gamma, unsigned int startFpt, unsigned int endFpt)
{
  const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x + startFpt;

  if (fpt >= endFpt)
    return;

   for (unsigned int slot = 0; slot < 2; slot ++)
   {
     /* Get states */
     double rho = U(fpt, 0, slot);
     double momx = U(fpt, 1, slot);
     double momy = U(fpt, 2, slot);
     double momz = U(fpt, 3, slot);
     double ene = U(fpt, 4, slot);

     /* Compute some primitive variables (keep pressure)*/
     double momF = (momx * momx + momy * momy + momz * momz) / rho;

     double P_d = (gamma - 1.0) * (ene - 0.5 * momF);
     P(fpt, slot) = P_d;

     double H = (ene + P_d) / rho;

     F(fpt, 0, 0, slot) = momx;
     F(fpt, 1, 0, slot) = momx * momx / rho + P_d;
     F(fpt, 2, 0, slot) = momx * momy / rho;
     F(fpt, 3, 0, slot) = momx * momz / rho;
     F(fpt, 4, 0, slot) = momx * H;

     F(fpt, 0, 1, slot) = momy;
     F(fpt, 1, 1, slot) = momy * momx / rho;
     F(fpt, 2, 1, slot) = momy * momy / rho + P_d;
     F(fpt, 3, 1, slot) = momy * momz / rho;
     F(fpt, 4, 1, slot) = momy * H;

     F(fpt, 0, 2, slot) = momz;
     F(fpt, 1, 2, slot) = momz * momx / rho;
     F(fpt, 2, 2, slot) = momz * momy / rho;
     F(fpt, 3, 2, slot) = momz * momz / rho + P_d;
     F(fpt, 4, 2, slot) = momz * H;
   }
}

void compute_Fconv_fpts_EulerNS_wrapper(mdvector_gpu<double> &F_gfpts, 
    mdvector_gpu<double> &U_gfpts, mdvector_gpu<double> &P_gfpts, 
    unsigned int nFpts, unsigned int nDims, double gamma,
    unsigned int startFpt, unsigned int endFpt)
{
  unsigned int threads = 192;
  //unsigned int blocks = (nFpts + threads - 1)/threads;
  unsigned int blocks = ((endFpt - startFpt + 1) + threads - 1)/threads;

  if (nDims == 2)
  {
    compute_Fconv_fpts_2D_EulerNS<<<blocks, threads>>>(F_gfpts, U_gfpts, P_gfpts, nFpts, gamma,
        startFpt, endFpt);
  }
  else 
  {
    compute_Fconv_fpts_3D_EulerNS<<<blocks, threads>>>(F_gfpts, U_gfpts, P_gfpts, nFpts, gamma, 
        startFpt, endFpt);
  }
}

template <unsigned int nDims>
__global__
void compute_Fvisc_fpts_AdvDiff(mdvector_gpu<double> Fvisc, mdvector_gpu<double> dU, 
    unsigned int nFpts, double AdvDiff_D, unsigned int startFpt,
    unsigned int endFpt)
{
  const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x + startFpt;

  if (fpt >= endFpt)
    return;

  for (unsigned int dim = 0; dim < nDims; dim++)
  {
      Fvisc(fpt, 0, dim, 0) = -AdvDiff_D * dU(fpt, 0, dim, 0);

      Fvisc(fpt, 0, dim, 1) = -AdvDiff_D * dU(fpt, 0, dim, 1);
  }
}

void compute_Fvisc_fpts_AdvDiff_wrapper(mdvector_gpu<double> &Fvisc, 
    mdvector_gpu<double> &dU, unsigned int nFpts, unsigned int nDims, 
    double AdvDiff_D, unsigned int startFpt, unsigned int endFpt)
{
  unsigned int threads = 192;
  unsigned int blocks = ((endFpt - startFpt + 1) + threads - 1)/threads;

  if (nDims == 2)
    compute_Fvisc_fpts_AdvDiff<2><<<blocks, threads>>>(Fvisc, dU, nFpts, AdvDiff_D,
        startFpt, endFpt);
  else 
    compute_Fvisc_fpts_AdvDiff<3><<<blocks, threads>>>(Fvisc, dU, nFpts, AdvDiff_D,
        startFpt, endFpt);
}


__global__
void compute_Fvisc_fpts_2D_EulerNS(mdvector_gpu<double> Fvisc, mdvector_gpu<double> U, 
    mdvector_gpu<double> dU, unsigned int nFpts, double gamma, double prandtl, 
    double mu_in, double c_sth, double rt, bool fix_vis, unsigned int startFpt,
    unsigned int endFpt)
{
  const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x + startFpt;

  if (fpt >= endFpt)
    return;

  for (unsigned int slot = 0; slot < 2; slot++)
  {
    /* Setting variables for convenience */
    /* States */
    double rho = U(fpt, 0, slot);
    double momx = U(fpt, 1, slot);
    double momy = U(fpt, 2, slot);
    double e = U(fpt, 3, slot);

    double u = momx / rho;
    double v = momy / rho;
    double e_int = e / rho - 0.5 * (u*u + v*v);

    /* Gradients */
    double rho_dx = dU(fpt, 0, 0, slot);
    double momx_dx = dU(fpt, 1, 0, slot);
    double momy_dx = dU(fpt, 2, 0, slot);
    double e_dx = dU(fpt, 3, 0, slot);
    
    double rho_dy = dU(fpt, 0, 1, slot);
    double momx_dy = dU(fpt, 1, 1, slot);
    double momy_dy = dU(fpt, 2, 1, slot);
    double e_dy = dU(fpt, 3, 1, slot);

    /* Set viscosity */
    double mu;
    if (fix_vis)
    {
      mu = mu_in;
    }
    /* If desired, use Sutherland's law */
    else
    {
      double rt_ratio = (gamma - 1.0) * e_int / (rt);
      mu = mu_in * pow(rt_ratio,1.5) * (1. + c_sth) / (rt_ratio + c_sth);
    }

    double du_dx = (momx_dx - rho_dx * u) / rho;
    double du_dy = (momx_dy - rho_dy * u) / rho;

    double dv_dx = (momy_dx - rho_dx * v) / rho;
    double dv_dy = (momy_dy - rho_dy * v) / rho;

    double dke_dx = 0.5 * (u*u + v*v) * rho_dx + rho * (u * du_dx + v * dv_dx);
    double dke_dy = 0.5 * (u*u + v*v) * rho_dy + rho * (u * du_dy + v * dv_dy);

    double de_dx = (e_dx - dke_dx - rho_dx * e_int) / rho;
    double de_dy = (e_dy - dke_dy - rho_dy * e_int) / rho;

    double diag = (du_dx + dv_dy) / 3.0;

    double tauxx = 2.0 * mu * (du_dx - diag);
    double tauxy = mu * (du_dy + dv_dx);
    double tauyy = 2.0 * mu * (dv_dy - diag);

    /* Set viscous flux values */
    Fvisc(fpt, 0, 0, slot) = 0.0;
    Fvisc(fpt, 1, 0, slot) = -tauxx;
    Fvisc(fpt, 2, 0, slot) = -tauxy;
    Fvisc(fpt, 3, 0, slot) = -(u * tauxx + v * tauxy + (mu / prandtl) *
        gamma * de_dx);

    Fvisc(fpt, 0, 1, slot) = 0.0;
    Fvisc(fpt, 1, 1, slot) = -tauxy;
    Fvisc(fpt, 2, 1, slot) = -tauyy;
    Fvisc(fpt, 3, 1, slot) = -(u * tauxy + v * tauyy + (mu / prandtl) *
        gamma * de_dy);
  }

}

__global__
void compute_Fvisc_fpts_3D_EulerNS(mdvector_gpu<double> Fvisc, mdvector_gpu<double> U, 
    mdvector_gpu<double> dU, unsigned int nFpts, double gamma, double prandtl, 
    double mu_in, double c_sth, double rt, bool fix_vis, unsigned int startFpt,
    unsigned int endFpt)
{
  const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x + startFpt;

  if (fpt >= endFpt)
    return;

  for (unsigned int slot = 0; slot < 2; slot++)
  {
    /* States */
    double rho = U(fpt, 0, slot);
    double momx = U(fpt, 1, slot);
    double momy = U(fpt, 2, slot);
    double momz = U(fpt, 3, slot);
    double e = U(fpt, 4, slot);

    double u = momx / rho;
    double v = momy / rho;
    double w = momz / rho;
    double e_int = e / rho - 0.5 * (u*u + v*v + w*w);

    /* Gradients */
    double rho_dx = dU(fpt, 0, 0, slot);
    double momx_dx = dU(fpt, 1, 0, slot);
    double momy_dx = dU(fpt, 2, 0, slot);
    double momz_dx = dU(fpt, 3, 0, slot);
    double e_dx = dU(fpt, 4, 0, slot);

    double rho_dy = dU(fpt, 0, 1, slot);
    double momx_dy = dU(fpt, 1, 1, slot);
    double momy_dy = dU(fpt, 2, 1, slot);
    double momz_dy = dU(fpt, 3, 1, slot);
    double e_dy = dU(fpt, 4, 1, slot);

    double rho_dz = dU(fpt, 0, 2, slot);
    double momx_dz = dU(fpt, 1, 2, slot);
    double momy_dz = dU(fpt, 2, 2, slot);
    double momz_dz = dU(fpt, 3, 2, slot);
    double e_dz = dU(fpt, 4, 2, slot);

    /* Set viscosity */
    double mu;
    if (fix_vis)
    {
    mu = mu_in;
    }
    else
    {
    double rt_ratio = (gamma - 1.0) * e_int / (rt);
    mu = mu_in * std::pow(rt_ratio,1.5) * (1. + c_sth) / (rt_ratio + c_sth);
    }

    double du_dx = (momx_dx - rho_dx * u) / rho;
    double du_dy = (momx_dy - rho_dy * u) / rho;
    double du_dz = (momx_dz - rho_dz * u) / rho;

    double dv_dx = (momy_dx - rho_dx * v) / rho;
    double dv_dy = (momy_dy - rho_dy * v) / rho;
    double dv_dz = (momy_dz - rho_dz * v) / rho;

    double dw_dx = (momz_dx - rho_dx * w) / rho;
    double dw_dy = (momz_dy - rho_dy * w) / rho;
    double dw_dz = (momz_dz - rho_dz * w) / rho;

    double dke_dx = 0.5 * (u*u + v*v + w*w) * rho_dx + rho * (u * du_dx + v * dv_dx + w * dw_dx);
    double dke_dy = 0.5 * (u*u + v*v + w*w) * rho_dy + rho * (u * du_dy + v * dv_dy + w * dw_dy);
    double dke_dz = 0.5 * (u*u + v*v + w*w) * rho_dz + rho * (u * du_dz + v * dv_dz + w * dw_dz);

    double de_dx = (e_dx - dke_dx - rho_dx * e_int) / rho;
    double de_dy = (e_dy - dke_dy - rho_dy * e_int) / rho;
    double de_dz = (e_dz - dke_dz - rho_dz * e_int) / rho;

    double diag = (du_dx + dv_dy + dw_dz) / 3.0;

    double tauxx = 2.0 * mu * (du_dx - diag);
    double tauyy = 2.0 * mu * (dv_dy - diag);
    double tauzz = 2.0 * mu * (dw_dz - diag);
    double tauxy = mu * (du_dy + dv_dx);
    double tauxz = mu * (du_dz + dw_dx);
    double tauyz = mu * (dv_dz + dw_dy);

    /* Set viscous flux values */
    Fvisc(fpt, 0, 0, slot) = 0;
    Fvisc(fpt, 1, 0, slot) = -tauxx;
    Fvisc(fpt, 2, 0, slot) = -tauxy;
    Fvisc(fpt, 3, 0, slot) = -tauxz;
    Fvisc(fpt, 4, 0, slot) = -(u * tauxx + v * tauxy + w * tauxz + (mu / prandtl) *
      gamma * de_dx);

    Fvisc(fpt, 0, 1, slot) = 0;
    Fvisc(fpt, 1, 1, slot) = -tauxy;
    Fvisc(fpt, 2, 1, slot) = -tauyy;
    Fvisc(fpt, 3, 1, slot) = -tauyz;
    Fvisc(fpt, 4, 1, slot) = -(u * tauxy + v * tauyy + w * tauyz + (mu / prandtl) *
      gamma * de_dy);

    Fvisc(fpt, 0, 2, slot) = 0;
    Fvisc(fpt, 1, 2, slot) = -tauxz;
    Fvisc(fpt, 2, 2, slot) = -tauyz;
    Fvisc(fpt, 3, 2, slot) = -tauzz;
    Fvisc(fpt, 4, 2, slot) = -(u * tauxz + v * tauyz + w * tauzz + (mu / prandtl) *
      gamma * de_dz);
  }

}

void compute_Fvisc_fpts_EulerNS_wrapper(mdvector_gpu<double> &Fvisc, 
    mdvector_gpu<double> &U, mdvector_gpu<double> &dU, unsigned int nFpts, unsigned int nDims, 
    double gamma, double prandtl, double mu_in, double c_sth, double rt, bool fix_vis,
    unsigned int startFpt, unsigned int endFpt)
{
  unsigned int threads = 192;
  unsigned int blocks = ((endFpt - startFpt + 1) + threads - 1)/threads;

  if (nDims == 2)
  {
    compute_Fvisc_fpts_2D_EulerNS<<<blocks, threads>>>(Fvisc, U, dU, nFpts, gamma, 
        prandtl, mu_in, c_sth, rt, fix_vis, startFpt, endFpt);
  }
  else
  {
    compute_Fvisc_fpts_3D_EulerNS<<<blocks, threads>>>(Fvisc, U, dU, nFpts, gamma, 
        prandtl, mu_in, c_sth, rt, fix_vis, startFpt, endFpt);
  }
}

template<unsigned int nVars, unsigned int nDims, unsigned int equation>
__global__
void apply_bcs(mdvector_gpu<double> U, unsigned int nFpts, unsigned int nGfpts_int, 
    unsigned int nGfpts_bnd, double rho_fs, 
    mdvector_gpu<double> V_fs, double P_fs, double gamma, double R_ref, double T_tot_fs, 
    double P_tot_fs, double T_wall, mdvector_gpu<double> V_wall, mdvector_gpu<double> norm_fs, 
    mdvector_gpu<double> norm, mdvector_gpu<unsigned int> gfpt2bnd, 
    mdvector_gpu<unsigned int> per_fpt_list, mdvector_gpu<int> LDG_bias, mdvector_gpu<int> bc_bias)
{
  const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x + nGfpts_int;

  if (fpt >= nGfpts_int + nGfpts_bnd)
    return;

  unsigned int bnd_id = gfpt2bnd(fpt - nGfpts_int);

  /* Apply specified boundary condition */
  switch(bnd_id)
  {
    case 1:/* Periodic */
    {
      unsigned int per_fpt = per_fpt_list(fpt - nGfpts_int);

      for (unsigned int n = 0; n < nVars; n++)
      {
        U(fpt, n, 1) = U(per_fpt, n, 0);
      }
      break;
    }
  
    case 2: /* Farfield and Supersonic Inlet */
    {
      if (equation == AdvDiff || equation == Burgers)
      {
        /* Set boundaries to zero */
        U(fpt, 0, 1) = 0;
      }
      else
      {
        /* Set boundaries to freestream values */
        U(fpt, 0, 1) = rho_fs;

        double Vsq = 0.0;
        for (unsigned int dim = 0; dim < nDims; dim++)
        {
          U(fpt, dim+1, 1) = rho_fs * V_fs(dim);
          Vsq += V_fs(dim) * V_fs(dim);
        }

        U(fpt, nDims + 1, 1) = P_fs/(gamma-1.0) + 0.5*rho_fs * Vsq; 
      }

      /* Set LDG bias */
      //LDG_bias(fpt) = -1;
      LDG_bias(fpt) = 0;
      bc_bias(fpt) = 1;

      break;
    }

    case 3: /* Supersonic Outlet */
    {
      /* Extrapolate boundary values from interior */
      for (unsigned int n = 0; n < nVars; n++)
        U(fpt, n, 1) = U(fpt, n, 0);

      /* Set LDG bias */
      //LDG_bias(fpt) = -1;
      LDG_bias(fpt) = 0;

      break;
    }

    case 4: /* Subsonic Inlet */
    {
      double VL[3]; double VR[3];
      /*
      if (!input->viscous)
        ThrowException("Subsonic inlet only for viscous flows currently!");
      */

      /* Get states for convenience */
      double rhoL = U(fpt, 0, 0);

      double Vsq = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        VL[dim] = U(fpt, dim+1, 0) / rhoL;
        Vsq += VL[dim] * VL[dim];
      }

      double eL = U(fpt, nDims + 1 ,0);
      double PL = (gamma - 1.0) * (eL - 0.5 * rhoL * Vsq);


      /* Compute left normal velocity and dot product of normal*/
      double VnL = 0.0;
      double alpha = 0.0;

      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        VnL += VL[dim] * norm(fpt, dim, 0);
        alpha += norm_fs(dim) * norm(fpt, dim, 0);
      }

      /* Compute speed of sound */
      double cL = std::sqrt(gamma * PL / rhoL);

      /* Extrapolate Riemann invariant */
      double R_plus  = VnL + 2.0 * cL / (gamma - 1.0);

      /* Specify total enthalpy */
      double H_tot = gamma * R_ref / (gamma - 1.0) * T_tot_fs;

      /* Compute total speed of sound squared */
      double c_tot_sq = (gamma - 1.0) * (H_tot - (eL + PL) / rhoL + 0.5 * Vsq) + cL * cL;

      /* Coefficients of Quadratic equation */
      double aa = 1.0 + 0.5 * (gamma - 1.0) * alpha * alpha;
      double bb = -(gamma - 1.0) * alpha * R_plus;
      double cc = 0.5 * (gamma - 1.0) * R_plus * R_plus - 2.0 * c_tot_sq / (gamma - 1.0);

      /* Solve quadratic for right velocity */
      double dd = bb * bb  - 4.0 * aa * cc;
      dd = std::sqrt(max(dd, 0.0));  // Max to keep from producing NaN
      double VR_mag = (dd - bb) / (2.0 * aa);
      VR_mag = max(VR_mag, 0.0);
      double VR_mag_sq = VR_mag * VR_mag;

      /* Compute right speed of sound and Mach */
      /* Note: Need to verify what is going on here. */
      double cR_sq = c_tot_sq - 0.5 * (gamma - 1.0) * VR_mag_sq;
      double Mach_sq = VR_mag_sq / cR_sq;
      Mach_sq = min(Mach_sq, 1.0); // Clamp to Mach = 1
      VR_mag_sq = Mach_sq * cR_sq;
      VR_mag = std::sqrt(VR_mag_sq);
      cR_sq = c_tot_sq - 0.5 * (gamma - 1.0) * VR_mag_sq;

      /* Compute right states */

      double TR = cR_sq / (gamma * R_ref);
      double PR = P_tot_fs * std::pow(TR / T_tot_fs, gamma/ (gamma - 1.0));

      U(fpt, 0, 1) = PR / (R_ref * TR);

      Vsq = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        VR[dim] = VR_mag * norm_fs(dim);
        U(fpt, dim+1, 1) = U(fpt, 0, 1) * VR[dim];
        Vsq += VR[dim] * VR[dim];
      }

      U(fpt, nDims + 1, 1) = PR / (gamma - 1.0) + 0.5 * U(fpt, 0, 1) * Vsq;

      /* Set LDG bias */
      //LDG_bias(fpt) = -1;
      LDG_bias(fpt) = 0;

      break;
    }

    case 5: /* Subsonic Outlet */
    { 
      /* Extrapolate Density */
      U(fpt, 0, 1) = U(fpt, 0, 0);

      /* Extrapolate Momentum */
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        U(fpt, dim+1, 1) =  U(fpt, dim+1, 0);
      }

      double momF = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        momF += U(fpt, dim + 1, 0) * U(fpt, dim + 1, 0);
      }

      momF /= U(fpt, 0, 0);

      /* Fix pressure */
      U(fpt, nDims + 1, 1) = P_fs/(gamma-1.0) + 0.5 * momF; 

      /* Set LDG bias */
      //LDG_bias(fpt) = -1;
      LDG_bias(fpt) = 0;

      break;

      /*
      if (!input->viscous)
        ThrowException("Subsonic outlet only for viscous flows currently!");
      */

      double VL[3]; double VR[3];

      /* Get states for convenience */
      double rhoL = U(fpt, 0, 0);

      double Vsq = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        VL[dim] = U(fpt, dim+1, 0) / rhoL;
        Vsq += VL[dim] * VL[dim];
      }

      double eL = U(fpt, nDims + 1, 0);
      double PL = (gamma - 1.0) * (eL - 0.5 * rhoL * Vsq);

      /* Compute left normal velocity */
      double VnL = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        VnL += VL[dim] * norm(fpt, dim, 0);
      }

      /* Compute speed of sound */
      double cL = std::sqrt(gamma * PL / rhoL);

      /* Extrapolate Riemann invariant */
      double R_plus  = VnL + 2.0 * cL / (gamma - 1.0);

      /* Extrapolate entropy */
      double s = PL / std::pow(rhoL, gamma);

      /* Fix pressure */
      double PR = P_fs;

      U(fpt, 0, 1) = std::pow(PR / s, 1.0 / gamma);

      /* Compute right speed of sound and velocity magnitude */
      double cR = std::sqrt(gamma * PR/ U(fpt, 0, 1));

      double VnR = R_plus - 2.0 * cR / (gamma - 1.0);

      Vsq = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        VR[dim] = VL[dim] + (VnR - VnL) * norm(fpt, dim, 0);
        U(fpt, dim + 1, 1) = U(fpt, 0, 1) * VR[dim];
        Vsq += VR[dim] * VR[dim];
      }

      U(fpt, nDims + 1, 1) = PR / (gamma - 1.0) + 0.5 * U(fpt, 0, 1) * Vsq;

      /* Set LDG bias */
      //LDG_bias(fpt) = -1;
      LDG_bias(fpt) = 0;

      break;
    }

    case 6: /* Characteristic (from HiFiLES) */
    {
      /* Compute wall normal velocities */
      double VnL = 0.0; double VnR = 0.0;

      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        VnL += U(fpt, dim+1, 0) / U(fpt, 0, 0) * norm(fpt, dim, 0);
        VnR += V_fs(dim) * norm(fpt, dim, 0);
      }
    

      /* Compute pressure. TODO: Compute pressure once!*/
      double momF = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        momF += U(fpt, dim + 1, 0) * U(fpt, dim + 1, 0);
      }

      momF /= U(fpt, 0, 0);

      double PL = (gamma - 1.0) * (U(fpt, nDims + 1, 0) - 0.5 * momF);
      double PR = P_fs;

      /* Compute Riemann Invariants */
      double Rp = VnL + 2.0 / (gamma - 1) * std::sqrt(gamma * PL / 
          U(fpt, 0,0));
      double Rn = VnR - 2.0 / (gamma - 1) * std::sqrt(gamma * PR / 
          rho_fs);

      double cstar = 0.25 * (gamma - 1) * (Rp - Rn);
      double ustarn = 0.5 * (Rp + Rn);

      if (VnL < 0.0) /* Case 1: Inflow */
      {
        double s_inv = std::pow(rho_fs, gamma) / PR;

        double Vsq = 0.0;
        for (unsigned int dim = 0; dim < nDims; dim++)
          Vsq += V_fs(dim) * V_fs(dim);

        double H_fs = gamma / (gamma - 1.0) * PR / rho_fs +
            0.5 * Vsq;

        double rhoR = std::pow(1.0 / gamma * (s_inv * cstar * cstar), 1.0/ 
            (gamma - 1.0));

        U(fpt, 0, 1) = rhoR;
        for (unsigned int dim = 0; dim < nDims; dim++)
          U(fpt, dim+1, 1) = rhoR * (ustarn * norm(fpt, dim, 0) + V_fs(dim) - VnR * 
            norm(fpt, dim, 0));

        PR = rhoR / gamma * cstar * cstar;
        U(fpt, nDims + 1, 1) = rhoR * H_fs - PR;
        
      }
      else  /* Case 2: Outflow */
      {
        double rhoL = U(fpt, 0, 0);
        double s_inv = std::pow(rhoL, gamma) / PL;

        double rhoR = std::pow(1.0 / gamma * (s_inv * cstar * cstar), 1.0/ 
            (gamma - 1.0));

        U(fpt, 0, 1) = rhoR;

        for (unsigned int dim = 0; dim < nDims; dim++)
          U(fpt, dim + 1, 1) = rhoR * (ustarn * norm(fpt, dim, 0) +(U(fpt, dim + 1, 0) / 
                U(fpt, 0, 0) - VnL * norm(fpt, dim, 0)));

        double PR = rhoR / gamma * cstar * cstar;

        double Vsq = 0.0;
        for (unsigned int dim = 0; dim < nDims; dim++)
          Vsq += U(fpt, dim+1, 1) * U(fpt, dim+1, 1) / (rhoR * rhoR) ;
        
        U(fpt, nDims + 1, 1) = PR / (gamma - 1.0) + 0.5 * rhoR * Vsq; 
      }

      /* Set LDG bias */
      //LDG_bias(fpt) = -1;
      LDG_bias(fpt) = 0;
      bc_bias(fpt) = 1;

      break;

    }
    case 7:
    case 8: /* Slip Wall */
    {
      double momN = 0.0;

      /* Compute wall normal momentum */
      for (unsigned int dim = 0; dim < nDims; dim++)
        momN += U(fpt, dim+1, 0) * norm(fpt, dim, 0);

      U(fpt, 0, 1) = U(fpt, 0, 0);

      for (unsigned int dim = 0; dim < nDims; dim++)
        /* Set boundary state to cancelled normal velocity (strong)*/
        //U(fpt, dim+1, 1) = U(fpt, dim+1, 0) - momN * norm(fpt, dim, 0);
        /* Set boundary state to reflect normal velocity */
        U(fpt, dim+1, 1) = U(fpt, dim+1, 0) - 2.0 * momN * norm(fpt, dim, 0);

      //U(fpt, nDims + 1, 1) = U(fpt, nDims + 1, 0) - 0.5 * (momN * momN) / U(fpt, 0, 0);
      U(fpt, nDims + 1, 1) = U(fpt, nDims + 1, 0);

      /* Set LDG bias */
      //LDG_bias(fpt) = -1;
      LDG_bias(fpt) = 0;
      bc_bias(fpt) = 1;

      break;
    }

    case 9: /* No-slip Wall (isothermal) */
    {
      /*
      if (!input->viscous)
        ThrowException("No slip wall boundary only for viscous flows!");
      */

      double momF = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        momF += U(fpt, dim + 1, 0) * U(fpt, dim + 1, 0);
      }

      momF /= U(fpt, 0, 0);

      double PL = (gamma - 1.0) * (U(fpt, nDims + 1, 0) - 0.5 * momF);
      double PR = PL;
      //double TR = T_wall;
      double TR = 1; // T_wall = T_fs (hardcoded for couette flow)
      
      U(fpt, 0, 1) = PR / (R_ref * TR);

      /* Set velocity to zero */
      for (unsigned int dim = 0; dim < nDims; dim++)
        U(fpt, dim+1, 1) = 0.0;

      U(fpt, nDims + 1, 1) = PR / (gamma - 1.0);

      /* Set LDG bias */
      LDG_bias(fpt) = -1;

      break;
    }

    case 10: /* No-slip Wall (isothermal and moving) */
    {
      /*
      if (!input->viscous)
        ThrowException("No slip wall boundary only for viscous flows!");
      */

      double momF = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        momF += U(fpt, dim + 1, 0) * U(fpt, dim + 1, 0);
      }

      momF /= U(fpt, 0, 0);

      double PL = (gamma - 1.0) * (U(fpt, nDims + 1, 0) - 0.5 * momF);

      double PR = PL;
      double TR = T_wall;
      
      U(fpt, 0, 1) = PR / (R_ref * TR);

      /* Set velocity to wall velocity */
      double V_wall_sq = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        U(fpt, dim+1, 1) = U(fpt, 0 , 1) * V_wall(dim);
        V_wall_sq += V_wall(dim) * V_wall(dim);
      }

      U(fpt, nDims + 1, 1) = PR / (gamma - 1.0) + 0.5 * U(fpt, 0 , 1) * V_wall_sq;

      /* Set LDG bias */
      LDG_bias(fpt) = -1;

      break;
    }

    case 11: /* No-slip Wall (adiabatic) */
    {
      /*
      if (!input->viscous)
        ThrowException("No slip wall boundary only for viscous flows!");
      */

      /* Extrapolate density */
      U(fpt, 0, 1) = U(fpt, 0, 0);

      /* Extrapolate pressure */
      double momF = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        momF += U(fpt, dim + 1, 0) * U(fpt, dim + 1, 0);
      }

      momF /= U(fpt, 0, 0);

      double PL = (gamma - 1.0) * (U(fpt, nDims + 1, 0) - 0.5 * momF);
      double PR = PL; 

      /* Set velocity to zero */
      for (unsigned int dim = 0; dim < nDims; dim++)
        U(fpt, dim+1, 1) = 0.0;
        //U(fpt, dim+1, 1) = -U(fpt, dim+1, 0);

      U(fpt, nDims + 1, 1) = PR / (gamma - 1.0);
      //U(fpt, nDims + 1, 1) = U(fpt, nDims + 1, 0);

      /* Set LDG bias */
      LDG_bias(fpt) = 1;

      break;
    }

    case 12: /* No-slip Wall (adiabatic and moving) */
    {
      /*
      if (!input->viscous)
        ThrowException("No slip wall boundary only for viscous flows!");
      */

      /* Extrapolate density */
      U(fpt, 0, 1) = U(fpt, 0, 0);

      /* Extrapolate pressure */
      double momF = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        momF += U(fpt, dim + 1, 0) * U(fpt, dim + 1, 0);
      }

      momF /= U(fpt, 0, 0);

      double PL = (gamma - 1.0) * (U(fpt, nDims + 1, 0) - 0.5 * momF);
      double PR = PL; 

      /* Set velocity to wall velocity */
      double V_wall_sq = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        U(fpt, dim+1, 1) = U(fpt, 0 , 1) * V_wall(dim);
        V_wall_sq += V_wall(dim) * V_wall(dim);
      }

      U(fpt, nDims + 1, 1) = PR / (gamma - 1.0) + 0.5 * U(fpt, 0, 1) * V_wall_sq;

      /* Set LDG bias */
      LDG_bias(fpt) = 1;

      break;
    }
  }

}

void apply_bcs_wrapper(mdvector_gpu<double> &U, unsigned int nFpts, unsigned int nGfpts_int, 
    unsigned int nGfpts_bnd, unsigned int nVars, unsigned int nDims, double rho_fs, 
    mdvector_gpu<double> &V_fs, double P_fs, double gamma, double R_ref, double T_tot_fs, 
    double P_tot_fs, double T_wall, mdvector_gpu<double> &V_wall, mdvector_gpu<double> &norm_fs, 
    mdvector_gpu<double> &norm, mdvector_gpu<unsigned int> &gfpt2bnd, 
    mdvector_gpu<unsigned int> &per_fpt_list, mdvector_gpu<int> &LDG_bias, mdvector_gpu<int> &bc_bias, unsigned int equation)
{
  unsigned int threads = 192;
  unsigned int blocks = (nGfpts_bnd + threads - 1)/threads;

  if (blocks != 0)
  {
    if (equation == AdvDiff)
    {
      if (nDims == 2)
        apply_bcs<1, 2, AdvDiff><<<blocks, threads>>>(U, nFpts, nGfpts_int, nGfpts_bnd, rho_fs, V_fs, P_fs, 
            gamma, R_ref,T_tot_fs, P_tot_fs, T_wall, V_wall, norm_fs, norm, gfpt2bnd, per_fpt_list, LDG_bias, bc_bias); 
      else
        apply_bcs<1, 3, AdvDiff><<<blocks, threads>>>(U, nFpts, nGfpts_int, nGfpts_bnd, rho_fs, V_fs, P_fs, 
            gamma, R_ref,T_tot_fs, P_tot_fs, T_wall, V_wall, norm_fs, norm, gfpt2bnd, per_fpt_list, LDG_bias, bc_bias); 
    }
    else if (equation == Burgers)
    {
      if (nDims == 2)
        apply_bcs<1, 2, Burgers><<<blocks, threads>>>(U, nFpts, nGfpts_int, nGfpts_bnd, rho_fs, V_fs, P_fs, 
            gamma, R_ref,T_tot_fs, P_tot_fs, T_wall, V_wall, norm_fs, norm, gfpt2bnd, per_fpt_list, LDG_bias, bc_bias); 
      else
        apply_bcs<1, 3, Burgers><<<blocks, threads>>>(U, nFpts, nGfpts_int, nGfpts_bnd, rho_fs, V_fs, P_fs, 
            gamma, R_ref,T_tot_fs, P_tot_fs, T_wall, V_wall, norm_fs, norm, gfpt2bnd, per_fpt_list, LDG_bias, bc_bias); 
    }
    else if (equation == EulerNS)
    {
      if (nDims == 2)
        apply_bcs<4, 2, EulerNS><<<blocks, threads>>>(U, nFpts, nGfpts_int, nGfpts_bnd, rho_fs, V_fs, P_fs, 
            gamma, R_ref,T_tot_fs, P_tot_fs, T_wall, V_wall, norm_fs, norm, gfpt2bnd, per_fpt_list, LDG_bias, bc_bias); 
      else
        apply_bcs<5, 3, EulerNS><<<blocks, threads>>>(U, nFpts, nGfpts_int, nGfpts_bnd, rho_fs, V_fs, P_fs, 
            gamma, R_ref,T_tot_fs, P_tot_fs, T_wall, V_wall, norm_fs, norm, gfpt2bnd, per_fpt_list, LDG_bias, bc_bias); 
    }
  }
}

template<unsigned int nVars, unsigned int nDims>
__global__
void apply_bcs_dU(mdvector_gpu<double> dU, mdvector_gpu<double> U, mdvector_gpu<double> norm_gfpt,
    unsigned int nFpts, unsigned int nGfpts_int, unsigned int nGfpts_bnd, 
    mdvector_gpu<unsigned int> gfpt2bnd, mdvector_gpu<unsigned int> per_fpt_list)
{
  const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x + nGfpts_int;

  if (fpt >= nGfpts_int + nGfpts_bnd)
    return;

  unsigned int bnd_id = gfpt2bnd(fpt - nGfpts_int);

  /* Apply specified boundary condition */
  if (bnd_id == 1) /* Periodic */
  {
    for (unsigned int dim = 0; dim < nDims; dim++)
    {
      for (unsigned int n = 0; n < nVars; n++)
      {
	  unsigned int per_fpt = per_fpt_list(fpt - nGfpts_int);
          dU(fpt, n, dim, 1) = dU(per_fpt, n, dim, 0);
      }
    }
  }
  else if(bnd_id == 11 || bnd_id == 12) /* Adibatic Wall */
  {
    double norm[nDims];

    for (unsigned int dim = 0; dim < nDims; dim++)
      norm[dim] = norm_gfpt(fpt, dim, 0);

    /* Extrapolate density gradient */
    for (unsigned int dim = 0; dim < nDims; dim++)
    {
      dU(fpt, 0, dim, 1) = dU(fpt, 0, dim, 0);
    }

    if (nDims == 2)
    {
      /* Compute energy gradient */
      /* Get right states and velocity gradients*/
      double rho = U(fpt, 0, 0);
      double momx = U(fpt, 1, 0);
      double momy = U(fpt, 2, 0);
      double E = U(fpt, 3, 0);

      double u = momx / rho;
      double v = momy / rho;
      //double e_int = e / rho - 0.5 * (u*u + v*v);

      double rho_dx = dU(fpt, 0, 0, 0);
      double momx_dx = dU(fpt, 1, 0, 0);
      double momy_dx = dU(fpt, 2, 0, 0);
      double E_dx = dU(fpt, 3, 0, 0);
      
      double rho_dy = dU(fpt, 0, 1, 0);
      double momx_dy = dU(fpt, 1, 1, 0);
      double momy_dy = dU(fpt, 2, 1, 0);
      double E_dy = dU(fpt, 3, 1, 0);

      double du_dx = (momx_dx - rho_dx * u) / rho;
      double du_dy = (momx_dy - rho_dy * u) / rho;

      double dv_dx = (momy_dx - rho_dx * v) / rho;
      double dv_dy = (momy_dy - rho_dy * v) / rho;

      /* Option 1: Extrapolate momentum gradients */
      dU(fpt, 1, 0, 1) = dU(fpt, 1, 0, 0);
      dU(fpt, 1, 1, 1) = dU(fpt, 1, 1, 0);
      dU(fpt, 2, 0, 1) = dU(fpt, 2, 0, 0);
      dU(fpt, 2, 1, 1) = dU(fpt, 2, 1, 0);

      /* Option 2: Enforce constraint on tangential velocity gradient */
      //double du_dn = du_dx * norm[0] + du_dy * norm[1];
      //double dv_dn = dv_dx * norm[0] + dv_dy * norm[1];

      //dU(fpt, 1, 0, 1) = rho * du_dn * norm[0];
      //dU(fpt, 1, 1, 1) = rho * du_dn * norm[1];
      //dU(fpt, 2, 0, 1) = rho * dv_dn * norm[0];
      //dU(fpt, 2, 1, 1) =  rho * dv_dn * norm[1];

     // double dke_dx = 0.5 * (u*u + v*v) * rho_dx + rho * (u * du_dx + v * dv_dx);
     // double dke_dy = 0.5 * (u*u + v*v) * rho_dy + rho * (u * du_dy + v * dv_dy);

      /* Compute temperature gradient (actually C_v * rho * dT) */
      double dT_dx = E_dx - rho_dx * E/rho - rho * (u * du_dx + v * dv_dx);
      double dT_dy = E_dy - rho_dy * E/rho - rho * (u * du_dy + v * dv_dy);

      /* Compute wall normal temperature gradient */
      double dT_dn = dT_dx * norm[0] + dT_dy * norm[1];

      /* Option 1: Simply remove contribution of dT from total energy gradient */
      dU(fpt, 3, 0, 1) = E_dx - dT_dn * norm[0]; 
      dU(fpt, 3, 1, 1) = E_dy - dT_dn * norm[1]; 

      /* Option 2: Reconstruct energy gradient using right states (E = E_r, u = 0, v = 0, rho = rho_r = rho_l) */
      //dU(fpt, 3, 0, 1) = (dT_dx - dT_dn * norm[0]) + rho_dx * U(fpt, 3, 1) / rho; 
      //dU(fpt, 3, 1, 1) = (dT_dy - dT_dn * norm[1]) + rho_dy * U(fpt, 3, 1) / rho; 
    }
    else
    {
      /* Compute energy gradient */
      /* Get right states and velocity gradients*/
      double rho = U(fpt, 0, 0);
      double momx = U(fpt, 1, 0);
      double momy = U(fpt, 2, 0);
      double momz = U(fpt, 3, 0);
      double E = U(fpt, 4, 0);

      double u = momx / rho;
      double v = momy / rho;
      double w = momz / rho;

      /* Gradients */
      double rho_dx = dU(fpt, 0, 0, 0);
      double momx_dx = dU(fpt, 1, 0, 0);
      double momy_dx = dU(fpt, 2, 0, 0);
      double momz_dx = dU(fpt, 3, 0, 0);
      double E_dx = dU(fpt, 4, 0, 0);

      double rho_dy = dU(fpt, 0, 1, 0);
      double momx_dy = dU(fpt, 1, 1, 0);
      double momy_dy = dU(fpt, 2, 1, 0);
      double momz_dy = dU(fpt, 3, 1, 0);
      double E_dy = dU(fpt, 4, 1, 0);

      double rho_dz = dU(fpt, 0, 2, 0);
      double momx_dz = dU(fpt, 1, 2, 0);
      double momy_dz = dU(fpt, 2, 2, 0);
      double momz_dz = dU(fpt, 3, 2, 0);
      double E_dz = dU(fpt, 4, 2, 0);

      double du_dx = (momx_dx - rho_dx * u) / rho;
      double du_dy = (momx_dy - rho_dy * u) / rho;
      double du_dz = (momx_dz - rho_dz * u) / rho;

      double dv_dx = (momy_dx - rho_dx * v) / rho;
      double dv_dy = (momy_dy - rho_dy * v) / rho;
      double dv_dz = (momy_dz - rho_dz * v) / rho;

      double dw_dx = (momz_dx - rho_dx * w) / rho;
      double dw_dy = (momz_dy - rho_dy * w) / rho;
      double dw_dz = (momz_dz - rho_dz * w) / rho;

      /* Option 1: Extrapolate momentum gradients */
      dU(fpt, 1, 0, 1) = dU(fpt, 1, 0, 0);
      dU(fpt, 1, 1, 1) = dU(fpt, 1, 1, 0);
      dU(fpt, 1, 2, 1) = dU(fpt, 1, 2, 0);

      dU(fpt, 2, 0, 1) = dU(fpt, 2, 0, 0);
      dU(fpt, 2, 1, 1) = dU(fpt, 2, 1, 0);
      dU(fpt, 2, 2, 1) = dU(fpt, 2, 2, 0);

      dU(fpt, 3, 0, 1) = dU(fpt, 3, 0, 0);
      dU(fpt, 3, 1, 1) = dU(fpt, 3, 1, 0);
      dU(fpt, 3, 2, 1) = dU(fpt, 3, 2, 0);

      /* Option 2: Enforce constraint on tangential velocity gradient */
      //double du_dn = du_dx * norm[0] + du_dy * norm[1] + du_dz * norm[2];
      //double dv_dn = dv_dx * norm[0] + dv_dy * norm[1] + dv_dz * norm[2];
      //double dw_dn = dw_dx * norm[0] + dw_dy * norm[1] + dw_dz * norm[2];

      //dU(fpt, 1, 0, 1) = rho * du_dn * norm[0];
      //dU(fpt, 1, 1, 1) = rho * du_dn * norm[1];
      //dU(fpt, 1, 2, 1) = rho * du_dn * norm[2];
      //dU(fpt, 2, 0, 1) = rho * dv_dn * norm[0];
      //dU(fpt, 2, 1, 1) =  rho * dv_dn * norm[1];
      //dU(fpt, 2, 2, 1) =  rho * dv_dn * norm[2];
      //dU(fpt, 3, 0, 1) = rho * dw_dn * norm[0];
      //dU(fpt, 3, 1, 1) =  rho * dw_dn * norm[1];
      //dU(fpt, 3, 2, 1) =  rho * dw_dn * norm[2];

     // double dke_dx = 0.5 * (u*u + v*v + w*w) * rho_dx + rho * (u * du_dx + v * dv_dx + w * dw_dx);
     // double dke_dy = 0.5 * (u*u + v*v + w*w) * rho_dy + rho * (u * du_dy + v * dv_dy + w * dw_dy);
     // double dke_dz = 0.5 * (u*u + v*v + w*w) * rho_dz + rho * (u * du_dz + v * dv_dz + w * dw_dz);

      /* Compute temperature gradient (actually C_v * rho * dT) */
      double dT_dx = E_dx - rho_dx * E/rho - rho * (u * du_dx + v * dv_dx + w * dw_dx);
      double dT_dy = E_dy - rho_dy * E/rho - rho * (u * du_dy + v * dv_dy + w * dw_dy);
      double dT_dz = E_dz - rho_dz * E/rho - rho * (u * du_dz + v * dv_dz + w * dw_dz);

      /* Compute wall normal temperature gradient */
      double dT_dn = dT_dx * norm[0] + dT_dy * norm[1] + dT_dz * norm[2];

      /* Option 1: Simply remove contribution of dT from total energy gradient */
      dU(fpt, 4, 0, 1) = E_dx - dT_dn * norm[0]; 
      dU(fpt, 4, 1, 1) = E_dy - dT_dn * norm[1]; 
      dU(fpt, 4, 2, 1) = E_dz - dT_dn * norm[2]; 

      /* Option 2: Reconstruct energy gradient using right states (E = E_r, u = 0, v = 0, rho = rho_r = rho_l) */
      //dU(fpt, 4, 0, 1) = (dT_dx - dT_dn * norm[0]) + rho_dx * U(fpt, 4, 1) / rho; 
      //dU(fpt, 4, 1, 1) = (dT_dy - dT_dn * norm[1]) + rho_dy * U(fpt, 4, 1) / rho; 
      //dU(fpt, 4, 2, 1) = (dT_dz - dT_dn * norm[2]) + rho_dz * U(fpt, 4, 1) / rho; 

    }

  }
  else
  {
    for (unsigned int dim = 0; dim < nDims; dim++)
    {
      for (unsigned int n = 0; n < nVars; n++)
      {
        dU(fpt, n, dim, 1) = dU(fpt, n, dim , 0);
      }
    }
  }

}


void apply_bcs_dU_wrapper(mdvector_gpu<double> &dU, mdvector_gpu<double> &U, mdvector_gpu<double> &norm, 
    unsigned int nFpts, unsigned int nGfpts_int, unsigned int nGfpts_bnd, unsigned int nVars, 
    unsigned int nDims, mdvector_gpu<unsigned int> &gfpt2bnd, mdvector_gpu<unsigned int> &per_fpt_list)
{
  unsigned int threads = 192;
  unsigned int blocks = (nGfpts_bnd + threads - 1)/threads;

  if (blocks != 0)
  {
    if (nDims == 2)
      apply_bcs_dU<4, 2><<<blocks, threads>>>(dU, U, norm, nFpts, nGfpts_int, nGfpts_bnd,
          gfpt2bnd, per_fpt_list);
    else
      apply_bcs_dU<5, 3><<<blocks, threads>>>(dU, U, norm, nFpts, nGfpts_int, nGfpts_bnd,
          gfpt2bnd, per_fpt_list);
  }
}

template<unsigned int nVars, unsigned int nDims, unsigned int equation>
__global__
void rusanov_flux(mdvector_gpu<double> U, mdvector_gpu<double> Fconv, 
    mdvector_gpu<double> Fcomm, mdvector_gpu<double> P, mdvector_gpu<double> AdvDiff_A, 
    mdvector_gpu<double> norm_gfpts, mdvector_gpu<int> outnorm_gfpts, 
    mdvector_gpu<double> waveSp_gfpts, mdvector_gpu<int> LDG_bias, mdvector_gpu<int> bc_bias,
    double gamma, double rus_k, unsigned int nFpts, unsigned int startFpt,
    unsigned int endFpt)
{
  const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x + startFpt;

  if (fpt >= endFpt)
    return;

  /* Apply central flux at boundaries */
  double k = rus_k;
  if (bc_bias(fpt))
  {
    k = 1.0;
  }

  double FL[nVars]; double FR[nVars];
  double WL[nVars]; double WR[nVars];
  double norm[nDims]; double outnorm[2];

  for (unsigned int dim = 0; dim < nDims; dim++)
  {
    norm[dim] = norm_gfpts(fpt, dim, 0);
  }

  outnorm[0] = outnorm_gfpts(fpt, 0);
  outnorm[1] = outnorm_gfpts(fpt, 1);


  /* Initialize FL, FR */
  for (unsigned int n = 0; n < nVars; n++)
  {
    FL[n] = 0.0; FR[n] = 0.0;
  }

  /* Get interface-normal flux components  (from L to R) */
  for (unsigned int dim = 0; dim < nDims; dim++)
  {
    for (unsigned int n = 0; n < nVars; n++)
    {
      FL[n] += Fconv(fpt, n, dim, 0) * norm[dim];
      FR[n] += Fconv(fpt, n, dim, 1) * norm[dim];
    }
  }

  /* If on boundary, set common to right state flux */
  if (LDG_bias(fpt) != 0)
  {
    for (unsigned int n = 0; n < nVars; n++)
    {
      Fcomm(fpt, n, 0) = FR[n] * outnorm[0];
      Fcomm(fpt, n, 1) = FR[n] * -outnorm[1];
    }

    return;
  }

  /* Get left and right state variables */
  for (unsigned int n = 0; n < nVars; n++)
  {
    WL[n] = U(fpt, n, 0); WR[n] = U(fpt, n, 1);
  }

  /* Get numerical wavespeed */
  double waveSp = 0.;
  if (equation == AdvDiff) 
  {
    for (unsigned int dim = 0; dim < nDims; dim++)
    {
      waveSp += AdvDiff_A(dim) * norm[dim];
    }

    waveSp_gfpts(fpt) = waveSp;

    waveSp = std::abs(waveSp);
  }
  else if (equation == Burgers) 
  {
    double AnL = 0;
    double AnR = 0;

    for (unsigned int dim = 0; dim < nDims; dim++)
    {
      AnL += WL[0] * norm[dim];
      AnR += WR[0] * norm[dim];
    }

    waveSp = max(std::abs(AnL), std::abs(AnR));

    // NOTE: Can I just store absolute of waveSp?
    waveSp_gfpts(fpt) = waveSp;
    waveSp = std::abs(waveSp);
  }
  else if (equation == EulerNS)
  {
    /* Compute speed of sound */
    double aL = std::sqrt(std::abs(gamma * P(fpt, 0) / WL[0]));
    double aR = std::sqrt(std::abs(gamma * P(fpt, 1) / WR[0]));

    /* Compute normal velocities */
    double VnL = 0.0; double VnR = 0.0;
    for (unsigned int dim = 0; dim < nDims; dim++)
    {
      VnL += WL[dim+1]/WL[0] * norm[dim];
      VnR += WR[dim+1]/WR[0] * norm[dim];
    }

    waveSp = max(std::abs(VnL) + aL, std::abs(VnR) + aR);

    // NOTE: Can I just store absolute of waveSp?
    waveSp_gfpts(fpt) = waveSp;
    waveSp = std::abs(waveSp);
  }

  /* Compute common normal flux */
  for (unsigned int n = 0; n < nVars; n++)
  {
    double F = 0.5 * (FR[n]+FL[n]) - 0.5 * waveSp * (1.0-k) * 
        (WR[n]-WL[n]);
    Fcomm(fpt, n, 0) = F * outnorm[0];
    Fcomm(fpt, n, 1) = F * -outnorm[1];
  }

}

void rusanov_flux_wrapper(mdvector_gpu<double> &U, mdvector_gpu<double> &Fconv, 
    mdvector_gpu<double> &Fcomm, mdvector_gpu<double> &P, mdvector_gpu<double> &AdvDiff_A, 
    mdvector_gpu<double> &norm, mdvector_gpu<int> &outnorm, mdvector_gpu<double> &waveSp, 
    mdvector_gpu<int> &LDG_bias, mdvector_gpu<int> &bc_bias, double gamma, double rus_k, unsigned int nFpts, unsigned int nVars, 
    unsigned int nDims, unsigned int equation, unsigned int startFpt, unsigned int endFpt)
{
  unsigned int threads = 256;
  //unsigned int blocks = (nFpts + threads - 1)/threads;
  unsigned int blocks = ((endFpt - startFpt + 1) + threads - 1)/threads;
  //int threads; int minBlocks; int blocks;

  //hipOccupancyMaxPotentialBlockSize(&minBlocks, &threads, (const void*)rusanov_flux, 0, nFpts);

  //blocks = (nFpts + threads - 1) / threads;

  if (equation == AdvDiff)
  {
    if (nDims == 2)
      rusanov_flux<1, 2, AdvDiff><<<blocks, threads>>>(U, Fconv, Fcomm, P, AdvDiff_A, norm, outnorm, 
          waveSp, LDG_bias, bc_bias, gamma, rus_k, nFpts, startFpt, endFpt);
    else
      rusanov_flux<1, 3, AdvDiff><<<blocks, threads>>>(U, Fconv, Fcomm, P, AdvDiff_A, norm, outnorm, 
          waveSp, LDG_bias, bc_bias, gamma, rus_k, nFpts, startFpt, endFpt);
  }
  else if (equation == Burgers)
  {
    if (nDims == 2)
      rusanov_flux<1, 2, Burgers><<<blocks, threads>>>(U, Fconv, Fcomm, P, AdvDiff_A, norm, outnorm, 
          waveSp, LDG_bias, bc_bias, gamma, rus_k, nFpts, startFpt, endFpt);
    else
      rusanov_flux<1, 3, Burgers><<<blocks, threads>>>(U, Fconv, Fcomm, P, AdvDiff_A, norm, outnorm, 
          waveSp, LDG_bias, bc_bias, gamma, rus_k, nFpts, startFpt, endFpt);
  }
  else if (equation == EulerNS)
  {
    if (nDims == 2)
      rusanov_flux<4, 2, EulerNS><<<blocks, threads>>>(U, Fconv, Fcomm, P, AdvDiff_A, norm, outnorm, 
          waveSp, LDG_bias, bc_bias, gamma, rus_k, nFpts, startFpt, endFpt);
    else
      rusanov_flux<5, 3, EulerNS><<<blocks, threads>>>(U, Fconv, Fcomm, P, AdvDiff_A, norm, outnorm, 
          waveSp, LDG_bias, bc_bias, gamma, rus_k, nFpts, startFpt, endFpt);

  }
}

template<unsigned int nVars, unsigned int nDims, unsigned int equation>
__global__
void roe_flux(mdvector_gpu<double> U, mdvector_gpu<double> Fconv, 
    mdvector_gpu<double> Fcomm, mdvector_gpu<double> norm_gfpts, 
    mdvector_gpu<int> outnorm_gfpts, mdvector_gpu<double> waveSp_gfpts, mdvector_gpu<int> bc_bias,
    double gamma, unsigned int nFpts, unsigned int startFpt, unsigned int endFpt)
{
  const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x + startFpt;

  if (fpt >= endFpt)
    return;

  /* Apply central flux at boundaries */
  double k = 0;
  if (bc_bias(fpt))
  {
    k = 1.0;
  }

  double FL[nVars]; double FR[nVars]; 
  double F[nVars]; double dW[nVars];
  double norm[nDims]; double outnorm[2];

  for (unsigned int dim = 0; dim < nDims; dim++)
  {
    norm[dim] = norm_gfpts(fpt, dim, 0);
  }

  outnorm[0] = outnorm_gfpts(fpt, 0);
  outnorm[1] = outnorm_gfpts(fpt, 1);


  /* Initialize FL, FR */
  for (unsigned int n = 0; n < nVars; n++)
  {
    FL[n] = 0.0; FR[n] = 0.0;
  }

  /* Get interface-normal flux components  (from L to R) */
  for (unsigned int dim = 0; dim < nDims; dim++)
  {
    for (unsigned int n = 0; n < nVars; n++)
    {
      FL[n] += Fconv(fpt, n, dim, 0) * norm[dim];
      FR[n] += Fconv(fpt, n, dim, 1) * norm[dim];
    }
  }

  /* Get difference in state variables */
  for (unsigned int n = 0; n < nVars; n++)
  {
    dW[n] = U(fpt, n, 1) - U(fpt, n, 0);
  }

  /* Get numerical wavespeed */
  if (equation == EulerNS)
  {
    /* Primitive Variables */
    double gam = gamma;
    double rhoL = U(fpt, 0, 0);
    double uL = U(fpt, 1, 0) / U(fpt, 0, 0);
    double vL = U(fpt, 2, 0) / U(fpt, 0, 0);
    double pL = (gam-1.0) * (U(fpt, 3, 0) - 0.5 * rhoL * (uL*uL + vL*vL));
    double hL = (U(fpt, 3, 0) + pL) / rhoL;

    double rhoR = U(fpt, 0, 1);
    double uR = U(fpt, 1, 1) / U(fpt, 0, 1);
    double vR = U(fpt, 2, 1) / U(fpt, 0, 1);
    double pR = (gam-1.0) * (U(fpt, 3, 1) - 0.5 * rhoR * (uR*uR + vR*vR));
    double hR = (U(fpt, 3, 0) + pL) / rhoL;

    /* Compute averaged values */
    double sq_rho = std::sqrt(rhoR / rhoL);
    double rrho = 1.0 / (sq_rho + 1.0);
    double um = rrho * (uL + sq_rho * uR);
    double vm = rrho * (vL + sq_rho * vR);
    double hm = rrho * (hL + sq_rho * hR);

    double Vmsq = 0.5 * (um*um + vm*vm);
    double am = std::sqrt((gam-1.0) * (hm - Vmsq));
    double Vnm = um * norm[0] + vm * norm[1];

    /* Compute Wavespeeds */
    double lambda0 = std::abs(Vnm);
    double lambdaP = std::abs(Vnm + am);
    double lambdaM = std::abs(Vnm - am);

    /* Entropy fix */
    double eps = 0.5 * (std::abs(FL[0] / rhoL - FR[0] / rhoR) + std::abs(std::sqrt(gam*pL/rhoL) - std::sqrt(gam*pR/rhoR)));
    if (lambda0 < 2.0 * eps)
      lambda0 = 0.25 * lambda0*lambda0 / eps + eps;
    if (lambdaP < 2.0 * eps)
      lambdaP = 0.25 * lambdaP*lambdaP / eps + eps;
    if (lambdaM < 2.0 * eps)
      lambdaM = 0.25 * lambdaM*lambdaM / eps + eps;

    /* Matrix terms */
    double a2 = 0.5 * (lambdaP + lambdaM) - lambda0;
    double a3 = 0.5 * (lambdaP - lambdaM) / am;
    double a1 = a2 * (gam-1.0) / (am*am);
    double a4 = a3 * (gam-1.0);
    double a5 = Vmsq * dW[0] - um * dW[1] - vm * dW[2] + dW[3];
    double a6 = Vnm * dW[0] - norm[0] * dW[1] - norm[1] * dW[2];
    double aL1 = a1 * a5 - a3 * a6;
    double bL1 = a4 * a5 - a2 * a6;

    F[0] = 0.5 * (FR[0] + FL[0]) - (1.0-k) * (lambda0 * dW[0] + aL1);
    F[1] = 0.5 * (FR[1] + FL[1]) - (1.0-k) * (lambda0 * dW[1] + aL1 * um + bL1 * norm[0]);
    F[2] = 0.5 * (FR[2] + FL[2]) - (1.0-k) * (lambda0 * dW[2] + aL1 * vm + bL1 * norm[1]);
    F[3] = 0.5 * (FR[3] + FL[3]) - (1.0-k) * (lambda0 * dW[3] + aL1 * hm + bL1 * Vnm);

    waveSp_gfpts(fpt) = max(max(lambda0, lambdaP), lambdaM);
  }

  /* Correct for positive parent space sign convention */
  for (unsigned int n = 0; n < nVars; n++)
  {
    Fcomm(fpt, n, 0) = F[n] * outnorm[0];
    Fcomm(fpt, n, 1) = F[n] * -outnorm[1];
  }
}

void roe_flux_wrapper(mdvector_gpu<double> &U, mdvector_gpu<double> &Fconv, 
    mdvector_gpu<double> &Fcomm, mdvector_gpu<double> &norm, mdvector_gpu<int> &outnorm, 
    mdvector_gpu<double> &waveSp, mdvector_gpu<int> &bc_bias, double gamma, unsigned int nFpts, unsigned int nVars, 
    unsigned int nDims, unsigned int equation, unsigned int startFpt, unsigned int endFpt)
{
  unsigned int threads = 256;
  unsigned int blocks = ((endFpt - startFpt + 1) + threads - 1)/threads;

  if (equation == EulerNS)
  {
    if (nDims == 2)
      roe_flux<4, 2, EulerNS><<<blocks, threads>>>(U, Fconv, Fcomm, norm, outnorm, 
          waveSp, bc_bias, gamma, nFpts, startFpt, endFpt);
    else
      ThrowException("Roe flux only implemented for 2D!");
  }
  else
  {
    ThrowException("Roe flux not implemented for this equation type!");
  }
}

template <unsigned int nVars, unsigned int nDims>
__global__
void LDG_flux(mdvector_gpu<double> U, mdvector_gpu<double> Fvisc, 
    mdvector_gpu<double> Fcomm, mdvector_gpu<double> Fcomm_no, mdvector_gpu<double> norm_gfpts, 
    mdvector_gpu<int> outnorm_gfpts, mdvector_gpu<int> LDG_bias, double beta, double tau, 
    unsigned int nFpts, unsigned int startFpt, unsigned int endFpt)
{
  const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x + startFpt;

  if (fpt >= endFpt)
    return;

  double FL[nVars]; double FR[nVars];
  double WL[nVars]; double WR[nVars];
  double Fcomm_temp[nVars][nDims];
  double norm[nDims];
  double outnorm[2];
   
  /* Zero out temporary array */
  for (unsigned int n = 0; n < nVars; n++)
    for (unsigned int dim = 0; dim < nDims; dim++)
      Fcomm_temp[n][dim] = 0.0;

  /* Initialize FL, FR */
  for (unsigned int n = 0; n < nVars; n++)
  {
    FL[n] = 0.0; FR[n] = 0.0;
  }

  for (unsigned int dim = 0; dim < nDims; dim++)
  {
    norm[dim] = norm_gfpts(fpt, dim, 0);
  }

  outnorm[0] = outnorm_gfpts(fpt, 0);
  outnorm[1] = outnorm_gfpts(fpt, 1);

  /* Setting sign of beta (from HiFiLES) */
  if (nDims == 2)
  {
    if (norm[0] + norm[1] < 0.0)
      beta = -beta;
  }
  else if (nDims == 3)
  {
    if (norm[0] + norm[1] + sqrt(2.) * norm[2] < 0.0)
      beta = -beta;
  }


  /* Get interface-normal flux components  (from L to R)*/
  for (unsigned int dim = 0; dim < nDims; dim++)
  {
    for (unsigned int n = 0; n < nVars; n++)
    {
      FL[n] += Fvisc(fpt, n, dim, 0) * norm[dim];
      FR[n] += Fvisc(fpt, n, dim, 1) * norm[dim];
    }
  }

  /* Get left and right state variables */
  for (unsigned int n = 0; n < nVars; n++)
  {
    WL[n] = U(fpt, n, 0); WR[n] = U(fpt, n, 1);
  }

  /* Compute common normal viscous flux and accumulate */
  /* If interior, use central */
  if (LDG_bias(fpt) == 0)
  {
    for (unsigned int dim = 0; dim < nDims; dim++)
    {
      for (unsigned int n = 0; n < nVars; n++)
      {
        Fcomm_temp[n][dim] += 0.5*(Fvisc(fpt, n, dim, 0) + Fvisc(fpt, n, dim, 1)) + 
          tau * norm[dim] * (WL[n] - WR[n]) + beta * norm[dim] * (FL[n] - FR[n]);
      }
    }
  }
  /* If boundary, use right state only */
  else
  {
    for (unsigned int dim = 0; dim < nDims; dim++)
    {
      for (unsigned int n = 0; n < nVars; n++)
      {
        Fcomm_temp[n][dim] += Fvisc(fpt, n, dim, 1) + tau * norm[dim] * (WL[n] - WR[n]);
      }
    }
  }

  for (unsigned int dim = 0; dim < nDims; dim++)
  {
    for (unsigned int n = 0; n < nVars; n++)
    {
      double F = Fcomm_temp[n][dim] * norm[dim];
      Fcomm(fpt, n, 0) += F * outnorm[0];
      Fcomm(fpt, n, 1) += F * -outnorm[1];
    }
  }

}

void LDG_flux_wrapper(mdvector_gpu<double> &U, mdvector_gpu<double> &Fvisc, 
    mdvector_gpu<double> &Fcomm, mdvector_gpu<double> &Fcomm_temp, mdvector_gpu<double> &norm, 
    mdvector_gpu<int> &outnorm, mdvector_gpu<int> &LDG_bias, double beta, double tau, 
    unsigned int nFpts, unsigned int nVars, unsigned int nDims, unsigned int equation,
    unsigned int startFpt, unsigned int endFpt)
{
  unsigned int threads = 256;
  unsigned int blocks = ((endFpt - startFpt + 1) + threads - 1)/threads;

  if (equation == AdvDiff || equation == Burgers)
  {
    if (nDims == 2)
      LDG_flux<1, 2><<<blocks, threads>>>(U, Fvisc, Fcomm, Fcomm_temp, norm, outnorm, LDG_bias, beta, tau, 
          nFpts, startFpt, endFpt);
    else
      LDG_flux<1, 3><<<blocks, threads>>>(U, Fvisc, Fcomm, Fcomm_temp, norm, outnorm, LDG_bias, beta, tau, 
          nFpts, startFpt, endFpt);
  }
  else if (equation == EulerNS)
  {
    if (nDims == 2)
      LDG_flux<4, 2><<<blocks, threads>>>(U, Fvisc, Fcomm, Fcomm_temp, norm, outnorm, LDG_bias, beta, tau, 
          nFpts, startFpt, endFpt);
    else
      LDG_flux<5, 3><<<blocks, threads>>>(U, Fvisc, Fcomm, Fcomm_temp, norm, outnorm, LDG_bias, beta, tau, 
          nFpts, startFpt, endFpt);
  }
}

template <unsigned int nDims>
__global__
void compute_common_U_LDG(mdvector_gpu<double> U, mdvector_gpu<double> Ucomm, 
    mdvector_gpu<double> norm, double beta, unsigned int nFpts, unsigned int nVars,
    mdvector_gpu<int> LDG_bias, unsigned int startFpt, unsigned int endFpt)
{
    const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x + startFpt;
    const unsigned int var = blockDim.y * blockIdx.y + threadIdx.y;

    if (fpt >= endFpt || var >= nVars)
      return;

    /* Setting sign of beta (from HiFiLES) */
    if (nDims == 2)
    {
      if (norm(fpt, 0, 0) + norm(fpt, 1, 0) < 0.0)
        beta = -beta;
    }
    else if (nDims == 3)
    {
      if (norm(fpt, 0,0) + norm(fpt, 1, 0) + sqrt(2.) * norm(fpt, 2, 0) < 0.0)
        beta = -beta;
    }

    double UL = U(fpt, var, 0); double UR = U(fpt, var, 1);

    if (LDG_bias(fpt) == 0)
    {
      double UC = 0.5*(UL + UR) - beta*(UL - UR);
      Ucomm(fpt, var, 0) = UC;
      Ucomm(fpt, var, 1) = UC;
    }
    /* If on boundary, don't use beta (this is from HiFILES. Need to check) */
    else
    {
      Ucomm(fpt, var, 0) = UR;
      Ucomm(fpt, var, 1) = UR;
      //Ucomm(fpt, var, 0) = 0.5*(UL + UR);
      //Ucomm(fpt, var, 1) = 0.5*(UL + UR);
    }
}

void compute_common_U_LDG_wrapper(mdvector_gpu<double> &U, mdvector_gpu<double> &Ucomm, 
    mdvector_gpu<double> &norm, double beta, unsigned int nFpts, unsigned int nVars, 
    unsigned int nDims, mdvector_gpu<int> &LDG_bias, unsigned int startFpt,
    unsigned int endFpt)
{
  dim3 threads(32,4);
  dim3 blocks(((endFpt - startFpt + 1) + threads.x - 1)/threads.x, (nVars + threads.y - 1)/threads.y);

  if (nDims == 2)
    compute_common_U_LDG<2><<<blocks, threads>>>(U, Ucomm, norm, beta, nFpts, nVars,
        LDG_bias, startFpt, endFpt);
  else
    compute_common_U_LDG<3><<<blocks, threads>>>(U, Ucomm, norm, beta, nFpts, nVars,
        LDG_bias, startFpt, endFpt);

}
__global__
void transform_flux_faces(mdvector_gpu<double> Fcomm, mdvector_gpu<double> dA, 
    unsigned int nFpts, unsigned int nVars)
{
    const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned int var = blockDim.y * blockIdx.y + threadIdx.y;

    if (fpt >= nFpts || var >= nVars)
      return;

    Fcomm(fpt, var, 0) *= dA(fpt);
    Fcomm(fpt, var, 1) *= dA(fpt);
}

void transform_flux_faces_wrapper(mdvector_gpu<double> &Fcomm, mdvector_gpu<double> &dA, 
    unsigned int nFpts, unsigned int nVars)
{
  dim3 threads(32,4);
  dim3 blocks((nFpts + threads.x - 1)/threads.x, (nVars + threads.y - 1)/threads.y);

  transform_flux_faces<<<blocks,threads>>>(Fcomm, dA, nFpts, nVars);

}

#ifdef _MPI
__global__
void pack_U(mdvector_gpu<double> U_sbuffs, mdvector_gpu<unsigned int> fpts, 
    mdvector_gpu<double> U, unsigned int nVars, unsigned int nFpts)
{
  const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int var = blockDim.y * blockIdx.y + threadIdx.y;

  if (i >= nFpts || var >= nVars)
    return;

  U_sbuffs(i, var) = U(fpts(i), var, 0);
}

void pack_U_wrapper(mdvector_gpu<double> &U_sbuffs, mdvector_gpu<unsigned int> &fpts, 
    mdvector_gpu<double> &U, unsigned int nVars)
{
  dim3 threads(32,4);
  dim3 blocks((fpts.size() + threads.x - 1)/threads.x, (nVars + threads.y - 1)/threads.y);

  pack_U<<<blocks,threads>>>(U_sbuffs, fpts, U, nVars, fpts.size());
}

__global__
void unpack_U(mdvector_gpu<double> U_rbuffs, mdvector_gpu<unsigned int> fpts, 
    mdvector_gpu<double> U, unsigned int nVars, unsigned int nFpts)
{
  const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int var = blockDim.y * blockIdx.y + threadIdx.y;

  if (i >= nFpts || var >= nVars)
    return;

  U(fpts(i), var, 1) = U_rbuffs(i, var);
}

void unpack_U_wrapper(mdvector_gpu<double> &U_rbuffs, mdvector_gpu<unsigned int> &fpts, 
    mdvector_gpu<double> &U, unsigned int nVars)
{
  dim3 threads(32,4);
  dim3 blocks((fpts.size() + threads.x - 1)/threads.x, (nVars + threads.y - 1)/threads.y);

  unpack_U<<<blocks,threads>>>(U_rbuffs, fpts, U, nVars, fpts.size());
}

template<unsigned int nDims>
__global__
void pack_dU(mdvector_gpu<double> U_sbuffs, mdvector_gpu<unsigned int> fpts, 
    mdvector_gpu<double> dU, unsigned int nVars, unsigned int nFpts)
{
  const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int var = blockDim.y * blockIdx.y + threadIdx.y;

  if (i >= nFpts || var >= nVars)
    return;

  for (unsigned int dim = 0; dim < nDims; dim++)
  {
    U_sbuffs(i, var, dim) = dU(fpts(i), var, dim, 0);
  }
}

void pack_dU_wrapper(mdvector_gpu<double> &U_sbuffs, mdvector_gpu<unsigned int> &fpts, 
    mdvector_gpu<double> &dU, unsigned int nVars, unsigned int nDims)
{
  dim3 threads(32,4);
  dim3 blocks((fpts.size() + threads.x - 1)/threads.x, (nVars + threads.y - 1)/threads.y);

  if (nDims == 2)
    pack_dU<2><<<blocks,threads>>>(U_sbuffs, fpts, dU, nVars, fpts.size());
  else
    pack_dU<3><<<blocks,threads>>>(U_sbuffs, fpts, dU, nVars, fpts.size());
}

template<unsigned int nDims>
__global__
void unpack_dU(mdvector_gpu<double> U_rbuffs, mdvector_gpu<unsigned int> fpts, 
    mdvector_gpu<double> dU, unsigned int nVars, unsigned int nFpts)
{
  const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int var = blockDim.y * blockIdx.y + threadIdx.y;

  if (i >= nFpts || var >= nVars)
    return;

  for (unsigned int dim = 0; dim < nDims; dim++)
  {
    dU(fpts(i), var, dim, 1) = U_rbuffs(i, var, dim);
  }
}

void unpack_dU_wrapper(mdvector_gpu<double> &U_rbuffs, mdvector_gpu<unsigned int> &fpts, 
    mdvector_gpu<double> &dU, unsigned int nVars, unsigned int nDims)
{
  dim3 threads(32,4);
  dim3 blocks((fpts.size() + threads.x - 1)/threads.x, (nVars + threads.y - 1)/threads.y);

  if (nDims == 2)
    unpack_dU<2><<<blocks,threads>>>(U_rbuffs, fpts, dU, nVars, fpts.size());
  else 
    unpack_dU<3><<<blocks,threads>>>(U_rbuffs, fpts, dU, nVars, fpts.size());
}

#endif
