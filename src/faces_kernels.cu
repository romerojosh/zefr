#include "hip/hip_runtime.h"
#include "faces_kernels.h"
#include "mdvector_gpu.h"

__global__
void compute_Fconv_fpts_2D_EulerNS(mdvector_gpu<double> F, mdvector_gpu<double> U, mdvector_gpu<double> P, 
    unsigned int nFpts, double gamma)
{
  const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x;

  if (fpt >= nFpts)
    return;

   for (unsigned int slot = 0; slot < 2; slot ++)
   {
     /* Compute some primitive variables (keep pressure)*/
     double momF = (U(fpt, 1, slot) * U(fpt, 1, slot) + U(fpt, 2, slot) * 
         U(fpt, 2, slot)) / U(fpt, 0, slot);

     P(fpt, slot) = (gamma - 1.0) * (U(fpt, 3, slot) - 0.5 * momF);
     double H = (U(fpt, 3, slot) + P(fpt, slot)) / U(fpt, 0, slot);

     F(fpt, 0, 0, slot) = U(fpt, 1, slot);
     F(fpt, 1, 0, slot) = U(fpt, 1, slot) * U(fpt, 1, slot) / U(fpt, 0, slot) + P(fpt, slot);
     F(fpt, 2, 0, slot) = U(fpt, 1, slot) * U(fpt, 2, slot) / U(fpt, 0, slot);
     F(fpt, 3, 0, slot) = U(fpt, 1, slot) * H;

     F(fpt, 0, 1, slot) = U(fpt, 2, slot);
     F(fpt, 1, 1, slot) = U(fpt, 1, slot) * U(fpt, 2, slot) / U(fpt, 0, slot);
     F(fpt, 2, 1, slot) = U(fpt, 2, slot) * U(fpt, 2, slot) / U(fpt, 0, slot) + P(fpt, slot);
     F(fpt, 3, 1, slot) = U(fpt, 2, slot) * H;
   }
}

void compute_Fconv_fpts_2D_EulerNS_wrapper(mdvector_gpu<double> F_gfpts, mdvector_gpu<double> U_gfpts, mdvector_gpu<double> P_gfpts, 
    unsigned int nFpts, double gamma)
{
  unsigned int threads = 192;
  unsigned int blocks = (nFpts + threads - 1)/threads;

  compute_Fconv_fpts_2D_EulerNS<<<blocks, threads>>>(F_gfpts, U_gfpts, P_gfpts, nFpts, gamma);
}

__global__
void compute_Fvisc_fpts_2D_EulerNS(mdvector_gpu<double> Fvisc, 
    mdvector_gpu<double> U, mdvector_gpu<double> dU, unsigned int nFpts, double gamma, 
        double prandtl, double mu_in, double c_sth, double rt, bool fix_vis)
{
  const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x;

  if (fpt >= nFpts)
    return;

  for (unsigned int slot = 0; slot < 2; slot++)
  {
    /* Setting variables for convenience */
    /* States */
    double rho = U(fpt, 0, slot);
    double momx = U(fpt, 1, slot);
    double momy = U(fpt, 2, slot);
    double e = U(fpt, 3, slot);

    double u = momx / rho;
    double v = momy / rho;
    double e_int = e / rho - 0.5 * (u*u + v*v);

    /* Gradients */
    double rho_dx = dU(fpt, 0, 0, slot);
    double momx_dx = dU(fpt, 1, 0, slot);
    double momy_dx = dU(fpt, 2, 0, slot);
    double e_dx = dU(fpt, 3, 0, slot);
    
    double rho_dy = dU(fpt, 0, 1, slot);
    double momx_dy = dU(fpt, 1, 1, slot);
    double momy_dy = dU(fpt, 2, 1, slot);
    double e_dy = dU(fpt, 3, 1, slot);

    /* Set viscosity */
    double mu;
    if (fix_vis)
    {
      mu = mu_in;
    }
    /* If desired, use Sutherland's law */
    else
    {
      double rt_ratio = (gamma - 1.0) * e_int / (rt);
      mu = mu_in * std::pow(rt_ratio,1.5) * (1. + c_sth) / (rt_ratio + c_sth);
    }

    double du_dx = (momx_dx - rho_dx * u) / rho;
    double du_dy = (momx_dy - rho_dy * u) / rho;

    double dv_dx = (momy_dx - rho_dx * v) / rho;
    double dv_dy = (momy_dy - rho_dy * v) / rho;

    double dke_dx = 0.5 * (u*u + v*v) * rho_dx + rho * (u * du_dx + v * dv_dx);
    double dke_dy = 0.5 * (u*u + v*v) * rho_dy + rho * (u * du_dy + v * dv_dy);

    double de_dx = (e_dx - dke_dx - rho_dx * e_int) / rho;
    double de_dy = (e_dy - dke_dy - rho_dy * e_int) / rho;

    double diag = (du_dx + dv_dy) / 3.0;

    double tauxx = 2.0 * mu * (du_dx - diag);
    double tauxy = mu * (du_dy + dv_dx);
    double tauyy = 2.0 * mu * (dv_dy - diag);

    /* Set viscous flux values */
    Fvisc(fpt, 0, 0, slot) = 0.0;
    Fvisc(fpt, 1, 0, slot) = -tauxx;
    Fvisc(fpt, 2, 0, slot) = -tauxy;
    Fvisc(fpt, 3, 0, slot) = -(u * tauxx + v * tauxy + (mu / prandtl) *
        gamma * de_dx);

    Fvisc(fpt, 0, 1, slot) = 0.0;
    Fvisc(fpt, 1, 1, slot) = -tauxy;
    Fvisc(fpt, 2, 1, slot) = -tauyy;
    Fvisc(fpt, 3, 1, slot) = -(u * tauxy + v * tauyy + (mu / prandtl) *
        gamma * de_dy);
  }

}

void compute_Fvisc_fpts_2D_EulerNS_wrapper(mdvector_gpu<double> Fvisc, 
    mdvector_gpu<double> U, mdvector_gpu<double> dU, unsigned int nFpts, double gamma, 
        double prandtl, double mu_in, double c_sth, double rt, bool fix_vis)
{
  unsigned int threads = 192;
  unsigned int blocks = (nFpts + threads - 1)/threads;

  compute_Fvisc_fpts_2D_EulerNS<<<threads, blocks>>>(Fvisc, U, dU, nFpts, gamma, 
      prandtl, mu_in, c_sth, rt, fix_vis);
}
__global__
void apply_bcs(mdvector_gpu<double> U, unsigned int nFpts, unsigned int nGfpts_int, 
    unsigned int nVars, unsigned int nDims, double rho_fs, mdvector_gpu<double> V_fs, 
    double P_fs, double gamma, double R_ref, double T_tot_fs, double P_tot_fs, double T_wall, 
    mdvector_gpu<double> V_wall, mdvector_gpu<double> norm_fs, mdvector_gpu<double> norm, 
    mdvector_gpu<unsigned int> gfpt2bnd, mdvector_gpu<unsigned int> per_fpt_list,
    mdvector_gpu<int> LDG_bias)
{
  const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x + nGfpts_int;

  if (fpt >= nFpts)
    return;

  unsigned int bnd_id = gfpt2bnd(fpt - nGfpts_int);

  /* Apply specified boundary condition */
  switch(bnd_id)
  {
    case 1:/* Periodic */
    {
      unsigned int per_fpt = per_fpt_list(fpt - nGfpts_int);

      for (unsigned int n = 0; n < nVars; n++)
      {
        U(fpt, n, 1) = U(per_fpt, n, 0);
      }
      break;
    }
  
    case 2: /* Farfield and Supersonic Inlet */
    {
      /* Set boundaries to freestream values */
      U(fpt, 0, 1) = rho_fs;

      double Vsq = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        U(fpt, dim+1, 1) = rho_fs * V_fs(dim);
        Vsq += V_fs(dim) * V_fs(dim);
      }

      U(fpt, 3, 1) = P_fs/(gamma-1.0) + 0.5*rho_fs * Vsq; 

      /* Set LDG bias */
      LDG_bias(fpt) = 1;

      break;
    }

    case 3: /* Supersonic Outlet */
    {
      /* Extrapolate boundary values from interior */
      for (unsigned int n = 0; n < nVars; n++)
        U(fpt, n, 1) = U(fpt, n, 0);

      /* Set LDG bias */
      LDG_bias(fpt) = 1;

      break;
    }

    case 4: /* Subsonic Inlet */
    {
      double VL[3]; double VR[3];
      /*
      if (!input->viscous)
        ThrowException("Subsonic inlet only for viscous flows currently!");
      */

      /* Get states for convenience */
      double rhoL = U(fpt, 0, 0);

      double Vsq = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        VL[dim] = U(fpt, dim+1, 0) / rhoL;
        Vsq += VL[dim] * VL[dim];
      }

      double eL = U(fpt, 3 ,0);
      double PL = (gamma - 1.0) * (eL - 0.5 * rhoL * Vsq);


      /* Compute left normal velocity and dot product of normal*/
      double VnL = 0.0;
      double alpha = 0.0;

      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        VnL += VL[dim] * norm(fpt, dim, 0);
        alpha += norm_fs(dim) * norm(fpt, dim, 0);
      }

      /* Compute speed of sound */
      double cL = std::sqrt(gamma * PL / rhoL);

      /* Extrapolate Riemann invariant */
      double R_plus  = VnL + 2.0 * cL / (gamma - 1.0);

      /* Specify total enthalpy */
      double H_tot = gamma * R_ref / (gamma - 1.0) * T_tot_fs;

      /* Compute total speed of sound squared */
      double c_tot_sq = (gamma - 1.0) * (H_tot - (eL + PL) / rhoL + 0.5 * Vsq) + cL * cL;

      /* Coefficients of Quadratic equation */
      double aa = 1.0 + 0.5 * (gamma - 1.0) * alpha * alpha;
      double bb = -(gamma - 1.0) * alpha * R_plus;
      double cc = 0.5 * (gamma - 1.0) * R_plus * R_plus - 2.0 * c_tot_sq / (gamma - 1.0);

      /* Solve quadratic for right velocity */
      double dd = bb * bb  - 4.0 * aa * cc;
      dd = std::sqrt(max(dd, 0.0));  // Max to keep from producing NaN
      double VR_mag = (dd - bb) / (2.0 * aa);
      VR_mag = max(VR_mag, 0.0);
      double VR_mag_sq = VR_mag * VR_mag;

      /* Compute right speed of sound and Mach */
      /* Note: Need to verify what is going on here. */
      double cR_sq = c_tot_sq - 0.5 * (gamma - 1.0) * VR_mag_sq;
      double Mach_sq = VR_mag_sq / cR_sq;
      Mach_sq = min(Mach_sq, 1.0); // Clamp to Mach = 1
      VR_mag_sq = Mach_sq * cR_sq;
      VR_mag = std::sqrt(VR_mag_sq);
      cR_sq = c_tot_sq - 0.5 * (gamma - 1.0) * VR_mag_sq;

      /* Compute right states */

      double TR = cR_sq / (gamma * R_ref);
      double PR = P_tot_fs * std::pow(TR / T_tot_fs, gamma/ (gamma - 1.0));

      U(fpt, 0, 1) = PR / (R_ref * TR);

      Vsq = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        VR[dim] = VR_mag * norm_fs(dim);
        U(fpt, dim+1, 1) = U(fpt, 0, 1) * VR[dim];
        Vsq += VR[dim] * VR[dim];
      }

      U(fpt, 3, 1) = PR / (gamma - 1.0) + 0.5 * U(fpt, 0, 1) * Vsq;

      /* Set LDG bias */
      LDG_bias(fpt) = 1;

      break;
    }

    case 5: /* Subsonic Outlet */
    {
      /*
      if (!input->viscous)
        ThrowException("Subsonic outlet only for viscous flows currently!");
      */

      double VL[3]; double VR[3];

      /* Get states for convenience */
      double rhoL = U(fpt, 0, 0);

      double Vsq = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        VL[dim] = U(fpt, dim+1, 0) / rhoL;
        Vsq += VL[dim] * VL[dim];
      }

      double eL = U(fpt, 3 ,0);
      double PL = (gamma - 1.0) * (eL - 0.5 * rhoL * Vsq);

      /* Compute left normal velocity */
      double VnL = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        VnL += VL[dim] * norm(fpt, dim, 0);
      }

      /* Compute speed of sound */
      double cL = std::sqrt(gamma * PL / rhoL);

      /* Extrapolate Riemann invariant */
      double R_plus  = VnL + 2.0 * cL / (gamma - 1.0);

      /* Extrapolate entropy */
      double s = PL / std::pow(rhoL, gamma);

      /* Fix pressure */
      double PR = P_fs;

      U(fpt, 0, 1) = std::pow(PR / s, 1.0 / gamma);

      /* Compute right speed of sound and velocity magnitude */
      double cR = std::sqrt(gamma * PR/ U(fpt, 0, 1));

      double VnR = R_plus - 2.0 * cR / (gamma - 1.0);

      Vsq = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        VR[dim] = VL[dim] + (VnR - VnL) * norm(fpt, dim, 0);
        U(fpt, dim+1, 1) = U(fpt, 0, 1) * VR[dim];
        Vsq += VR[dim] * VR[dim];
      }

      U(fpt, 3, 1) = PR / (gamma - 1.0) + 0.5 * U(fpt, 0, 1) * Vsq;

      /* Set LDG bias */
      LDG_bias(fpt) = 1;

      break;
    }

    case 6: /* Characteristic (from HiFiLES) */
    {
      /* Compute wall normal velocities */
      double VnL = 0.0; double VnR = 0.0;

      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        VnL += U(fpt, dim+1, 0) / U(fpt, 0, 0) * norm(fpt, dim, 0);
        VnR += V_fs(dim) * norm(fpt, dim, 0);
      }
    

      /* Compute pressure. TODO: Compute pressure once!*/
      double momF = (U(fpt, 1, 0) * U(fpt, 1, 0) + U(fpt, 2, 0) * 
          U(fpt, 2, 0)) / U(fpt, 0, 0);

      double PL = (gamma - 1.0) * (U(fpt, 3, 0) - 0.5 * momF);
      double PR = P_fs;

      /* Compute Riemann Invariants */
      double Rp = VnL + 2.0 / (gamma - 1) * std::sqrt(gamma * PL / 
          U(fpt, 0,0));
      double Rn = VnR - 2.0 / (gamma - 1) * std::sqrt(gamma * PR / 
          rho_fs);

      double cstar = 0.25 * (gamma - 1) * (Rp - Rn);
      double ustarn = 0.5 * (Rp + Rn);

      if (VnL < 0.0) /* Case 1: Inflow */
      {
        double s_inv = std::pow(rho_fs, gamma) / PR;

        double Vsq = 0.0;
        for (unsigned int dim = 0; dim < nDims; dim++)
          Vsq += V_fs(dim) * V_fs(dim);

        double H_fs = gamma / (gamma - 1.0) * PR / rho_fs +
            0.5 * Vsq;

        double rhoR = std::pow(1.0 / gamma * (s_inv * cstar * cstar), 1.0/ 
            (gamma - 1.0));

        U(fpt, 0, 1) = rhoR;
        for (unsigned int dim = 0; dim < nDims; dim++)
          U(fpt, dim+1, 1) = rhoR * (ustarn * norm(fpt, dim, 0) + V_fs(dim) - VnR * 
            norm(fpt, dim, 0));

        PR = rhoR / gamma * cstar * cstar;
        U(fpt, 3, 1) = rhoR * H_fs - PR;
        
      }
      else  /* Case 2: Outflow */
      {
        double rhoL = U(fpt, 0, 0);
        double s_inv = std::pow(rhoL, gamma) / PL;

        double rhoR = std::pow(1.0 / gamma * (s_inv * cstar * cstar), 1.0/ 
            (gamma - 1.0));

        U(fpt, 0, 1) = rhoR;
        U(fpt, 1, 1) = rhoR * (ustarn * norm(fpt, 0, 0) +(U(fpt, 1, 0) / 
              U(fpt, 0, 0) - VnL * norm(fpt, 0, 0)));
        U(fpt, 2, 1) = rhoR * (ustarn * norm(fpt, 1, 0) +(U(fpt, 2, 0) / 
              U(fpt, 0, 0) - VnL * norm(fpt, 1, 0)));
        double PR = rhoR / gamma * cstar * cstar;

        double Vsq = 0.0;
        for (unsigned int dim = 0; dim < nDims; dim++)
          Vsq += U(fpt, dim+1, 1) * U(fpt, dim+1, 1) / (rhoL * rhoL) ;
        
        U(fpt, 3, 1) = PR / (gamma - 1.0) + 0.5 * rhoR * Vsq; 
      }

      /* Set LDG bias */
      LDG_bias(fpt) = 1;

      break;

    }
    case 7: /* Slip Wall */
    {
      double momN = 0.0;

      /* Compute wall normal momentum */
      for (unsigned int dim = 0; dim < nDims; dim++)
        momN += U(fpt, dim+1, 0) * norm(fpt, dim, 0);

      U(fpt, 0, 1) = U(fpt, 0, 0);

      /* Set boundary state to cancel normal velocity */
      for (unsigned int dim = 0; dim < nDims; dim++)
        U(fpt, dim+1, 1) = U(fpt, dim+1, 0) - momN * norm(fpt, dim, 0);

      U(fpt, 3, 1) = U(fpt, 3, 0) - 0.5 * (momN * momN) / U(fpt, 0, 0);

      /* Set LDG bias */
      LDG_bias(fpt) = 1;

      break;
    }

    case 8: /* No-slip Wall (isothermal) */
    {
      /*
      if (!input->viscous)
        ThrowException("No slip wall boundary only for viscous flows!");
      */

      double momF = (U(fpt, 1, 0) * U(fpt, 1, 0) + U(fpt, 2, 0) * 
          U(fpt, 2, 0)) / U(fpt, 0, 0);

      double PL = (gamma - 1.0) * (U(fpt, 3, 0) - 0.5 * momF);

      double PR = PL;
      double TR = T_wall;
      
      U(fpt, 0, 1) = PR / (R_ref * TR);

      /* Set velocity to zero */
      for (unsigned int dim = 0; dim < nDims; dim++)
        U(fpt, dim+1, 1) = 0.0;

      U(fpt, 3, 1) = PR / (gamma - 1.0);

      /* Set LDG bias */
      LDG_bias(fpt) = 1;

      break;
    }

    case 9: /* No-slip Wall (isothermal and moving) */
    {
      /*
      if (!input->viscous)
        ThrowException("No slip wall boundary only for viscous flows!");
      */

      double momF = (U(fpt, 1, 0) * U(fpt, 1, 0) + U(fpt, 2, 0) * 
          U(fpt, 2, 0)) / U(fpt, 0, 0);

      double PL = (gamma - 1.0) * (U(fpt, 3, 0) - 0.5 * momF);

      double PR = PL;
      double TR = T_wall;
      
      U(fpt, 0, 1) = PR / (R_ref * TR);

      /* Set velocity to wall velocity */
      double V_wall_sq = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        U(fpt, dim+1, 1) = U(fpt, 0 , 1) * V_wall(dim);
        V_wall_sq += V_wall(dim) * V_wall(dim);
      }

      U(fpt, 3, 1) = PR / (gamma - 1.0) + 0.5 * U(fpt, 0 , 1) * V_wall_sq;

      /* Set LDG bias */
      LDG_bias(fpt) = 1;

      break;
    }

    case 10: /* No-slip Wall (adiabatic) */
    {
      /*
      if (!input->viscous)
        ThrowException("No slip wall boundary only for viscous flows!");
      */

      /* Extrapolate density */
      U(fpt, 0, 1) = U(fpt, 0, 0);

      /* Extrapolate pressure */
      double momF = (U(fpt, 1, 0) * U(fpt, 1, 0) + U(fpt, 2, 0) * 
          U(fpt, 2, 0)) / U(fpt, 0, 0);

      double PL = (gamma - 1.0) * (U(fpt, 3, 0) - 0.5 * momF);
      double PR = PL; 

      /* Set velocity to zero */
      for (unsigned int dim = 0; dim < nDims; dim++)
        U(fpt, dim+1, 1) = 0.0;

      U(fpt, 3, 1) = PR / (gamma - 1.0);

      break;
    }

    case 11: /* No-slip Wall (adiabatic and moving) */
    {
      /*
      if (!input->viscous)
        ThrowException("No slip wall boundary only for viscous flows!");
      */

      /* Extrapolate density */
      U(fpt, 0, 1) = U(fpt, 0, 0);

      /* Extrapolate pressure */
      double momF = (U(fpt, 1, 0) * U(fpt, 1, 0) + U(fpt, 2, 0) * 
          U(fpt, 2, 0)) / U(fpt, 0, 0);

      double PL = (gamma - 1.0) * (U(fpt, 3, 0) - 0.5 * momF);
      double PR = PL; 

      /* Set velocity to wall velocity */
      double V_wall_sq = 0.0;
      for (unsigned int dim = 0; dim < nDims; dim++)
      {
        U(fpt, dim+1, 1) = U(fpt, 0 , 1) * V_wall(dim);
        V_wall_sq += V_wall(dim) * V_wall(dim);
      }

      U(fpt, 3, 1) = PR / (gamma - 1.0) + 0.5 * U(fpt, 0, 1) * V_wall_sq;

      break;
    }
  }

}

void apply_bcs_wrapper(mdvector_gpu<double> U, unsigned int nFpts, unsigned int nGfpts_int, 
    unsigned int nVars, unsigned int nDims, double rho_fs, mdvector_gpu<double> V_fs, 
    double P_fs, double gamma, double R_ref, double T_tot_fs, double P_tot_fs, double T_wall, 
    mdvector_gpu<double> V_wall, mdvector_gpu<double> norm_fs, mdvector_gpu<double> norm, 
    mdvector_gpu<unsigned int> gfpt2bnd, mdvector_gpu<unsigned int> per_fpt_list,
    mdvector_gpu<int> LDG_bias)
{
  unsigned int threads = 192;
  unsigned int blocks = ((nFpts - nGfpts_int) + threads - 1)/threads;

  apply_bcs<<<threads, blocks>>>(U, nFpts, nGfpts_int, nVars, nDims, rho_fs, V_fs, P_fs, gamma, R_ref, 
      T_tot_fs, P_tot_fs, T_wall, V_wall, norm_fs, norm, gfpt2bnd, per_fpt_list, LDG_bias); 
}

__global__
void apply_bcs_dU(mdvector_gpu<double> dU, mdvector_gpu<double> U, unsigned int nFpts, 
    unsigned int nGfpts_int, unsigned int nVars, unsigned int nDims,
    mdvector_gpu<unsigned int> gfpt2bnd, mdvector_gpu<unsigned int> per_fpt_list)
{
  const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x + nGfpts_int;

  if (fpt >= nFpts)
    return;

  unsigned int bnd_id = gfpt2bnd(fpt - nGfpts_int);

  /* Apply specified boundary condition */
  if (bnd_id == 1) /* Periodic */
  {
    for (unsigned int dim = 0; dim < nDims; dim++)
    {
      for (unsigned int n = 0; n < nVars; n++)
      {
          //unsigned int per_fpt = per_fpt_pairs[fpt];
          unsigned int per_fpt = per_fpt_list(fpt);
          dU(fpt, n, dim, 1) = dU(per_fpt, n, dim, 0);
      }
    }
  }
  else if(bnd_id == 10 || bnd_id == 11) /* Adibatic Wall */
  {
    /* Extrapolate gradients except for energy */
    for (unsigned int dim = 0; dim < nDims; dim++)
    {
      for (unsigned int n = 0; n < nVars - 1; n++)
      {
          dU(fpt, n, dim, 1) = dU(fpt, n, dim, 0);
      }
    }

    /* Compute energy gradient */
    /* Get right states and velocity gradients*/
    double rho = U(fpt, 0, 1);
    double momx = U(fpt, 1, 1);
    double momy = U(fpt, 2, 1);
    double e = U(fpt, 3, 1);

    double u = momx / rho;
    double v = momy / rho;
    double e_int = e / rho - 0.5 * (u*u + v*v);

    double rho_dx = dU(fpt, 0, 0, 1);
    double momx_dx = dU(fpt, 1, 0, 1);
    double momy_dx = dU(fpt, 2, 0, 1);
    
    double rho_dy = dU(fpt, 0, 1, 1);
    double momx_dy = dU(fpt, 1, 1, 1);
    double momy_dy = dU(fpt, 2, 1, 1);

    double du_dx = (momx_dx - rho_dx * u) / rho;
    double du_dy = (momx_dy - rho_dy * u) / rho;

    double dv_dx = (momy_dx - rho_dx * v) / rho;
    double dv_dy = (momy_dy - rho_dy * v) / rho;

    double dke_dx = 0.5 * (u*u + v*v) * rho_dx + rho * (u * du_dx + v * dv_dx);
    double dke_dy = 0.5 * (u*u + v*v) * rho_dy + rho * (u * du_dy + v * dv_dy);

    dU(fpt, 3, 0, 1) = (dke_dx + rho_dx * e_int);
    dU(fpt, 3, 1, 1) = (dke_dy + rho_dy * e_int);

    }

}


void apply_bcs_dU_wrapper(mdvector_gpu<double> dU, mdvector_gpu<double> U, unsigned int nFpts, 
    unsigned int nGfpts_int, unsigned int nVars, unsigned int nDims,
    mdvector_gpu<unsigned int> gfpt2bnd, mdvector_gpu<unsigned int> per_fpt_list)
{
  unsigned int threads = 192;
  unsigned int blocks = ((nFpts - nGfpts_int) + threads - 1)/threads;

  apply_bcs_dU<<<threads, blocks>>>(dU, U, nFpts, nGfpts_int, nVars, nDims, 
      gfpt2bnd, per_fpt_list);
}

__global__
void rusanov_flux(mdvector_gpu<double> U, mdvector_gpu<double> Fconv, 
    mdvector_gpu<double> Fcomm, mdvector_gpu<double> P, mdvector_gpu<double> norm,
    mdvector_gpu<int> outnorm, mdvector_gpu<double> waveSp, double gamma, double rus_k,
    unsigned int nFpts, unsigned int nVars, unsigned int nDims)
{
  const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x;

  if (fpt >= nFpts)
    return;

  /* Currently hardcoded for Euler. Need to pass nVars as template arg. */
  double FL[4]; double FR[4];
  double WL[4]; double WR[4];

  /* Initialize FL, FR */
  for (unsigned int i = 0; i < 4; i++)
  {
    FL[i] = 0.0; FR[i] = 0.0;
  }

  /* Get interface-normal flux components  (from L to R)*/
  for (unsigned int dim = 0; dim < nDims; dim++)
  {
    for (unsigned int n = 0; n < nVars; n++)
    {
      FL[n] += Fconv(fpt, n, dim, 0) * norm(fpt, dim, 0);
      FR[n] += Fconv(fpt, n, dim, 1) * norm(fpt, dim, 0);
    }
  }

  /* Get left and right state variables */
  for (unsigned int n = 0; n < nVars; n++)
  {
    WL[n] = U(fpt, n, 0); WR[n] = U(fpt, n, 1);
  }

  /* Get numerical wavespeed */
  /*
  if (input->equation == "AdvDiff")
  {
    waveSp[fpt] = 0.0;

    for (unsigned int dim = 0; dim < nDims; dim++)
      waveSp[fpt] += input->AdvDiff_A(dim) * norm(fpt, dim, 0);
  }
  else if (input->equation == "EulerNS")
  {
  */
  /* Compute speed of sound */
  double aL = std::sqrt(std::abs(gamma * P(fpt, 0) / WL[0]));
  double aR = std::sqrt(std::abs(gamma * P(fpt, 1) / WR[0]));

  /* Compute normal velocities */
  double VnL = 0.0; double VnR = 0.0;
  for (unsigned int dim = 0; dim < nDims; dim++)
  {
    VnL += WL[dim+1]/WL[0] * norm(fpt, dim, 0);
    VnR += WR[dim+1]/WR[0] * norm(fpt, dim, 0);
  }

  waveSp(fpt) = max(std::abs(VnL) + aL, std::abs(VnR) + aR);
  //}

  /* Compute common normal flux */
  for (unsigned int n = 0; n < nVars; n++)
  {
    Fcomm(fpt, n, 0) = 0.5 * (FR[n]+FL[n]) - 0.5 * std::abs(waveSp(fpt))*(1.0-rus_k) * (WR[n]-WL[n]);
    Fcomm(fpt, n, 1) = 0.5 * (FR[n]+FL[n]) - 0.5 * std::abs(waveSp(fpt))*(1.0-rus_k) * (WR[n]-WL[n]);

    /* Correct for positive parent space sign convention */
    Fcomm(fpt, n, 0) *= outnorm(fpt, 0);
    Fcomm(fpt, n, 1) *= -outnorm(fpt, 1);
  }

}

void rusanov_flux_wrapper(mdvector_gpu<double> U, mdvector_gpu<double> Fconv, 
    mdvector_gpu<double> Fcomm, mdvector_gpu<double> P, mdvector_gpu<double> norm,
    mdvector_gpu<int> outnorm, mdvector_gpu<double> waveSp, double gamma, double rus_k,
    unsigned int nFpts, unsigned int nVars, unsigned int nDims)
{
  unsigned int threads = 192;
  unsigned int blocks = (nFpts + threads - 1)/threads;

  rusanov_flux<<<threads, blocks>>>(U, Fconv, Fcomm, P, norm, outnorm, waveSp, gamma, rus_k, 
      nFpts, nVars, nDims);
}

__global__
void LDG_flux(mdvector_gpu<double> U, mdvector_gpu<double> Fvisc, 
    mdvector_gpu<double> Fcomm, mdvector_gpu<double> Fcomm_temp, mdvector_gpu<double> norm, 
    mdvector_gpu<int> outnorm, mdvector_gpu<int> LDG_bias, double beta, double tau, 
    unsigned int nFpts, unsigned int nVars, unsigned int nDims)
{
  const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x;

  if (fpt >= nFpts)
    return;

  /* Hardcoded for EulerNS. Will need to add nVar template */
  double FL[4]; double FR[4];
  double WL[4]; double WR[4];
   
  /* Zero out temporary array */
  for (unsigned int n = 0; n < nVars; n++)
    for (unsigned int dim = 0; dim < nDims; dim++)
      Fcomm_temp(fpt, n, dim) = 0.0;

  /* Initialize FL, FR */
  for (unsigned int n = 0; n < 4; n++)
  {
    FL[n] = 0.0; FR[n] = 0.0;
  }


  /* Setting sign of beta (from HiFiLES) */
  if (norm(fpt, 0, 0) + norm(fpt, 1, 0) < 0.0)
    beta = -beta;


  /* Get interface-normal flux components  (from L to R)*/
  for (unsigned int dim = 0; dim < nDims; dim++)
  {
    for (unsigned int n = 0; n < nVars; n++)
    {
      FL[n] += Fvisc(fpt, n, dim, 0) * norm(fpt, dim, 0);
      FR[n] += Fvisc(fpt, n, dim, 1) * norm(fpt, dim, 0);
    }
  }

  /* Get left and right state variables */
  for (unsigned int n = 0; n < nVars; n++)
  {
    WL[n] = U(fpt, n, 0); WR[n] = U(fpt, n, 1);
  }

  /* Compute common normal viscous flux and accumulate */
  if (!LDG_bias(fpt))
  {
    for (unsigned int n = 0; n < nVars; n++)
    {
      Fcomm_temp(fpt, n, 0) += 0.5*(Fvisc(fpt, n, 0, 0) + Fvisc(fpt, n, 0, 1)) + tau * norm(fpt, 0, 0)* (WL[n]
          - WR[n]) + beta * norm(fpt, 0, 0)* (FL[n] - FR[n]);
      Fcomm_temp(fpt, n, 1) += 0.5*(Fvisc(fpt, n, 1, 0) + Fvisc(fpt, n, 1, 1)) + tau * norm(fpt, 1, 0)* (WL[n]
          - WR[n]) + beta * norm(fpt, 1, 0)* (FL[n] - FR[n]);
    }
  }
  /* Else, only use left flux state */
  else
  {
    for (unsigned int n = 0; n < nVars; n++)
    {
      Fcomm_temp(fpt, n, 0) += Fvisc(fpt, n, 0, 0) + tau * norm(fpt, 0, 0)* (WL[n] - WR[n]);
      Fcomm_temp(fpt, n, 1) += Fvisc(fpt, n, 1, 0) + tau * norm(fpt, 1, 0)* (WL[n] - WR[n]);
    }
  }

  for (unsigned int dim = 0; dim < nDims; dim++)
  {
    for (unsigned int n = 0; n < nVars; n++)
    {
      Fcomm(fpt, n, 0) += (Fcomm_temp(fpt, n, dim) * norm(fpt, dim, 0)) * outnorm(fpt, 0);
      Fcomm(fpt, n, 1) += (Fcomm_temp(fpt, n, dim) * norm(fpt, dim, 0)) * -outnorm(fpt, 1);
    }
  }

}

void LDG_flux_wrapper(mdvector_gpu<double> U, mdvector_gpu<double> Fvisc, 
    mdvector_gpu<double> Fcomm, mdvector_gpu<double> Fcomm_temp, mdvector_gpu<double> norm, 
    mdvector_gpu<int> outnorm, mdvector_gpu<int> LDG_bias, double beta, double tau, 
    unsigned int nFpts, unsigned int nVars, unsigned int nDims)
{
  unsigned int threads = 192;
  unsigned int blocks = (nFpts + threads - 1)/threads;

  LDG_flux<<<threads,blocks>>>(U, Fvisc, Fcomm, Fcomm_temp, norm, outnorm, LDG_bias, beta, tau, 
      nFpts, nVars, nDims);
}
__global__
void compute_common_U_LDG(mdvector_gpu<double> U, mdvector_gpu<double> Ucomm, 
    mdvector_gpu<double> norm, double beta, unsigned int nFpts, unsigned int nVars)
{
    const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned int var = blockDim.y * blockIdx.y + threadIdx.y;

    if (fpt >= nFpts || var >= nVars)
      return;

    /* Setting sign of beta (from HiFiLES) */
    if (norm(fpt, 0, 0) + norm(fpt, 1, 0) < 0.0)
      beta = -beta;

    double UL = U(fpt, var, 0); double UR = U(fpt, var, 1);

    Ucomm(fpt, var, 0) = 0.5*(UL + UR) - beta*(UL - UR);
    Ucomm(fpt, var, 1) = 0.5*(UL + UR) - beta*(UL - UR);

}

void compute_common_U_LDG_wrapper(mdvector_gpu<double> U, mdvector_gpu<double> Ucomm, 
    mdvector_gpu<double> norm, double beta, unsigned int nFpts, unsigned int nVars)
{
  dim3 threads(32,4);
  dim3 blocks((nFpts + threads.x - 1)/threads.x, (nVars + threads.y - 1)/threads.y);

  compute_common_U_LDG<<<threads, blocks>>>(U, Ucomm, norm, beta, nFpts, nVars);

}
__global__
void transform_flux_faces(mdvector_gpu<double> Fcomm, mdvector_gpu<double> dA, 
    unsigned int nFpts, unsigned int nVars)
{
    const unsigned int fpt = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned int var = blockDim.y * blockIdx.y + threadIdx.y;

    if (fpt >= nFpts || var >= nVars)
      return;

    Fcomm(fpt, var, 0) *= dA(fpt);
    Fcomm(fpt, var, 1) *= dA(fpt);
}

void transform_flux_faces_wrapper(mdvector_gpu<double> Fcomm, mdvector_gpu<double> dA, 
    unsigned int nFpts, unsigned int nVars)
{
  dim3 threads(32,4);
  dim3 blocks((nFpts + threads.x - 1)/threads.x, (nVars + threads.y - 1)/threads.y);

  transform_flux_faces<<<threads, blocks>>>(Fcomm, dA, nFpts, nVars);

}
