#include "hip/hip_runtime.h"
/* NOTE: This file is directly included into solver_kernels.cu. */

#include "input.hpp"

__device__
double compute_source_term_dev(double x, double y, double z, double t, unsigned int var, unsigned int nDims, unsigned int equation)
{
  double val = 0.;
  if (equation == AdvDiff)
  {
    if (nDims == 2)
    {
      val =  -M_PI * (std::cos(M_PI * x) + M_PI * std::sin(M_PI * x) + 
             std::cos(M_PI * y) + M_PI * std::sin(M_PI * y));
    }
    else
    {
      val =  -M_PI * (std::cos(M_PI * x) + M_PI * std::sin(M_PI * x) + 
             std::cos(M_PI * y) + M_PI * std::sin(M_PI * y) + 
             std::cos(M_PI * z) + M_PI * std::sin(M_PI * z));
    }
  }
  else
  {
    // NOT DEFINED. Cannot throw exception.
  }

  return val;
}

__device__
double get_cfl_limit_dev(int order)
{
 switch(order)
  {
    case 0:
      return 1.392;

    case 1:
      return 0.4642; 

    case 2:
      return 0.2351;

    case 3:
      return 0.1453;

    case 4:
      return 0.1000;

    case 5:
      return 0.0736;

    case 6:
      return 0.048639193282486;
    
    case 7:
      return 0.034554530245757;
    
    case 8:
      return 0.023910375650672;
    
    case 9:
      return 0.015583791814472;
      
    case 10:
      return 0.008892412680298;

    default:
      return 0.0;
  }

}


