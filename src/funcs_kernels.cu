#include "hip/hip_runtime.h"
/* NOTE: This file is directly included into solver_kernels.cu. */

#include "input.hpp"

__device__
double compute_source_term_dev(double x, double y, double z, double t, unsigned int var, unsigned int nDims, unsigned int equation)
{
  double val = 0.;
  if (equation == AdvDiff || equation == Burgers)
  {
    if (nDims == 2)
    {
      val =  -M_PI * (std::cos(M_PI * x) + M_PI * std::sin(M_PI * x) + 
             std::cos(M_PI * y) + M_PI * std::sin(M_PI * y));
    }
    else
    {
      val =  -M_PI * (std::cos(M_PI * x) + M_PI * std::sin(M_PI * x) + 
             std::cos(M_PI * y) + M_PI * std::sin(M_PI * y) + 
             std::cos(M_PI * z) + M_PI * std::sin(M_PI * z));
    }
  }
  else
  {
    // NOT DEFINED. Cannot throw exception.
  }

  return val;
}

__device__
double get_cfl_limit_adv_dev(int order)
{
  /* Upwinded */
  switch(order)
  {
    case 0:
      return 1.392;

    case 1:
      return 0.4642; 

    case 2:
      return 0.2351;

    case 3:
      return 0.1453;

    case 4:
      return 0.1000;

    case 5:
      return 0.0736;

    default:
      return 0.0;
  }
}

__device__
double get_cfl_limit_diff_dev(double beta, int order)
{
  /* Centered */
  if (beta == 0)
  {
    switch(order)
    {
      case 0:
        return 2.785;

      case 1:
        return 0.17408; 

      case 2:
        return 0.04264;

      case 3:
        return 0.015800;

      case 4:
        return 0.007193;

      case 5:
        return 0.003730;

      default:
        return 0.0;
    }
  }

  /* Upwinded */
  else
  {
    switch(order)
    {
      case 0:
        return 0.6963;

      case 1:
        return 0.07736; 

      case 2:
        return 0.018786;

      case 3:
        return 0.006345;

      case 4:
        return 0.002664;

      case 5:
        return 0.0012999;

      default:
        return 0.0;
    }
  }
}
