#include "hip/hip_runtime.h"
/* NOTE: This file is directly included into solver_kernels.cu. */

#include "input.hpp"

__device__
double compute_source_term_dev(double x, double y, double z, double t, unsigned int var, unsigned int nDims, unsigned int equation)
{
  double val = 0.;
  if (equation == AdvDiff)
  {
    if (nDims == 2)
    {
      val =  -M_PI * (std::cos(M_PI * x) + M_PI * std::sin(M_PI * x) + 
             std::cos(M_PI * y) + M_PI * std::sin(M_PI * y));

      if(x*x + y*y <= 0.09)
        val = 1;
    }
    else
    {
      val =  -M_PI * (std::cos(M_PI * x) + M_PI * std::sin(M_PI * x) + 
             std::cos(M_PI * y) + M_PI * std::sin(M_PI * y) + 
             std::cos(M_PI * z) + M_PI * std::sin(M_PI * z));
    }
  }
  else
  {
    // NOT DEFINED. Cannot throw exception.
  }

  return val;
}

__device__
double get_cfl_limit_dev(int order)
{
 switch(order)
  {
    case 0:
      return 1.392;

    case 1:
      return 0.4642; 

    case 2:
      return 0.2351;

    case 3:
      return 0.1453;

    case 4:
      return 0.1000;

    case 5:
      return 0.0736;

    default:
      return 0.0;
  }

}


