#include "hip/hip_runtime.h"
/* Copyright (C) 2016 Aerospace Computing Laboratory (ACL).
 * See AUTHORS for contributors to this source code.
 *
 * This file is part of ZEFR.
 *
 * ZEFR is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ZEFR is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with ZEFR.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "mdvector_gpu.h"

#include <thrust/device_vector.h>
#include <thrust/extrema.h>

__global__
void normalize_data(mdvector_gpu<double> U_spts, double normalTol, unsigned int nSpts,
    unsigned int nEles, unsigned int nVars)
{
  const unsigned int ele = blockDim.x * blockIdx.x + threadIdx.x;

  if (ele >= nEles) return;

  for (unsigned int var = 0; var < nVars; var++)
  {
    // Find element maximum and minimum
    double uMax = U_spts(0, ele, var), uMin = U_spts(0, ele, var);

    for (unsigned int spt = 0; spt < nSpts; spt++)
    {
      uMax = max(uMax, U_spts(spt, ele, var));
      uMin = min(uMin, U_spts(spt, ele, var));
    }
    
    if (uMax - uMin > normalTol)
    {
      for (unsigned int spt = 0; spt < nSpts; spt++)
        U_spts(spt,ele,var) = (U_spts(spt,ele,var) - uMin) / (uMax - uMin);
    }
  }
}

void normalize_data_wrapper(mdvector_gpu<double>& U_spts, double normalTol, unsigned int nSpts,
    unsigned int nEles, unsigned int nVars)
{
  unsigned int threads = 128;
  unsigned int blocks = (nEles + threads - 1)/threads;

  normalize_data<<<blocks, threads>>>(U_spts, normalTol, nSpts, nEles, nVars);
}

__global__
void compute_max_sensor(mdvector_gpu<double> KS, mdvector_gpu<double> sensor, 
    unsigned int order, unsigned int nSpts, unsigned int nEles, unsigned int nVars)
{
  const unsigned int ele = blockDim.x * blockIdx.x + threadIdx.x;

  if (ele >= nEles) return;

  double max_sen = 0.0;

  for (unsigned int var = 0; var < nVars; var++)
  {
    double sen = 0.0;
    for (unsigned int row = 0; row < 2 * nSpts; row++)
    {
      KS(row, ele, var) = order * (KS(row, ele, var) * KS(row, ele, var));
      sen = max(sen, KS(row, ele, var));
    }
    max_sen = max(max_sen, sen);
  }

  sensor(ele) = max_sen;


}

void compute_max_sensor_wrapper(mdvector_gpu<double>& KS, mdvector_gpu<double>& sensor, 
    unsigned int order, double& max_sensor, unsigned int nSpts, unsigned int nEles, unsigned int nVars)
{
  unsigned int threads = 128;
  unsigned int blocks = (nEles + threads - 1)/threads;

  compute_max_sensor<<<blocks, threads>>>(KS, sensor, order, nSpts, nEles, nVars);


  /* Get max sensor value using thrust */
  thrust::device_ptr<double> s_ptr = thrust::device_pointer_cast(sensor.data());
  thrust::device_ptr<double> max_ptr = thrust::max_element(s_ptr, s_ptr + nEles);
  max_sensor = max_ptr[0];

}

__global__
void copy_filtered_solution(mdvector_gpu<double> U_spts_filt, mdvector_gpu<double> U_spts, 
    mdvector_gpu<double> sensor, double threshJ, unsigned int nSpts, unsigned int nEles, unsigned int nVars)
{
  const unsigned int ele = blockDim.x * blockIdx.x + threadIdx.x;

  if (ele >= nEles) return;

  // Check for sensor value
  if (sensor(ele) < threshJ) return; //TODO: This causes divergence. Need to address.

  for (unsigned int var = 0; var < nVars; var++)
    for (unsigned int spt = 0; spt < nSpts; spt++)
      U_spts(spt, ele, var) = U_spts_filt(spt, ele, var);

}

void copy_filtered_solution_wrapper(mdvector_gpu<double>& U_spts_filt, mdvector_gpu<double>& U_spts, 
    mdvector_gpu<double>& sensor, double threshJ, unsigned int nSpts, unsigned int nEles, unsigned int nVars)
{
  unsigned int threads = 128;
  unsigned int blocks = (nEles + threads - 1)/threads;

  copy_filtered_solution<<<blocks, threads>>>(U_spts_filt, U_spts, sensor, threshJ, nSpts, nEles, nVars);
}
