#include "hip/hip_runtime.h"
/* Copyright (C) 2016 Aerospace Computing Laboratory (ACL).
 * See AUTHORS for contributors to this source code.
 *
 * This file is part of ZEFR.
 *
 * ZEFR is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ZEFR is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with ZEFR.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "mdvector_gpu.h"

__global__
void normalize_data(mdvector_gpu<double> U_spts, double normalTol, unsigned int nSpts,
    unsigned int nEles, unsigned int nVars)
{
  const unsigned int ele = blockDim.x * blockIdx.x + threadIdx.x;

  if (ele >= nEles) return;

  for (unsigned int var = 0; var < nVars; var++)
  {
    // Find element maximum and minimum
    double uMax = U_spts(0, ele, var), uMin = U_spts(0, ele, var);

    for (unsigned int spt = 1; spt < nSpts; spt++)
    {
      uMax = max(uMax, U_spts(spt, ele, var));
      uMin = min(uMin, U_spts(spt, ele, var));
    }
    
    if (uMax - uMin > normalTol)
    {
      for (unsigned int spt = 0; spt < nSpts; spt++)
        U_spts(spt,ele,var) = (U_spts(spt,ele,var) - uMin) / (uMax - uMin);
    }
  }
}

void normalize_data_wrapper(mdvector_gpu<double>& U_spts, double normalTol, unsigned int nSpts,
    unsigned int nEles, unsigned int nVars)
{
  unsigned int threads = 128;
  unsigned int blocks = (nEles + threads - 1)/threads;

  normalize_data<<<blocks, threads>>>(U_spts, normalTol, nSpts, nEles, nVars);
}

__global__
void compute_sensor(mdvector_gpu<double> KS, mdvector_gpu<double> sensor,
    double threshJ, unsigned int nSpts, unsigned int nEles, unsigned int nVars, 
    unsigned int nDims, unsigned int nSptsKS, double Q, double epsilon)
{
  const unsigned int ele = blockDim.x * blockIdx.x + threadIdx.x;

  if (ele >= nEles) return;

  double max_sen = 0.0;

  for (unsigned int var = 0; var < nVars; var++)
  {
    double sen = 0.0;
    for (unsigned int row = 0; row < nDims * nSptsKS; row++)
    {
      KS(row, ele, var) = pow(1.0/epsilon , Q/2.0) * pow(abs(KS(row, ele, var)), Q);
      sen = max(sen, KS(row, ele, var));
    }
    max_sen = max(max_sen, sen);
  }

  sensor(ele) = max_sen;
}

void compute_sensor_wrapper(mdvector_gpu<double>& KS, mdvector_gpu<double>& sensor,
    double threshJ, unsigned int nSpts, unsigned int nEles, unsigned int nVars, 
    unsigned int nDims, unsigned int nSptsKS, double Q, double epsilon)
{
  unsigned int threads = 192;
  unsigned int blocks = (nEles + threads - 1)/threads;

  compute_sensor<<<blocks, threads>>>(KS, sensor, threshJ, nSpts, nEles, nVars, nDims, nSptsKS, Q, epsilon);

}

__global__
void copy_filtered_solution(mdvector_gpu<double> U_filt, mdvector_gpu<double> U_spts,
    mdvector_gpu<double> sensor, double threshJ, unsigned int nSpts, unsigned int nEles, unsigned int nVars)
{
  const unsigned int ele = blockDim.x * blockIdx.x + threadIdx.x;

  if (ele >= nEles) return;

  // Check for sensor value
  if (sensor(ele) < threshJ)
    return; 

  for (unsigned int var = 0; var < nVars; var++)
    for (unsigned int spt = 0; spt < nSpts; spt++)
      U_spts(spt, ele, var) = U_filt(spt, ele, var);

}

void copy_filtered_solution_wrapper(mdvector_gpu<double>& U_filt, mdvector_gpu<double>& U_spts,
    mdvector_gpu<double>& sensor, double threshJ, unsigned int nSpts, unsigned int nEles, unsigned int nVars)
{
  unsigned int threads = 192;
  unsigned int blocks = (nEles + threads - 1)/threads;

  copy_filtered_solution<<<blocks, threads>>>(U_filt, U_spts, sensor, threshJ, nSpts, nEles, nVars);
}

